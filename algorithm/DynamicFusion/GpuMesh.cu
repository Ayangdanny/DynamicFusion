#include "hip/hip_runtime.h"
#include "GpuMesh.h"

namespace dfusion
{
	__device__ __forceinline__ PixelRGBA copy_uchar4_pixelRGBA(uchar4 a)
	{
		PixelRGBA p;
		p.r = a.x;
		p.g = a.y;
		p.b = a.z;
		p.a = a.w;
		return p;
	}

	__global__ void copy_invert_y_kernel(PtrStepSz<uchar4> gldata,
		PtrStepSz<PixelRGBA> img)
	{
		int u = threadIdx.x + blockIdx.x * blockDim.x;
		int v = threadIdx.y + blockIdx.y * blockDim.y;

		if (u >= img.cols || v >= img.rows)
			return;

		img(v, u) = copy_uchar4_pixelRGBA(gldata(img.rows-1-v, u));
	}

	void GpuMesh::copy_invert_y(const uchar4* gldata, ColorMap& img)
	{
		dim3 block(32, 8);
		dim3 grid(1, 1, 1);
		grid.x = divUp(m_width, block.x);
		grid.y = divUp(m_height, block.y);

		PtrStepSz<uchar4> gldataptr;
		gldataptr.data = (uchar4*)gldata;
		gldataptr.rows = m_height;
		gldataptr.cols = m_width;
		gldataptr.step = m_width*sizeof(uchar4);

		img.create(m_height, m_width);

		copy_invert_y_kernel << <grid, block >> >(gldataptr, img);
		cudaSafeCall(hipGetLastError(), "GpuMesh::copy");
		hipDeviceSynchronize();
	}
}