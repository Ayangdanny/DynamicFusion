#include "hip/hip_runtime.h"
#include "GpuMesh.h"

namespace dfusion
{
	__device__ __forceinline__ PixelRGBA copy_uchar4_pixelRGBA(uchar4 a)
	{
		PixelRGBA p;
		p.r = a.x;
		p.g = a.y;
		p.b = a.z;
		p.a = a.w;
		return p;
	}

	__global__ void copy_invert_y_kernel(PtrStepSz<uchar4> gldata,
		PtrStepSz<PixelRGBA> img)
	{
		int u = threadIdx.x + blockIdx.x * blockDim.x;
		int v = threadIdx.y + blockIdx.y * blockDim.y;

		if (u >= img.cols || v >= img.rows)
			return;

		img(v, u) = copy_uchar4_pixelRGBA(gldata(img.rows-1-v, u));
	}

	void GpuMesh::copy_invert_y(const uchar4* gldata, ColorMap& img)
	{
		dim3 block(32, 8);
		dim3 grid(1, 1, 1);
		grid.x = divUp(m_width, block.x);
		grid.y = divUp(m_height, block.y);

		PtrStepSz<uchar4> gldataptr;
		gldataptr.data = (uchar4*)gldata;
		gldataptr.rows = m_height;
		gldataptr.cols = m_width;
		gldataptr.step = m_width*sizeof(uchar4);

		img.create(m_height, m_width);

		copy_invert_y_kernel << <grid, block >> >(gldataptr, img);
		cudaSafeCall(hipGetLastError(), "GpuMesh::copy_invert_y");
		hipDeviceSynchronize();
	}

	__global__ void copy_gldepth_to_depthmap_kernel(PtrStepSz<uchar4> gldata,
		PtrStepSz<depthtype> img, float s1, float s2, float camNear)
	{
		int u = threadIdx.x + blockIdx.x * blockDim.x;
		int v = threadIdx.y + blockIdx.y * blockDim.y;

		if (u >= img.cols || v >= img.rows)
			return;

		uchar4 p = gldata(img.rows - 1 - v, u);
		float val = (p.x + (p.y << 8) + p.z / 255.f) / 65525.f;
		val = s1 / (2 * val - 1 + s2) * 1000.f;
		if (val <= camNear*1000.f)
			val = 0;
		img(v, u) = val;
	}

	void GpuMesh::copy_gldepth_to_depthmap(const uchar4* gldata, DepthMap& depth, 
		float s1, float s2, float camNear)
	{
		dim3 block(32, 8);
		dim3 grid(1, 1, 1);
		grid.x = divUp(m_width, block.x);
		grid.y = divUp(m_height, block.y);

		PtrStepSz<uchar4> gldataptr;
		gldataptr.data = (uchar4*)gldata;
		gldataptr.rows = m_height;
		gldataptr.cols = m_width;
		gldataptr.step = m_width*sizeof(uchar4);

		depth.create(m_height, m_width);

		copy_gldepth_to_depthmap_kernel << <grid, block >> >(gldataptr, depth, s1, s2, camNear);
		cudaSafeCall(hipGetLastError(), "GpuMesh::copy_gldepth_to_depthmap");
		hipDeviceSynchronize();
	}
}