#include "hip/hip_runtime.h"
#include "GpuMesh.h"
#include "WarpField.h"
#include "TsdfVolume.h"
#include <hip/hip_vector_types.h>
namespace dfusion
{
	__device__ __forceinline__ PixelRGBA copy_uchar4_pixelRGBA(uchar4 a)
	{
		PixelRGBA p;
		p.r = a.x;
		p.g = a.y;
		p.b = a.z;
		p.a = a.w;
		return p;
	}

	__device__ __forceinline__ PixelRGBA copy_float4_pixelRGBA(float4 a)
	{
		PixelRGBA p;
		p.r = a.x * 255;
		p.g = a.y * 255;
		p.b = a.z * 255;
		p.a = a.w * 255;
		return p;
	}

	__device__ __forceinline__ WarpField::IdxType get_by_arrayid(WarpField::KnnIdx knn, int i)
	{
		return ((WarpField::IdxType*)(&knn))[i];
	}

	__global__ void copy_invert_y_kernel(PtrStepSz<float4> gldata,
		PtrStepSz<PixelRGBA> img)
	{
		int u = threadIdx.x + blockIdx.x * blockDim.x;
		int v = threadIdx.y + blockIdx.y * blockDim.y;

		if (u >= img.cols || v >= img.rows)
			return;

		img(v, u) = copy_float4_pixelRGBA(gldata(img.rows-1-v, u));
	}

	void GpuMesh::copy_invert_y(const float4* gldata, ColorMap& img)
	{
		dim3 block(32, 8);
		dim3 grid(1, 1, 1);
		grid.x = divUp(img.cols(), block.x);
		grid.y = divUp(img.rows(), block.y);

		PtrStepSz<float4> gldataptr;
		gldataptr.data = (float4*)gldata;
		gldataptr.rows = img.rows();
		gldataptr.cols = img.cols();
		gldataptr.step = img.cols()*sizeof(float4);

		copy_invert_y_kernel << <grid, block >> >(gldataptr, img);
		cudaSafeCall(hipGetLastError(), "GpuMesh::copy_invert_y");
		hipDeviceSynchronize();
	}

	__global__ void copy_gldepth_to_depthmap_kernel(PtrStepSz<float4> gldata,
		PtrStepSz<depthtype> img, float s1, float s2, float camNear)
	{
		int u = threadIdx.x + blockIdx.x * blockDim.x;
		int v = threadIdx.y + blockIdx.y * blockDim.y;

		if (u >= img.cols || v >= img.rows)
			return;

		float4 p = gldata(img.rows - 1 - v, u);
		float val = p.x;
		val = s1 / (2 * val - 1 + s2) * 1000.f;
		if (val <= camNear*1000.f)
			val = 0;
		img(v, u) = val;
	}

	void GpuMesh::copy_gldepth_to_depthmap(const float4* gldata, DepthMap& depth, 
		float s1, float s2, float camNear)
	{
		dim3 block(32, 8);
		dim3 grid(1, 1, 1);
		grid.x = divUp(depth.cols(), block.x);
		grid.y = divUp(depth.rows(), block.y);

		PtrStepSz<float4> gldataptr;
		gldataptr.data = (float4*)gldata;
		gldataptr.rows = depth.rows();
		gldataptr.cols = depth.cols();
		gldataptr.step = depth.cols()*sizeof(float4);

		copy_gldepth_to_depthmap_kernel << <grid, block >> >(gldataptr, depth, s1, s2, camNear);
		cudaSafeCall(hipGetLastError(), "GpuMesh::copy_gldepth_to_depthmap");
		hipDeviceSynchronize();
	}


	__global__ void copy_canoview_kernel(PtrStepSz<float4> gldata,
		PtrStepSz<float4> map)
	{
		int u = threadIdx.x + blockIdx.x * blockDim.x;
		int v = threadIdx.y + blockIdx.y * blockDim.y;

		if (u >= map.cols || v >= map.rows)
			return;

		map(v,u) = gldata(map.rows - 1 - v, u);
	}

	void GpuMesh::copy_canoview(const float4* gldata, DeviceArray2D<float4>& map)
	{
		dim3 block(32, 8);
		dim3 grid(1, 1, 1);
		grid.x = divUp(map.cols(), block.x);
		grid.y = divUp(map.rows(), block.y);

		PtrStepSz<float4> gldataptr;
		gldataptr.data = (float4*)gldata;
		gldataptr.rows = map.rows();
		gldataptr.cols = map.cols();
		gldataptr.step = map.cols()*sizeof(float4);

		copy_canoview_kernel << <grid, block >> >(gldataptr, map);
		cudaSafeCall(hipGetLastError(), "GpuMesh::copy_canoview");
		hipDeviceSynchronize();
	}


	__global__ void copy_warp_node_to_gl_buffer_kernel(
		float4* gldata, int* glindex,
		Tbx::Transfo trans, const float4* nodes, 
		hipTextureObject_t knnTex, hipTextureObject_t nodesDqVwTex, float3 origion, float invVoxelSize,
		const WarpField::KnnIdx* nodesKnn, 
		int n, int node_start_id)
	{
		int i = threadIdx.x + blockIdx.x * blockDim.x;

		if (i < n)
		{
			float4 node = nodes[i*3+2];

			Tbx::Dual_quat_cu dq_blend = WarpField::calc_dual_quat_blend_on_p(knnTex,
				nodesDqVwTex, make_float3(node.x, node.y, node.z), origion, invVoxelSize);
			Tbx::Vec3 t = trans * dq_blend.transform(Tbx::Point3(node.x, node.y, node.z));
			node.x = t.x;
			node.y = t.y;
			node.z = t.z;

			gldata[i] = node;

			if (glindex && nodesKnn)
			{
				WarpField::IdxType* knnIdx = (WarpField::IdxType*)&(nodesKnn[i]);
				int start = 2 * WarpField::KnnK * i;
				for (int k = 0; k < WarpField::KnnK; k++)
				{
					int nn = knnIdx[k];
					if (nn >= WarpField::MaxNodeNum || nn < 0)
						nn = i-WarpField::MaxNodeNum;
					glindex[start + k*2 + 0] = i + node_start_id;
					glindex[start + k*2 + 1] = nn + node_start_id + WarpField::MaxNodeNum;
				}
			}
		}
	}

	void GpuMesh::copy_warp_node_to_gl_buffer(float4* gldata, const WarpField* warpField)
	{
		int* glindex = (int*)(gldata + WarpField::MaxNodeNum * WarpField::GraphLevelNum);
		int node_start_id = 0;

		hipTextureObject_t nodesDqVwTex = warpField->bindNodesDqVwTexture();
		hipTextureObject_t knnTex = warpField->bindKnnFieldTexture();
		float3 origion = warpField->getVolume()->getOrigion();
		float invVsz = 1.f/warpField->getVolume()->getVoxelSize();

		for (int lv = 0; lv < warpField->getNumLevels(); lv++, 
			gldata += WarpField::MaxNodeNum, 
			node_start_id += WarpField::MaxNodeNum,
			glindex += WarpField::MaxNodeNum*2*WarpField::KnnK)
		{
			int n = warpField->getNumNodesInLevel(lv);
			if (n == 0)
				return;
			const float4* nodes = warpField->getNodesDqVwPtr(lv);
			const WarpField::KnnIdx* indices = nullptr;
			if (lv < warpField->getNumLevels() - 1)
				indices = warpField->getNodesEdgesPtr(lv);
			Tbx::Transfo tr = warpField->get_rigidTransform();
			dim3 block(32);
			dim3 grid(divUp(n, block.x));
			copy_warp_node_to_gl_buffer_kernel << <grid, block >> >(
				gldata, glindex, tr, nodes, 
				knnTex, nodesDqVwTex, origion, invVsz,
				indices, n, node_start_id);
			cudaSafeCall(hipGetLastError(), "GpuMesh::copy_warp_node_to_gl_buffer");
		}
		warpField->unBindKnnFieldTexture(knnTex);
		warpField->unBindNodesDqVwTexture(nodesDqVwTex);
	}

}