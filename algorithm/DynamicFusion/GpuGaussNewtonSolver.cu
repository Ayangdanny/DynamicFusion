#include "hip/hip_runtime.h"
#include "GpuGaussNewtonSolver.h"
#include "device_utils.h"
#include "cudpp\cudpp_wrapper.h"
#include "cudpp\thrust_wrapper.h"
#include "cudpp\ModerGpuWrapper.h"
namespace dfusion
{
	texture<WarpField::KnnIdx, hipTextureType1D, hipReadModeElementType> g_nodesKnnTex;
	texture<float4, hipTextureType1D, hipReadModeElementType> g_nodesVwTex;
	texture<float, hipTextureType1D, hipReadModeElementType> g_twistTex;
	texture<float, hipTextureType1D, hipReadModeElementType> g_JrtValTex;
	texture<int, hipTextureType1D, hipReadModeElementType> g_JrtCidxTex;

	__device__ __forceinline__ float4 get_nodesVw(int i)
	{
		return tex1Dfetch(g_nodesVwTex, i);
	}

	__device__ __forceinline__ WarpField::KnnIdx get_nodesKnn(int i)
	{
		return tex1Dfetch(g_nodesKnnTex, i);
	}

	__device__ __forceinline__ void get_twist(int i, Tbx::Vec3& r, Tbx::Vec3& t)
	{
		int i6 = i * 6;
		r.x = tex1Dfetch(g_twistTex, i6++);
		r.y = tex1Dfetch(g_twistTex, i6++);
		r.z = tex1Dfetch(g_twistTex, i6++);
		t.x = tex1Dfetch(g_twistTex, i6++);
		t.y = tex1Dfetch(g_twistTex, i6++);
		t.z = tex1Dfetch(g_twistTex, i6++);
	}

	__device__ __forceinline__ float get_JrtVal(int i)
	{
		return tex1Dfetch(g_JrtValTex, i);
	}
	__device__ __forceinline__ int get_JrtCidx(int i)
	{
		return tex1Dfetch(g_JrtCidxTex, i);
	}

	// map the lower part to full 6x6 matrix
	__constant__ int g_lower_2_full_6x6[21] = {
		0,
		6, 7,
		12, 13, 14,
		18, 19, 20, 21,
		24, 25, 26, 27, 28,
		30, 31, 32, 33, 34, 35
	};
	__constant__ int g_lower_2_rowShift_6x6[21] = {
		0,
		1, 1,
		2, 2, 2,
		3, 3, 3, 3,
		4, 4, 4, 4, 4,
		5, 5, 5, 5, 5, 5
	};
	__constant__ int g_lower_2_colShift_6x6[21] = {
		0,
		0, 1,
		0, 1, 2,
		0, 1, 2, 3,
		0, 1, 2, 3, 4,
		0, 1, 2, 3, 4, 5
	};
	__constant__ int g_lfull_2_lower_6x6[6][6] = {
		{ 0, -1, -1, -1, -1, -1 },
		{ 1, 2, -1, -1, -1, -1 },
		{ 3, 4, 5, -1, -1, -1 },
		{ 6, 7, 8, 9, -1, -1 },
		{ 10, 11, 12, 13, 14, -1 },
		{ 15, 16, 17, 18, 19, 20 },
	};

#define D_1_DIV_6 0.166666667

	__device__ __forceinline__ float3 read_float3_4(float4 a)
	{
		return make_float3(a.x, a.y, a.z);
	}

	__device__ __forceinline__ float sqr(float a)
	{
		return a*a;
	}

	__device__ __forceinline__ float pow3(float a)
	{
		return a*a*a;
	}

	__device__ __forceinline__ float sign(float a)
	{
		return (a>0.f) - (a<0.f);
	}

	__device__ __forceinline__ WarpField::IdxType& knn_k(WarpField::KnnIdx& knn, int k)
	{
		return ((WarpField::IdxType*)(&knn))[k];
	}
	__device__ __forceinline__ const WarpField::IdxType& knn_k(const WarpField::KnnIdx& knn, int k)
	{
		return ((WarpField::IdxType*)(&knn))[k];
	}

	__device__ __forceinline__ void sort_knn(WarpField::KnnIdx& knn)
	{
		for (int i = 1; i < WarpField::KnnK; i++)
		{
			WarpField::IdxType x = knn_k(knn,i);
			int	j = i;
			while (j > 0 && knn_k(knn, j - 1) > x)
			{
				knn_k(knn, j) = knn_k(knn, j - 1);
				j = j - 1;
			}
			knn_k(knn, j) = x;
		}
	}

#pragma region --bind textures
	void GpuGaussNewtonSolver::bindTextures()
	{
		if (1)
		{
			size_t offset;
			hipChannelFormatDesc desc = hipCreateChannelDesc<WarpField::KnnIdx>();
			hipBindTexture(&offset, &g_nodesKnnTex, m_nodesKnn.ptr(), &desc,
				m_nodesKnn.size() * sizeof(WarpField::KnnIdx));
			if (offset != 0)
				throw std::exception("GpuGaussNewtonSolver::bindTextures(): non-zero-offset error!");
		}
		if (1)
		{
			size_t offset;
			hipChannelFormatDesc desc = hipCreateChannelDesc<float4>();
			hipBindTexture(&offset, &g_nodesVwTex, m_nodesVw.ptr(), &desc,
				m_nodesVw.size() * sizeof(float4));
			if (offset != 0)
				throw std::exception("GpuGaussNewtonSolver::bindTextures(): non-zero-offset error!");
		}
		if (1)
		{
			size_t offset;
			hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
			hipBindTexture(&offset, &g_twistTex, m_twist.ptr(), &desc,
				m_twist.size() * sizeof(float));
			if (offset != 0)
				throw std::exception("GpuGaussNewtonSolver::bindTextures(): non-zero-offset error!");
		}
		if (1)
		{
			size_t offset;
			hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
			hipBindTexture(&offset, &g_JrtValTex, m_Jrt_val.ptr(), &desc,
				m_Jrt_val.size() * sizeof(float));
			if (offset != 0)
				throw std::exception("GpuGaussNewtonSolver::bindTextures(): non-zero-offset error!");
		}
		if (1)
		{
			size_t offset;
			hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
			hipBindTexture(&offset, &g_JrtCidxTex, m_Jrt_ColIdx.ptr(), &desc,
				m_Jrt_ColIdx.size() * sizeof(int));
			if (offset != 0)
				throw std::exception("GpuGaussNewtonSolver::bindTextures(): non-zero-offset error!");
		}
	}

	void GpuGaussNewtonSolver::unBindTextures()
	{
		hipUnbindTexture(g_twistTex);
		hipUnbindTexture(g_nodesVwTex);
		hipUnbindTexture(g_nodesKnnTex);
	}
#pragma endregion

#pragma region --calc data term

//#define ENABLE_GPU_DUMP_DEBUG

	struct DataTermCombined
	{
		typedef WarpField::KnnIdx KnnIdx;
		typedef WarpField::IdxType IdxType;
		enum
		{
			CTA_SIZE_X = GpuGaussNewtonSolver::CTA_SIZE_X,
			CTA_SIZE_Y = GpuGaussNewtonSolver::CTA_SIZE_Y,
			CTA_SIZE = CTA_SIZE_X * CTA_SIZE_Y,
			KnnK = WarpField::KnnK,
			VarPerNode = GpuGaussNewtonSolver::VarPerNode,
			VarPerNode2 = VarPerNode*VarPerNode,
			LowerPartNum = GpuGaussNewtonSolver::LowerPartNum,
		};

		PtrStep<float4> vmap_live;
		PtrStep<float4> nmap_live;
		PtrStep<float4> vmap_warp;
		PtrStep<float4> nmap_warp;
		PtrStep<float4> vmap_cano;
		PtrStep<float4> nmap_cano;
		PtrStep<KnnIdx> vmapKnn;
		float* Hd_;
		float* g_;

		Intr intr;
		Tbx::Transfo Tlw;

		int imgWidth;
		int imgHeight;
		int nNodes;

		float distThres;
		float angleThres;
		float psi_data;

#ifdef ENABLE_GPU_DUMP_DEBUG
		// for debug
		float* debug_buffer_pixel_sum2;
		float* debug_buffer_pixel_val;
#endif

		__device__ __forceinline__ float data_term_penalty(float f)const
		{
			return f * sqr(max(0.f, 1.f - sqr(f / psi_data)));
			//// the robust Tukey penelty gradient
			//if (abs(f) <= psi_data)
			//	return f * sqr(1 - sqr(f / psi_data));
			//else
			//	return 0;
		}

		__device__ __forceinline__ float trace_AtB(Tbx::Transfo A, Tbx::Transfo B)const
		{
			float sum = 0;
			for (int i = 0; i < 16; i++)
				sum += A[i] * B[i];
			return sum;
		}

		__device__ __forceinline__ Tbx::Transfo compute_p_f_p_T(const Tbx::Vec3& n,
			const Tbx::Point3& v, const Tbx::Point3& vl, const Tbx::Dual_quat_cu& dq)const
		{
			//Tbx::Transfo T = Tlw*dq.to_transformation_after_normalize();
			//Tbx::Transfo nvt = outer_product(n, v);
			//Tbx::Transfo vlnt = outer_product(n, vl).transpose();
			//Tbx::Transfo p_f_p_T = T*(nvt + nvt.transpose()) - vlnt;
			Tbx::Vec3 Tn = Tlw*dq.rotate(n);
			Tbx::Point3 Tv(Tlw*dq.transform(v) - vl);
			return Tbx::Transfo(
				Tn.x*v.x + n.x*Tv.x, Tn.x*v.y + n.y*Tv.x, Tn.x*v.z + n.z*Tv.x, Tn.x,
				Tn.y*v.x + n.x*Tv.y, Tn.y*v.y + n.y*Tv.y, Tn.y*v.z + n.z*Tv.y, Tn.y,
				Tn.z*v.x + n.x*Tv.z, Tn.z*v.y + n.y*Tv.z, Tn.z*v.z + n.z*Tv.z, Tn.z,
				n.x, n.y, n.z, 0
				);
		}

		__device__ __forceinline__ Tbx::Transfo p_T_p_alphak_func(const Tbx::Dual_quat_cu& p_qk_p_alpha,
			const Tbx::Dual_quat_cu& dq_bar, const Tbx::Dual_quat_cu& dq, float inv_norm_dq_bar, float wk_k)const
		{
			Tbx::Transfo p_T_p_alphak = Tbx::Transfo::empty();

			float pdot = dq_bar.get_non_dual_part().dot(p_qk_p_alpha.get_non_dual_part())
				* sqr(inv_norm_dq_bar);

			//// evaluate p_dqi_p_alphak, heavily hard code here
			//// this hard code is crucial to the performance 
			// 0:
			// (0, -z0, y0, x1,
			// z0, 0, -x0, y1,
			//-y0, x0, 0, z1,
			// 0, 0, 0, 0) * 2;
			float p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[0] - dq_bar[0] * pdot
				);
			p_T_p_alphak[1] += -dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[2] += dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[3] += dq[5] * p_dqi_p_alphak;
			p_T_p_alphak[4] += dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[6] += -dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[7] += dq[6] * p_dqi_p_alphak;
			p_T_p_alphak[8] += -dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[9] += dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[11] += dq[7] * p_dqi_p_alphak;

			// 1
			//( 0, y0, z0, -w1,
			//	y0, -2 * x0, -w0, -z1,
			//	z0, w0, -2 * x0, y1,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[1] - dq_bar[1] * pdot
				);
			p_T_p_alphak[1] += dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[2] += dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[3] += -dq[4] * p_dqi_p_alphak;
			p_T_p_alphak[4] += dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[5] += -dq[1] * p_dqi_p_alphak * 2;
			p_T_p_alphak[6] += -dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[7] += -dq[7] * p_dqi_p_alphak;
			p_T_p_alphak[8] += dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[9] += dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[10] += -dq[1] * p_dqi_p_alphak * 2;
			p_T_p_alphak[11] += dq[6] * p_dqi_p_alphak;

			// 2.
			// (-2 * y0, x0, w0, z1,
			//	x0, 0, z0, -w1,
			//	-w0, z0, -2 * y0, -x1,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[2] - dq_bar[2] * pdot
				);
			p_T_p_alphak[0] += -dq[2] * p_dqi_p_alphak * 2;
			p_T_p_alphak[1] += dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[2] += dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[3] += dq[7] * p_dqi_p_alphak;
			p_T_p_alphak[4] += dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[6] += dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[7] += -dq[4] * p_dqi_p_alphak;
			p_T_p_alphak[8] += -dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[9] += dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[10] += -dq[2] * p_dqi_p_alphak * 2;
			p_T_p_alphak[11] += -dq[5] * p_dqi_p_alphak;

			// 3.
			// (-2 * z0, -w0, x0, -y1,
			//	w0, -2 * z0, y0, x1,
			//	x0, y0, 0, -w1,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[3] - dq_bar[3] * pdot
				);
			p_T_p_alphak[0] += -dq[3] * p_dqi_p_alphak * 2;
			p_T_p_alphak[1] += -dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[2] += dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[3] += -dq[6] * p_dqi_p_alphak;
			p_T_p_alphak[4] += dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[5] += -dq[3] * p_dqi_p_alphak * 2;
			p_T_p_alphak[6] += dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[7] += dq[5] * p_dqi_p_alphak;
			p_T_p_alphak[8] += dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[9] += dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[11] += -dq[4] * p_dqi_p_alphak;

			// 4.
			//( 0, 0, 0, -x0,
			//	0, 0, 0, -y0,
			//	0, 0, 0, -z0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[4] - dq_bar[4] * pdot
				);
			p_T_p_alphak[3] += -dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[7] += -dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[11] += -dq[3] * p_dqi_p_alphak;

			// 5. 
			// (0, 0, 0, w0,
			//	0, 0, 0, z0,
			//	0, 0, 0, -y0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[5] - dq_bar[5] * pdot
				);
			p_T_p_alphak[3] += dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[7] += dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[11] += -dq[2] * p_dqi_p_alphak;

			// 6. 
			// (0, 0, 0, -z0,
			//	0, 0, 0, w0,
			//	0, 0, 0, x0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[6] - dq_bar[6] * pdot
				);
			p_T_p_alphak[3] += -dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[7] += dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[11] += dq[1] * p_dqi_p_alphak;

			// 7.
			// (0, 0, 0, y0,
			//	0, 0, 0, -x0,
			//	0, 0, 0, w0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[7] - dq_bar[7] * pdot
				);
			p_T_p_alphak[3] += dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[7] += -dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[11] += dq[0] * p_dqi_p_alphak;

			p_T_p_alphak = Tlw * p_T_p_alphak;
			return p_T_p_alphak;
		}

		__device__ __forceinline__ bool search(int x, int y, Tbx::Point3& vl) const
		{
			float3 vwarp = read_float3_4(vmap_warp(y, x));
			float3 nwarp = read_float3_4(nmap_warp(y, x));

			if (isnan(nwarp.x))
				return false;

			float3 uvd = intr.xyz2uvd(vwarp);
			int2 ukr = make_int2(uvd.x + 0.5, uvd.y + 0.5);

			// we use opengl coordinate, thus world.z should < 0
			if (ukr.x < 0 || ukr.y < 0 || ukr.x >= imgWidth || ukr.y >= imgHeight || vwarp.z >= 0)
				return false;

			float3 vlive = read_float3_4(vmap_live[ukr.y*imgWidth + ukr.x]);
			float3 nlive = read_float3_4(nmap_live[ukr.y*imgWidth + ukr.x]);
			if (isnan(nlive.x))
				return false;

			float dist = norm(vwarp - vlive);
			if (!(dist <= distThres))
				return false;

			float sine = norm(cross(nwarp, nlive));
			if (!(sine < angleThres))
				return false;

			vl = Tbx::Point3(vlive.x, vlive.y, vlive.z);

			return true;
		}

		__device__ __forceinline__ void operator () () const
		{
			const int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
			const int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

			Tbx::Point3 vl;
			bool found_coresp = false;
			if (x < imgWidth && y < imgHeight)
				found_coresp = search(x, y, vl);

			if (found_coresp)
			{
				Tbx::Point3 v(convert(read_float3_4(vmap_cano(y, x))));
				Tbx::Vec3 n(convert(read_float3_4(nmap_cano(y, x))));

				const KnnIdx knn = vmapKnn(y, x);
				Tbx::Dual_quat_cu dq(Tbx::Quat_cu(0, 0, 0, 0), Tbx::Quat_cu(0, 0, 0, 0));
				Tbx::Dual_quat_cu dqk_0;
				float wk[KnnK];
				for (int k = 0; k < KnnK; k++)
				{
					int knnNodeId = knn_k(knn, k);
					if (knnNodeId < nNodes)
					{
						Tbx::Vec3 r, t;
						get_twist(knnNodeId, r, t);
						float4 nodeVw = get_nodesVw(knnNodeId);
						Tbx::Point3 nodesV(convert(read_float3_4(nodeVw)));
						float invNodesW = nodeVw.w;
						Tbx::Dual_quat_cu dqk_k;
						dqk_k.from_twist(r, t);
						// note: we store inv radius as vw.w, thus using * instead of / here
						wk[k] = __expf(-(v - nodesV).dot(v - nodesV)*(2 * invNodesW * invNodesW));
						if (k == 0)
							dqk_0 = dqk_k;
						if (dqk_0.get_non_dual_part().dot(dqk_k.get_non_dual_part()) < 0)
							wk[k] = -wk[k];
						dq = dq + dqk_k * wk[k];
					}
				}

				Tbx::Dual_quat_cu dq_bar = dq;
				float inv_norm_dq_bar = 1.f / dq_bar.get_non_dual_part().norm();
				dq = dq * inv_norm_dq_bar; // normalize

				// the grad energy f
				const float f = data_term_penalty((Tlw*dq.rotate(n)).dot(Tlw*dq.transform(v) - vl));

				// paitial_f_partial_T
				const Tbx::Transfo p_f_p_T = compute_p_f_p_T(n, v, vl, dq);

				for (int knnK = 0; knnK < KnnK; knnK++)
				{
					float p_f_p_alpha[VarPerNode];
					int knnNodeId = knn_k(knn, knnK);
					float wk_k = wk[knnK] * inv_norm_dq_bar * 2;
					if (knnNodeId < nNodes)
					{
						//// comput partial_T_partial_alphak, hard code here.
						Tbx::Dual_quat_cu p_qk_p_alpha;
						Tbx::Transfo p_T_p_alphak;
						Tbx::Vec3 t, r;
						float b, c;
						Tbx::Quat_cu q1;
						get_twist(knnNodeId, r, t);
						{
							float n = r.norm();
							float sin_n, cos_n;
							sincos(n, &sin_n, &cos_n);
							b = n > Tbx::Dual_quat_cu::epsilon() ? sin_n / n : 1;
							c = n > Tbx::Dual_quat_cu::epsilon() ? (cos_n - b) / (n*n) : 0;
							q1 = Tbx::Quat_cu(cos_n*0.5f, r.x*b*0.5f, r.y*b*0.5f, r.z*b*0.5f);
						}

						// alpha0
						p_qk_p_alpha[0] = -r[0] * b;
						p_qk_p_alpha[1] = b + r[0] * r[0] * c;
						p_qk_p_alpha[2] = r[0] * r[1] * c;
						p_qk_p_alpha[3] = r[0] * r[2] * c;
						p_qk_p_alpha = Tbx::Dual_quat_cu::dual_quat_from(p_qk_p_alpha.get_non_dual_part(), t);
						p_T_p_alphak = p_T_p_alphak_func(p_qk_p_alpha, dq_bar, dq,
							inv_norm_dq_bar, wk_k);
						p_f_p_alpha[0] = trace_AtB(p_f_p_T, p_T_p_alphak);

						// alpha1
						p_qk_p_alpha[0] = -r[1] * b;
						p_qk_p_alpha[1] = r[1] * r[0] * c;
						p_qk_p_alpha[2] = b + r[1] * r[1] * c;
						p_qk_p_alpha[3] = r[1] * r[2] * c;
						p_qk_p_alpha = Tbx::Dual_quat_cu::dual_quat_from(p_qk_p_alpha.get_non_dual_part(), t);
						p_T_p_alphak = p_T_p_alphak_func(p_qk_p_alpha, dq_bar, dq,
							inv_norm_dq_bar, wk_k);
						p_f_p_alpha[1] = trace_AtB(p_f_p_T, p_T_p_alphak);

						// alpha2
						p_qk_p_alpha[0] = -r[2] * b;
						p_qk_p_alpha[1] = r[2] * r[0] * c;
						p_qk_p_alpha[2] = r[2] * r[1] * c;
						p_qk_p_alpha[3] = b + r[2] * r[2] * c;
						p_qk_p_alpha = Tbx::Dual_quat_cu::dual_quat_from(p_qk_p_alpha.get_non_dual_part(), t);
						p_T_p_alphak = p_T_p_alphak_func(p_qk_p_alpha, dq_bar, dq,
							inv_norm_dq_bar, wk_k);
						p_f_p_alpha[2] = trace_AtB(p_f_p_T, p_T_p_alphak);

						// alpha3
						p_qk_p_alpha = Tbx::Dual_quat_cu(Tbx::Quat_cu(0, 0, 0, 0),
							Tbx::Quat_cu(-q1[1], q1[0], -q1[3], q1[2]));
						p_T_p_alphak = p_T_p_alphak_func(p_qk_p_alpha, dq_bar, dq,
							inv_norm_dq_bar, wk_k);
						p_f_p_alpha[3] = trace_AtB(p_f_p_T, p_T_p_alphak);

						// alpha4
						p_qk_p_alpha = Tbx::Dual_quat_cu(Tbx::Quat_cu(0, 0, 0, 0),
							Tbx::Quat_cu(-q1[2], q1[3], q1[0], -q1[1]));
						p_T_p_alphak = p_T_p_alphak_func(p_qk_p_alpha, dq_bar, dq,
							inv_norm_dq_bar, wk_k);
						p_f_p_alpha[4] = trace_AtB(p_f_p_T, p_T_p_alphak);

						// alpha5
						p_qk_p_alpha = Tbx::Dual_quat_cu(Tbx::Quat_cu(0, 0, 0, 0),
							Tbx::Quat_cu(-q1[3], -q1[2], q1[1], q1[0]));
						p_T_p_alphak = p_T_p_alphak_func(p_qk_p_alpha, dq_bar, dq,
							inv_norm_dq_bar, wk_k);
						p_f_p_alpha[5] = trace_AtB(p_f_p_T, p_T_p_alphak);

						//// reduce--------------------------------------------------
						int shift = knnNodeId * VarPerNode2;
						int shift_g = knnNodeId * VarPerNode;
						for (int i = 0; i < VarPerNode; ++i)
						{
#pragma unroll
							for (int j = 0; j <= i; ++j)
							{
								atomicAdd(&Hd_[shift + j], p_f_p_alpha[i] * p_f_p_alpha[j]);
#ifdef ENABLE_GPU_DUMP_DEBUG
// debug
if (knnNodeId == 390 && i == 5 && j == 1
	&& debug_buffer_pixel_sum2 && debug_buffer_pixel_val
	)
{
	for (int k = 0; k < VarPerNode; k++)
		debug_buffer_pixel_val[(y*imgWidth + x)*VarPerNode + k] =
		p_f_p_alpha[k];
	debug_buffer_pixel_sum2[y*imgWidth + x] = Hd_[shift + j];
}
#endif
							}
							atomicAdd(&g_[shift_g + i], p_f_p_alpha[i] * f);
							shift += VarPerNode;
						}// end for i
					}// end if knnNodeId < nNodes
				}// end for knnK
			}// end if found corr
		}// end function ()
	};

	__global__ void dataTermCombinedKernel(const DataTermCombined cs)
	{
		cs();
	}

	void GpuGaussNewtonSolver::calcDataTerm()
	{
		DataTermCombined cs;
		cs.angleThres = m_param->fusion_nonRigid_angleThreSin;
		cs.distThres = m_param->fusion_nonRigid_distThre;
		cs.Hd_ = m_Hd;
		cs.g_ = m_g;
		cs.imgHeight = m_vmap_cano->rows();
		cs.imgWidth = m_vmap_cano->cols();
		cs.intr = m_intr;
		cs.nmap_cano = *m_nmap_cano;
		cs.nmap_live = *m_nmap_live;
		cs.nmap_warp = *m_nmap_warp;
		cs.vmap_cano = *m_vmap_cano;
		cs.vmap_live = *m_vmap_live;
		cs.vmap_warp = *m_vmap_warp;
		cs.vmapKnn = m_vmapKnn;
		cs.nNodes = m_numNodes;
		cs.Tlw = m_pWarpField->get_rigidTransform();
		cs.psi_data = m_param->fusion_psi_data;

#ifdef ENABLE_GPU_DUMP_DEBUG
		// debugging
		DeviceArray<float> pixelSum2, pixelVal;
		pixelSum2.create(cs.imgHeight*cs.imgWidth);
		hipMemset(pixelSum2.ptr(), 0, pixelSum2.sizeBytes());
		pixelVal.create(cs.imgHeight*cs.imgWidth*VarPerNode);
		hipMemset(pixelVal.ptr(), 0, pixelVal.sizeBytes());
		cs.debug_buffer_pixel_sum2 = pixelSum2;
		cs.debug_buffer_pixel_val = pixelVal;
#endif

		//////////////////////////////
		dim3 block(CTA_SIZE_X, CTA_SIZE_Y);
		dim3 grid(1, 1, 1);
		grid.x = divUp(cs.imgWidth, block.x);
		grid.y = divUp(cs.imgHeight, block.y);
		dataTermCombinedKernel << <grid, block >> >(cs);
		cudaSafeCall(hipGetLastError(), "dataTermCombinedKernel");

		// debugging
#ifdef ENABLE_GPU_DUMP_DEBUG
		{
			std::vector<float> ps, pv;
			pixelSum2.download(ps);
			pixelVal.download(pv);

			FILE* pFile = fopen("D:/tmp/gpu_pixel.txt", "w");
			for (int i = 0; i < ps.size(); i++)
			{
				fprintf(pFile, "%ef %ef %ef %ef %ef %ef %ef\n",
					pv[i * 6 + 0], pv[i * 6 + 1], pv[i * 6 + 2],
					pv[i * 6 + 3], pv[i * 6 + 4], pv[i * 6 + 5],
					ps[i]);
			}
			fclose(pFile);
		}
#endif
	}
#pragma endregion

#pragma region --define sparse structure
	__global__ void count_Jr_rows_kernel(int* rctptr, int nMaxNodes)
	{
		int i = threadIdx.x + blockIdx.x*blockDim.x;
		if (i < nMaxNodes)
		{
			WarpField::KnnIdx knn = get_nodesKnn(i);
			int numK = -1;
			for (int k = 0; k < WarpField::KnnK; ++k)
			{
				if (knn_k(knn, k) < nMaxNodes)
					numK = k;
			}

			// each node generate 6*maxK rows
			rctptr[i] = (numK+1) * 6;
		}
		if (i == nMaxNodes)
			rctptr[i] = 0;
	}

	__global__ void compute_row_map_kernel(GpuGaussNewtonSolver::JrRow2NodeMapper* row2nodeId, 
		const int* rctptr, int nMaxNodes)
	{
		int iNode = threadIdx.x + blockIdx.x*blockDim.x;
		if (iNode < nMaxNodes)
		{
			int row_b = rctptr[iNode];
			int row_e = rctptr[iNode+1];
			for (int r = row_b; r < row_e; r++)
			{
				GpuGaussNewtonSolver::JrRow2NodeMapper mp;
				mp.nodeId = iNode;
				mp.k = (r - row_b) / 6;
				mp.ixyz = r - 6 * mp.k;
				row2nodeId[r] = mp;
			}
		}
	}

	__global__ void compute_Jr_rowPtr_colIdx_kernel(
		int* rptr, int* rptr_coo, int* colIdx,
		const GpuGaussNewtonSolver::JrRow2NodeMapper* row2nodeId, 
		int nMaxNodes, int nRows)
	{
		enum{
			VarPerNode = GpuGaussNewtonSolver::VarPerNode,
			ColPerRow = VarPerNode * 2
		};
		const int iRow = threadIdx.x + blockIdx.x*blockDim.x;
		if (iRow < nRows)
		{
			const int iNode = row2nodeId[iRow].nodeId;
			if (iNode < nMaxNodes)
			{
				WarpField::KnnIdx knn = get_nodesKnn(iNode);
				int knnNodeId = knn_k(knn, row2nodeId[iRow].k);
				if (knnNodeId < nMaxNodes)
				{
					int col_b = iRow*ColPerRow;
					rptr[iRow] = col_b;

					// each row 2*VerPerNode Cols
					// 1. self
					for (int j = 0; j < VarPerNode; j++, col_b++)
					{
						rptr_coo[col_b] = iRow;
						colIdx[col_b] = iNode*VarPerNode + j;
					}// j
					// 2. neighbor
					for (int j = 0; j < VarPerNode; j++, col_b++)
					{
						rptr_coo[col_b] = iRow;
						colIdx[col_b] = knnNodeId*VarPerNode + j;
					}// j
				}// end if knnNodeId
			}
		}// end if iRow < nRows
		if (iRow == nRows)
			rptr[nRows] = nRows * ColPerRow;
	}

	__global__ void calc_B_cidx_kernel(int* B_rptr_coo, int* B_cidx, 
		const int* B_rptr, int nRows, int nMaxNodes, int nLv0Nodes)
	{
		int iRow = threadIdx.x + blockIdx.x*blockDim.x;
		if (iRow < nRows)
		{
			int iNode = iRow / GpuGaussNewtonSolver::VarPerNode;

			WarpField::KnnIdx knn = get_nodesKnn(iNode);
			int col_b = B_rptr[iRow];
			for (int k = 0; k < WarpField::KnnK; ++k)
			{
				int knnNodeId = knn_k(knn, k);
				if (knnNodeId < nMaxNodes)
				{
					// 2. neighbor
					for (int j = 0; j < GpuGaussNewtonSolver::VarPerNode; j++, col_b++)
					{
						B_rptr_coo[col_b] = iRow;
						B_cidx[col_b] = (knnNodeId-nLv0Nodes)*GpuGaussNewtonSolver::VarPerNode + j;
					}// j
				}
			}
		}
	}

	void GpuGaussNewtonSolver::initSparseStructure()
	{
		// 1. compute Jr structure ==============================================
		// 1.0. decide the total rows we have for each nodes
		{
			dim3 block(CTA_SIZE);
			dim3 grid(divUp(m_numNodes, block.x));
			count_Jr_rows_kernel << <grid, block >> >(m_Jr_RowCounter.ptr(), m_numNodes);
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::initSparseStructure::count_Jr_rows_kernel");
			thrust_wrapper::exclusive_scan(m_Jr_RowCounter.ptr(), m_Jr_RowCounter.ptr(), m_numNodes + 1);
			cudaSafeCall(hipMemcpy(&m_Jrrows, m_Jr_RowCounter.ptr() + m_numNodes,
				sizeof(int), hipMemcpyDeviceToHost), "copy Jr rows to host");
		}

		// 1.1. collect nodes edges info:
		//	each low-level nodes are connected to k higher level nodes
		//	but the connections are not stored for the higher level nodes
		//  thus when processing each node, we add 2*k edges, w.r.t. 2*k*3 rows: each (x,y,z) a row
		//	for each row, there are exactly 2*VarPerNode values
		//	after this step, we can get the CSR/COO structure
		{
			dim3 block(CTA_SIZE);
			dim3 grid(divUp(m_numNodes, block.x));
			compute_row_map_kernel << <grid, block >> >(m_Jr_RowMap2NodeId.ptr(), m_Jr_RowCounter.ptr(), m_numNodes);
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::initSparseStructure::compute_row_map_kernel");
		}
		{
			dim3 block(CTA_SIZE);
			dim3 grid(divUp(m_Jrrows, block.x));
			compute_Jr_rowPtr_colIdx_kernel << <grid, block >> >(m_Jr_RowPtr.ptr(),
				m_Jr_RowPtr_coo.ptr(), m_Jr_ColIdx.ptr(), m_Jr_RowMap2NodeId.ptr(), m_numNodes, m_Jrrows);
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::initSparseStructure::compute_Jr_rowPtr_kernel");
			cudaSafeCall(hipMemcpy(&m_Jrnnzs, m_Jr_RowPtr.ptr() + m_Jrrows,
				sizeof(int), hipMemcpyDeviceToHost), "copy Jr nnz to host");
		}

		// 2. compute Jrt structure ==============================================
		// 2.1. fill (row, col) as (col, row) from Jr and sort.
		hipMemcpy(m_Jrt_RowPtr_coo.ptr(), m_Jr_ColIdx.ptr(), m_Jrnnzs*sizeof(int), hipMemcpyDeviceToDevice);
		hipMemcpy(m_Jrt_ColIdx.ptr(), m_Jr_RowPtr_coo.ptr(), m_Jrnnzs*sizeof(int), hipMemcpyDeviceToDevice);
		// !!!NOTE: we must use mergesort here, it can guarentees the order of values of the same key
		modergpu_wrapper::mergesort_by_key(m_Jrt_RowPtr_coo.ptr(), m_Jrt_ColIdx.ptr(), m_Jrnnzs);
		cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::initSparseStructure::mergesort_by_key");

		// 2.2. extract CSR rowptr info.
		if (HIPSPARSE_STATUS_SUCCESS != hipsparseXcoo2csr(m_cuSparseHandle,
			m_Jrt_RowPtr_coo.ptr(), m_Jrnnzs, m_Jrcols,
			m_Jrt_RowPtr.ptr(), HIPSPARSE_INDEX_BASE_ZERO))
			throw std::exception("GpuGaussNewtonSolver::initSparseStructure::hipsparseXcoo2csr failed");

		// 3. compute B structure ==============================================
		// 3.1 the row ptr of B is the same with the first L0 rows of Jrt.
		hipMemcpy(m_B_RowPtr.ptr(), m_Jrt_RowPtr.ptr(), (m_Brows + 1)*sizeof(int), hipMemcpyDeviceToDevice);
		cudaSafeCall(hipMemcpy(&m_Bnnzs, m_B_RowPtr.ptr() + m_Brows,
			sizeof(int), hipMemcpyDeviceToHost), "copy B nnz to host");
		
		// 3.2 the col-idx of B
		{
			dim3 block(CTA_SIZE);
			dim3 grid(divUp(m_Brows, block.x));
			calc_B_cidx_kernel << <grid, block >> >(m_B_RowPtr_coo.ptr(),
				m_B_ColIdx.ptr(), m_B_RowPtr.ptr(), m_Brows, m_numNodes, m_numLv0Nodes);
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::initSparseStructure::calc_B_cidx_kernel");
		}

		// 3.3 sort to compute Bt
		hipMemcpy(m_Bt_RowPtr_coo.ptr(), m_B_ColIdx.ptr(), m_Bnnzs*sizeof(int), hipMemcpyDeviceToDevice);
		hipMemcpy(m_Bt_ColIdx.ptr(), m_B_RowPtr_coo.ptr(), m_Bnnzs*sizeof(int), hipMemcpyDeviceToDevice);
		modergpu_wrapper::mergesort_by_key(m_Bt_RowPtr_coo.ptr(), m_Bt_ColIdx.ptr(), m_Bnnzs);
		cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::initSparseStructure::mergesort_by_key");
		if (HIPSPARSE_STATUS_SUCCESS != hipsparseXcoo2csr(m_cuSparseHandle,
			m_Bt_RowPtr_coo.ptr(), m_Bnnzs, m_Bcols,
			m_Bt_RowPtr.ptr(), HIPSPARSE_INDEX_BASE_ZERO))
			throw std::exception("GpuGaussNewtonSolver::initSparseStructure::hipsparseXcoo2csr failed");
	}

#pragma endregion

#pragma region --calc reg term
	struct RegTermJacobi
	{
		typedef WarpField::KnnIdx KnnIdx;
		typedef WarpField::IdxType IdxType;
		typedef GpuGaussNewtonSolver::JrRow2NodeMapper Mapper;
		enum
		{
			KnnK = WarpField::KnnK,
			VarPerNode = GpuGaussNewtonSolver::VarPerNode,
			VarPerNode2 = VarPerNode*VarPerNode,
			ColPerRow = VarPerNode * 2
		};

		int nNodes;
		int nRows;
		const Mapper* rows2nodeIds;
		const int* rptr;
		const int* cidx;
		mutable float* vptr;
		mutable float* fptr;

		float psi_reg;
		float lambda;

		__device__ __forceinline__  Tbx::Dual_quat_cu p_qk_p_alpha_func(Tbx::Dual_quat_cu dq, int i)const
		{
			Tbx::Vec3 t, r;
			float b, c, n;
			Tbx::Quat_cu q0(0, 0, 0, 0), q1 = dq.get_non_dual_part();
			switch (i)
			{
			case 0:
				dq.to_twist(r, t);
				n = r.norm();
				if (n > Tbx::Dual_quat_cu::epsilon())
				{
					b = sin(n) / n;
					c = (cos(n) - b) / (n*n);
					q0.coeff0 = -r.x * b;
					q0.coeff1 = b + r.x*r.x*c;
					q0.coeff2 = r.x*r.y*c;
					q0.coeff3 = r.x*r.z*c;
				}
				else
				{
					q0.coeff0 = 0;
					q0.coeff1 = 1;
					q0.coeff2 = 0;
					q0.coeff3 = 0;
				}

				q1.coeff0 = (t.x * q0.coeff1 + t.y * q0.coeff2 + t.z * q0.coeff3) * (-0.5);
				q1.coeff1 = (t.x * q0.coeff0 + t.y * q0.coeff3 - t.z * q0.coeff2) * 0.5;
				q1.coeff2 = (-t.x * q0.coeff3 + t.y * q0.coeff0 + t.z * q0.coeff1) * 0.5;
				q1.coeff3 = (t.x * q0.coeff2 - t.y * q0.coeff1 + t.z * q0.coeff0) * 0.5;
				return Tbx::Dual_quat_cu(q0, q1);
			case 1:
				dq.to_twist(r, t);
				n = r.norm();
				if (n > Tbx::Dual_quat_cu::epsilon())
				{
					b = sin(n) / n;
					c = (cos(n) - b) / (n*n);
					q0.coeff0 = -r.y * b;
					q0.coeff1 = r.y*r.x*c;
					q0.coeff2 = b + r.y*r.y*c;
					q0.coeff3 = r.y*r.z*c;
				}
				else
				{
					q0.coeff0 = 0;
					q0.coeff1 = 0;
					q0.coeff2 = 1;
					q0.coeff3 = 0;
				}

				q1.coeff0 = (t.x * q0.coeff1 + t.y * q0.coeff2 + t.z * q0.coeff3) * (-0.5);
				q1.coeff1 = (t.x * q0.coeff0 + t.y * q0.coeff3 - t.z * q0.coeff2) * 0.5;
				q1.coeff2 = (-t.x * q0.coeff3 + t.y * q0.coeff0 + t.z * q0.coeff1) * 0.5;
				q1.coeff3 = (t.x * q0.coeff2 - t.y * q0.coeff1 + t.z * q0.coeff0) * 0.5;
				return Tbx::Dual_quat_cu(q0, q1);
			case 2:
				dq.to_twist(r, t);
				n = r.norm();
				if (n > Tbx::Dual_quat_cu::epsilon())
				{
					b = sin(n) / n;
					c = (cos(n) - b) / (n*n);

					q0.coeff0 = -r.z * b;
					q0.coeff1 = r.z*r.x*c;
					q0.coeff2 = r.z*r.y*c;
					q0.coeff3 = b + r.z*r.z*c;
				}
				else
				{
					q0.coeff0 = 0;
					q0.coeff1 = 0;
					q0.coeff2 = 0;
					q0.coeff3 = 1;
				}

				q1.coeff0 = (t.x * q0.coeff1 + t.y * q0.coeff2 + t.z * q0.coeff3) * (-0.5);
				q1.coeff1 = (t.x * q0.coeff0 + t.y * q0.coeff3 - t.z * q0.coeff2) * 0.5;
				q1.coeff2 = (-t.x * q0.coeff3 + t.y * q0.coeff0 + t.z * q0.coeff1) * 0.5;
				q1.coeff3 = (t.x * q0.coeff2 - t.y * q0.coeff1 + t.z * q0.coeff0) * 0.5;
				return Tbx::Dual_quat_cu(q0, q1);
			case 3:
				return Tbx::Dual_quat_cu(q0, Tbx::Quat_cu(-q1.coeff1, q1.coeff0, -q1.coeff3, q1.coeff2))*0.5;
			case 4:
				return Tbx::Dual_quat_cu(q0, Tbx::Quat_cu(-q1.coeff2, q1.coeff3, q1.coeff0, -q1.coeff1))*0.5;
			case 5:
				return Tbx::Dual_quat_cu(q0, Tbx::Quat_cu(-q1.coeff3, -q1.coeff2, q1.coeff1, q1.coeff0))*0.5;
			default:
				return Tbx::Dual_quat_cu();
			}
		}

		__device__ __forceinline__  Tbx::Vec3 reg_term_penalty(Tbx::Vec3 f)const
		{
			// the robust Huber penelty gradient
			Tbx::Vec3 df;
			float norm = f.norm();
			if (norm < psi_reg)
				df = f;
			else
			for (int k = 0; k < 3; k++)
				df[k] = sign(f[k])*psi_reg;
			return df;
		}

		__device__ __forceinline__  Tbx::Transfo p_SE3_p_alpha_func(Tbx::Dual_quat_cu dq, int i)const
		{
			Tbx::Transfo T = Tbx::Transfo::empty();
			Tbx::Dual_quat_cu p_dq_p_alphai = p_qk_p_alpha_func(dq, i) * 2.f;

			//// evaluate p_dqi_p_alphak, heavily hard code here
			//// this hard code is crucial to the performance 
			// 0:
			// (0, -z0, y0, x1,
			// z0, 0, -x0, y1,
			//-y0, x0, 0, z1,
			// 0, 0, 0, 0) * 2;
			float p_dqi_p_alphak = p_dq_p_alphai[0];
			T[1] += -dq[3] * p_dqi_p_alphak;
			T[2] += dq[2] * p_dqi_p_alphak;
			T[3] += dq[5] * p_dqi_p_alphak;
			T[4] += dq[3] * p_dqi_p_alphak;
			T[6] += -dq[1] * p_dqi_p_alphak;
			T[7] += dq[6] * p_dqi_p_alphak;
			T[8] += -dq[2] * p_dqi_p_alphak;
			T[9] += dq[1] * p_dqi_p_alphak;
			T[11] += dq[7] * p_dqi_p_alphak;

			// 1
			//( 0, y0, z0, -w1,
			//	y0, -2 * x0, -w0, -z1,
			//	z0, w0, -2 * x0, y1,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[1];
			T[1] += dq[2] * p_dqi_p_alphak;
			T[2] += dq[3] * p_dqi_p_alphak;
			T[3] += -dq[4] * p_dqi_p_alphak;
			T[4] += dq[2] * p_dqi_p_alphak;
			T[5] += -dq[1] * p_dqi_p_alphak * 2;
			T[6] += -dq[0] * p_dqi_p_alphak;
			T[7] += -dq[7] * p_dqi_p_alphak;
			T[8] += dq[3] * p_dqi_p_alphak;
			T[9] += dq[0] * p_dqi_p_alphak;
			T[10] += -dq[1] * p_dqi_p_alphak * 2;
			T[11] += dq[6] * p_dqi_p_alphak;

			// 2.
			// (-2 * y0, x0, w0, z1,
			//	x0, 0, z0, -w1,
			//	-w0, z0, -2 * y0, -x1,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[2];
			T[0] += -dq[2] * p_dqi_p_alphak * 2;
			T[1] += dq[1] * p_dqi_p_alphak;
			T[2] += dq[0] * p_dqi_p_alphak;
			T[3] += dq[7] * p_dqi_p_alphak;
			T[4] += dq[1] * p_dqi_p_alphak;
			T[6] += dq[3] * p_dqi_p_alphak;
			T[7] += -dq[4] * p_dqi_p_alphak;
			T[8] += -dq[0] * p_dqi_p_alphak;
			T[9] += dq[3] * p_dqi_p_alphak;
			T[10] += -dq[2] * p_dqi_p_alphak * 2;
			T[11] += -dq[5] * p_dqi_p_alphak;

			// 3.
			// (-2 * z0, -w0, x0, -y1,
			//	w0, -2 * z0, y0, x1,
			//	x0, y0, 0, -w1,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[3];
			T[0] += -dq[3] * p_dqi_p_alphak * 2;
			T[1] += -dq[0] * p_dqi_p_alphak;
			T[2] += dq[1] * p_dqi_p_alphak;
			T[3] += -dq[6] * p_dqi_p_alphak;
			T[4] += dq[0] * p_dqi_p_alphak;
			T[5] += -dq[3] * p_dqi_p_alphak * 2;
			T[6] += dq[2] * p_dqi_p_alphak;
			T[7] += dq[5] * p_dqi_p_alphak;
			T[8] += dq[1] * p_dqi_p_alphak;
			T[9] += dq[2] * p_dqi_p_alphak;
			T[11] += -dq[4] * p_dqi_p_alphak;

			// 4.
			//( 0, 0, 0, -x0,
			//	0, 0, 0, -y0,
			//	0, 0, 0, -z0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[4];
			T[3] += -dq[1] * p_dqi_p_alphak;
			T[7] += -dq[2] * p_dqi_p_alphak;
			T[11] += -dq[3] * p_dqi_p_alphak;

			// 5. 
			// (0, 0, 0, w0,
			//	0, 0, 0, z0,
			//	0, 0, 0, -y0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[5];
			T[3] += dq[0] * p_dqi_p_alphak;
			T[7] += dq[3] * p_dqi_p_alphak;
			T[11] += -dq[2] * p_dqi_p_alphak;

			// 6. 
			// (0, 0, 0, -z0,
			//	0, 0, 0, w0,
			//	0, 0, 0, x0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[6];
			T[3] += -dq[3] * p_dqi_p_alphak;
			T[7] += dq[0] * p_dqi_p_alphak;
			T[11] += dq[1] * p_dqi_p_alphak;

			// 7.
			// (0, 0, 0, y0,
			//	0, 0, 0, -x0,
			//	0, 0, 0, w0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[7];
			T[3] += dq[2] * p_dqi_p_alphak;
			T[7] += -dq[1] * p_dqi_p_alphak;
			T[11] += dq[0] * p_dqi_p_alphak;

			return T;
		}

		__device__ __forceinline__ void operator () () const
		{
			const int iRow = (threadIdx.x + blockIdx.x * blockDim.x)*6;
		
			if (iRow >= nRows)
				return;

			Mapper mapper = rows2nodeIds[iRow];
			int knnNodeId = knn_k(get_nodesKnn(mapper.nodeId), mapper.k);

			if (knnNodeId >= nNodes)
				return;

			int cooPos = rptr[iRow];

			Tbx::Dual_quat_cu dqi, dqj;
			Tbx::Vec3 ri, ti, rj, tj;
			get_twist(mapper.nodeId, ri, ti);
			get_twist(knnNodeId, rj, tj);
			dqi.from_twist(ri, ti);
			dqj.from_twist(rj, tj);

			float4 nodeVwi = get_nodesVw(mapper.nodeId);
			float4 nodeVwj = get_nodesVw(knnNodeId);
			Tbx::Point3 vi(convert(read_float3_4(nodeVwi)));
			Tbx::Point3 vj(convert(read_float3_4(nodeVwj)));
			float alpha_ij = max(1.f / nodeVwi.w, 1.f / nodeVwj.w);
			float ww = sqrt(lambda * alpha_ij);

			// energy=============================================
			Tbx::Vec3 val = dqi.transform(Tbx::Point3(vj)) - dqj.transform(Tbx::Point3(vj));
			val = reg_term_penalty(val);

			fptr[iRow + 0] = val.x * ww;
			fptr[iRow + 1] = val.y * ww;
			fptr[iRow + 2] = val.z * ww;

			Tbx::Vec3 val1 = dqi.transform(Tbx::Point3(vi)) - dqj.transform(Tbx::Point3(vi));
			val1 = reg_term_penalty(val1);
			fptr[iRow + 3] = val1.x * ww;
			fptr[iRow + 4] = val1.y * ww;
			fptr[iRow + 5] = val1.z * ww;

			// jacobi=============================================
			for (int ialpha = 0; ialpha < VarPerNode; ialpha++)
			{
				Tbx::Transfo p_Ti_p_alpha = p_SE3_p_alpha_func(dqi, ialpha);
				Tbx::Transfo p_Tj_p_alpha = p_SE3_p_alpha_func(dqj, ialpha);

				// partial_psi_partial_alpha
				Tbx::Vec3 p_psi_p_alphai_j = (p_Ti_p_alpha * vj) * ww;
				Tbx::Vec3 p_psi_p_alphaj_j = (p_Tj_p_alpha * vj) * (-ww);
				Tbx::Vec3 p_psi_p_alphai_i = (p_Ti_p_alpha * vi) * ww;
				Tbx::Vec3 p_psi_p_alphaj_i = (p_Tj_p_alpha * vi) * (-ww);

				for (int ixyz = 0; ixyz < 3; ixyz++)
				{
					int pos = cooPos + ixyz*ColPerRow + ialpha;
					vptr[pos] = p_psi_p_alphai_j[ixyz];
					vptr[pos + VarPerNode] = p_psi_p_alphaj_j[ixyz];
					pos += 3 * ColPerRow;
					vptr[pos] = p_psi_p_alphai_i[ixyz];
					vptr[pos + VarPerNode] = p_psi_p_alphaj_i[ixyz];
				}
			}// end for ialpha
		}// end function ()
	};

	__global__ void calcRegTerm_kernel(RegTermJacobi rj)
	{
		rj();
	}

	void GpuGaussNewtonSolver::calcRegTerm()
	{
		RegTermJacobi rj;
		rj.cidx = m_Jr_ColIdx.ptr();
		rj.lambda = m_param->fusion_lambda;
		rj.nNodes = m_numNodes;
		rj.nRows = m_Jrrows;
		rj.psi_reg = m_param->fusion_psi_reg;
		rj.rows2nodeIds = m_Jr_RowMap2NodeId;
		rj.rptr = m_Jr_RowPtr.ptr();
		rj.vptr = m_Jr_val.ptr();
		rj.fptr = m_f_r.ptr();

		dim3 block(CTA_SIZE);
		dim3 grid(divUp(m_Jrrows / 6, block.x));

		calcRegTerm_kernel << <grid, block >> >(rj);
		cudaSafeCall(hipGetLastError(), "calcRegTerm_kernel");

		// 2. compute Jrt ==============================================
		// 2.1. fill (row, col) as (col, row) from Jr and sort.
		hipMemcpy(m_Jrt_RowPtr_coo.ptr(), m_Jr_ColIdx.ptr(), m_Jrnnzs*sizeof(int), hipMemcpyDeviceToDevice);
		hipMemcpy(m_Jrt_val.ptr(), m_Jr_val.ptr(), m_Jrnnzs*sizeof(float), hipMemcpyDeviceToDevice);
		modergpu_wrapper::mergesort_by_key(m_Jrt_RowPtr_coo.ptr(), m_Jrt_val.ptr(), m_Jrnnzs);
		cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::calcRegTerm::mergesort_by_key");
	}
#pragma endregion

#pragma region --calc Hessian
#define ENABLE_GPU_DUMP_DEBUG_H
	__global__ void calcJr0tJr0_add_to_Hd_kernel(float* Hd, int nLv0Nodes, const int* Jrt_rptr)
	{
		enum
		{
			VarPerNode = GpuGaussNewtonSolver::VarPerNode,
			VarPerNode2 = VarPerNode*VarPerNode,
			LowerPartNum = GpuGaussNewtonSolver::LowerPartNum
		};

		int tid = threadIdx.x + blockIdx.x * blockDim.x;
		int iNode = tid / LowerPartNum;
		if (iNode >= nLv0Nodes)
			return;
		int eleLowerShift = tid - iNode*LowerPartNum;
		int rowShift = g_lower_2_rowShift_6x6[eleLowerShift];
		int colShift = g_lower_2_colShift_6x6[eleLowerShift];
		int row0 = iNode*VarPerNode;

		const int row0_begin = Jrt_rptr[row0 + rowShift];
		const int row_len = Jrt_rptr[row0 + rowShift + 1] - row0_begin;
		const int row1_begin = Jrt_rptr[row0 + colShift];

		float sum = 0.f;
		for (int i = 0; i < row_len; i++)
			sum += get_JrtVal(row1_begin + i) * get_JrtVal(row0_begin + i);

		Hd[iNode * VarPerNode2 + g_lower_2_full_6x6[eleLowerShift]] += sum;
	}
	
	__global__ void calcB_kernel(
		float* B_val, const int* B_rptr_coo, const int* B_cidx, 
		int nBrows, int Bnnz, const int* Jrt_rptr)
	{
		enum{VarPerNode = GpuGaussNewtonSolver::VarPerNode};

		int tid = threadIdx.x + blockIdx.x * blockDim.x;
		if (tid >= Bnnz)
			return;

		int iBrow = B_rptr_coo[tid];
		int iBcol = B_cidx[tid];

		int Jr0t_cb = Jrt_rptr[iBrow];
		int Jr0t_ce = Jrt_rptr[iBrow + 1];

		int Jr1_rb = Jrt_rptr[iBcol + nBrows];
		int Jr1_re = Jrt_rptr[iBcol + nBrows + 1];

		float sum = 0.f;
		for (int i0 = Jr0t_cb, i1 = Jr1_rb; i0 < Jr0t_ce && i1 < Jr1_re; )
		{
			int Jr0t_c = get_JrtCidx(i0);
			int Jr1_r = get_JrtCidx(i1);
			if (Jr0t_c == Jr1_r)
			{
				for (int k = 0; k < VarPerNode; k++)
					sum += get_JrtVal(i0 + k) * get_JrtVal(i1 + k);
				i0 += VarPerNode;
				i1 += VarPerNode;
			}

			i0 += (Jr0t_c < Jr1_r) * VarPerNode;
			i1 += (Jr0t_c > Jr1_r) * VarPerNode;
		}// i

		B_val[tid] = sum;
	}

	void GpuGaussNewtonSolver::calcHessian()
	{
		// 1. compute Jr0'Jr0 and accumulate into Hd
		{
			dim3 block(CTA_SIZE);
			dim3 grid(divUp(m_numLv0Nodes*LowerPartNum, block.x));
			calcJr0tJr0_add_to_Hd_kernel << <grid, block >> >(m_Hd, m_numLv0Nodes, 
				m_Jrt_RowPtr.ptr());
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::calcHessian::calcJr0tJr0_add_to_Hd_kernel");
		}

		// 2. compute B = Jr0'Jr1
		{
			dim3 block(CTA_SIZE);
			dim3 grid(divUp(m_Bnnzs, block.x));
			calcB_kernel << <grid, block >> >(m_B_val.ptr(), m_B_RowPtr_coo.ptr(), 
				m_B_ColIdx.ptr(), m_Brows, m_Bnnzs, m_Jrt_RowPtr.ptr());
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::calcHessian::calcB_kernel");
		}

		// 3. compute Bt
		hipMemcpy(m_Bt_RowPtr_coo.ptr(), m_B_ColIdx.ptr(), m_Bnnzs*sizeof(int), hipMemcpyDeviceToDevice);
		hipMemcpy(m_Bt_val.ptr(), m_B_val.ptr(), m_Bnnzs*sizeof(float), hipMemcpyDeviceToDevice);
		modergpu_wrapper::mergesort_by_key(m_Bt_RowPtr_coo.ptr(), m_Bt_val.ptr(), m_Bnnzs);
		cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::calcHessian::mergesort_by_key");
	}
#pragma endregion
}