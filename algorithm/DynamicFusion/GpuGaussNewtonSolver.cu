#include "hip/hip_runtime.h"
#include "GpuGaussNewtonSolver.h"
#include "device_utils.h"
#include "cudpp\cudpp_wrapper.h"
#include "cudpp\thrust_wrapper.h"
#include "cudpp\ModerGpuWrapper.h"
namespace dfusion
{
	texture<WarpField::KnnIdx, hipTextureType1D, hipReadModeElementType> g_nodesKnnTex;
	texture<float4, hipTextureType1D, hipReadModeElementType> g_nodesVwTex;
	texture<float, hipTextureType1D, hipReadModeElementType> g_twistTex;

	__device__ __forceinline__ float4 get_nodesVw(int i)
	{
		return tex1Dfetch(g_nodesVwTex, i);
	}

	__device__ __forceinline__ WarpField::KnnIdx get_nodesKnn(int i)
	{
		return tex1Dfetch(g_nodesKnnTex, i);
	}

	__device__ __forceinline__ void get_twist(int i, Tbx::Vec3& r, Tbx::Vec3& t)
	{
		int i6 = i * 6;
		r.x = tex1Dfetch(g_twistTex, i6++);
		r.y = tex1Dfetch(g_twistTex, i6++);
		r.z = tex1Dfetch(g_twistTex, i6++);
		t.x = tex1Dfetch(g_twistTex, i6++);
		t.y = tex1Dfetch(g_twistTex, i6++);
		t.z = tex1Dfetch(g_twistTex, i6++);
	}

	// map the lower part to full 6x6 matrix
	__constant__ int g_lower_2_full_6x6[21] = {
		0,
		6, 7,
		12, 13, 14,
		18, 19, 20, 21,
		24, 25, 26, 27, 28,
		30, 31, 32, 33, 34, 35
	};
	__constant__ int g_lfull_2_lower_6x6[6][6] = {
		{ 0, -1, -1, -1, -1, -1 },
		{ 1, 2, -1, -1, -1, -1 },
		{ 3, 4, 5, -1, -1, -1 },
		{ 6, 7, 8, 9, -1, -1 },
		{ 10, 11, 12, 13, 14, -1 },
		{ 15, 16, 17, 18, 19, 20 },
	};

#define D_1_DIV_6 0.166666667

	__device__ __forceinline__ float3 read_float3_4(float4 a)
	{
		return make_float3(a.x, a.y, a.z);
	}

	__device__ __forceinline__ float sqr(float a)
	{
		return a*a;
	}

	__device__ __forceinline__ float pow3(float a)
	{
		return a*a*a;
	}

	__device__ __forceinline__ float sign(float a)
	{
		return (a>0.f) - (a<0.f);
	}

	__device__ __forceinline__ WarpField::IdxType& knn_k(WarpField::KnnIdx& knn, int k)
	{
		return ((WarpField::IdxType*)(&knn))[k];
	}
	__device__ __forceinline__ const WarpField::IdxType& knn_k(const WarpField::KnnIdx& knn, int k)
	{
		return ((WarpField::IdxType*)(&knn))[k];
	}

	__device__ __forceinline__ void sort_knn(WarpField::KnnIdx& knn)
	{
		for (int i = 1; i < WarpField::KnnK; i++)
		{
			WarpField::IdxType x = knn_k(knn,i);
			int	j = i;
			while (j > 0 && knn_k(knn, j - 1) > x)
			{
				knn_k(knn, j) = knn_k(knn, j - 1);
				j = j - 1;
			}
			knn_k(knn, j) = x;
		}
	}

#pragma region --bind textures
	void GpuGaussNewtonSolver::bindTextures()
	{
		if (1)
		{
			size_t offset;
			hipChannelFormatDesc desc = hipCreateChannelDesc<WarpField::KnnIdx>();
			hipBindTexture(&offset, &g_nodesKnnTex, m_nodesKnn.ptr(), &desc,
				m_nodesKnn.size() * sizeof(WarpField::KnnIdx));
			if (offset != 0)
				throw std::exception("GpuGaussNewtonSolver::bindTextures(): non-zero-offset error!");
		}
		if (1)
		{
			size_t offset;
			hipChannelFormatDesc desc = hipCreateChannelDesc<float4>();
			hipBindTexture(&offset, &g_nodesVwTex, m_nodesVw.ptr(), &desc,
				m_nodesVw.size() * sizeof(float4));
			if (offset != 0)
				throw std::exception("GpuGaussNewtonSolver::bindTextures(): non-zero-offset error!");
		}
		if (1)
		{
			size_t offset;
			hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
			hipBindTexture(&offset, &g_twistTex, m_twist.ptr(), &desc,
				m_twist.size() * sizeof(float));
			if (offset != 0)
				throw std::exception("GpuGaussNewtonSolver::bindTextures(): non-zero-offset error!");
		}
	}

	void GpuGaussNewtonSolver::unBindTextures()
	{
		hipUnbindTexture(g_twistTex);
		hipUnbindTexture(g_nodesVwTex);
		hipUnbindTexture(g_nodesKnnTex);
	}
#pragma endregion

#pragma region --calc data term

//#define ENABLE_GPU_DUMP_DEBUG

	struct DataTermCombined
	{
		typedef WarpField::KnnIdx KnnIdx;
		typedef WarpField::IdxType IdxType;
		enum
		{
			CTA_SIZE_X = GpuGaussNewtonSolver::CTA_SIZE_X,
			CTA_SIZE_Y = GpuGaussNewtonSolver::CTA_SIZE_Y,
			CTA_SIZE = CTA_SIZE_X * CTA_SIZE_Y,
			KnnK = WarpField::KnnK,
			VarPerNode = GpuGaussNewtonSolver::VarPerNode,
			VarPerNode2 = VarPerNode*VarPerNode,
			LowerPartNum = GpuGaussNewtonSolver::LowerPartNum,
		};

		PtrStep<float4> vmap_live;
		PtrStep<float4> nmap_live;
		PtrStep<float4> vmap_warp;
		PtrStep<float4> nmap_warp;
		PtrStep<float4> vmap_cano;
		PtrStep<float4> nmap_cano;
		PtrStep<KnnIdx> vmapKnn;
		float* Hd_;
		float* g_;

		Intr intr;
		Tbx::Transfo Tlw;

		int imgWidth;
		int imgHeight;
		int nNodes;

		float distThres;
		float angleThres;
		float psi_data;

#ifdef ENABLE_GPU_DUMP_DEBUG
		// for debug
		float* debug_buffer_pixel_sum2;
		float* debug_buffer_pixel_val;
#endif

		__device__ __forceinline__ float data_term_penalty(float f)const
		{
			return f * sqr(max(0.f, 1.f - sqr(f / psi_data)));
			//// the robust Tukey penelty gradient
			//if (abs(f) <= psi_data)
			//	return f * sqr(1 - sqr(f / psi_data));
			//else
			//	return 0;
		}

		__device__ __forceinline__ float trace_AtB(Tbx::Transfo A, Tbx::Transfo B)const
		{
			float sum = 0;
			for (int i = 0; i < 16; i++)
				sum += A[i] * B[i];
			return sum;
		}

		__device__ __forceinline__ Tbx::Transfo compute_p_f_p_T(const Tbx::Vec3& n,
			const Tbx::Point3& v, const Tbx::Point3& vl, const Tbx::Dual_quat_cu& dq)const
		{
			//Tbx::Transfo T = Tlw*dq.to_transformation_after_normalize();
			//Tbx::Transfo nvt = outer_product(n, v);
			//Tbx::Transfo vlnt = outer_product(n, vl).transpose();
			//Tbx::Transfo p_f_p_T = T*(nvt + nvt.transpose()) - vlnt;
			Tbx::Vec3 Tn = Tlw*dq.rotate(n);
			Tbx::Point3 Tv(Tlw*dq.transform(v) - vl);
			return Tbx::Transfo(
				Tn.x*v.x + n.x*Tv.x, Tn.x*v.y + n.y*Tv.x, Tn.x*v.z + n.z*Tv.x, Tn.x,
				Tn.y*v.x + n.x*Tv.y, Tn.y*v.y + n.y*Tv.y, Tn.y*v.z + n.z*Tv.y, Tn.y,
				Tn.z*v.x + n.x*Tv.z, Tn.z*v.y + n.y*Tv.z, Tn.z*v.z + n.z*Tv.z, Tn.z,
				n.x, n.y, n.z, 0
				);
		}

		__device__ __forceinline__ Tbx::Transfo p_T_p_alphak_func(const Tbx::Dual_quat_cu& p_qk_p_alpha,
			const Tbx::Dual_quat_cu& dq_bar, const Tbx::Dual_quat_cu& dq, float inv_norm_dq_bar, float wk_k)const
		{
			Tbx::Transfo p_T_p_alphak = Tbx::Transfo::empty();

			float pdot = dq_bar.get_non_dual_part().dot(p_qk_p_alpha.get_non_dual_part())
				* sqr(inv_norm_dq_bar);

			//// evaluate p_dqi_p_alphak, heavily hard code here
			//// this hard code is crucial to the performance 
			// 0:
			// (0, -z0, y0, x1,
			// z0, 0, -x0, y1,
			//-y0, x0, 0, z1,
			// 0, 0, 0, 0) * 2;
			float p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[0] - dq_bar[0] * pdot
				);
			p_T_p_alphak[1] += -dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[2] += dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[3] += dq[5] * p_dqi_p_alphak;
			p_T_p_alphak[4] += dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[6] += -dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[7] += dq[6] * p_dqi_p_alphak;
			p_T_p_alphak[8] += -dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[9] += dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[11] += dq[7] * p_dqi_p_alphak;

			// 1
			//( 0, y0, z0, -w1,
			//	y0, -2 * x0, -w0, -z1,
			//	z0, w0, -2 * x0, y1,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[1] - dq_bar[1] * pdot
				);
			p_T_p_alphak[1] += dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[2] += dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[3] += -dq[4] * p_dqi_p_alphak;
			p_T_p_alphak[4] += dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[5] += -dq[1] * p_dqi_p_alphak * 2;
			p_T_p_alphak[6] += -dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[7] += -dq[7] * p_dqi_p_alphak;
			p_T_p_alphak[8] += dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[9] += dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[10] += -dq[1] * p_dqi_p_alphak * 2;
			p_T_p_alphak[11] += dq[6] * p_dqi_p_alphak;

			// 2.
			// (-2 * y0, x0, w0, z1,
			//	x0, 0, z0, -w1,
			//	-w0, z0, -2 * y0, -x1,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[2] - dq_bar[2] * pdot
				);
			p_T_p_alphak[0] += -dq[2] * p_dqi_p_alphak * 2;
			p_T_p_alphak[1] += dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[2] += dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[3] += dq[7] * p_dqi_p_alphak;
			p_T_p_alphak[4] += dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[6] += dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[7] += -dq[4] * p_dqi_p_alphak;
			p_T_p_alphak[8] += -dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[9] += dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[10] += -dq[2] * p_dqi_p_alphak * 2;
			p_T_p_alphak[11] += -dq[5] * p_dqi_p_alphak;

			// 3.
			// (-2 * z0, -w0, x0, -y1,
			//	w0, -2 * z0, y0, x1,
			//	x0, y0, 0, -w1,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[3] - dq_bar[3] * pdot
				);
			p_T_p_alphak[0] += -dq[3] * p_dqi_p_alphak * 2;
			p_T_p_alphak[1] += -dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[2] += dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[3] += -dq[6] * p_dqi_p_alphak;
			p_T_p_alphak[4] += dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[5] += -dq[3] * p_dqi_p_alphak * 2;
			p_T_p_alphak[6] += dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[7] += dq[5] * p_dqi_p_alphak;
			p_T_p_alphak[8] += dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[9] += dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[11] += -dq[4] * p_dqi_p_alphak;

			// 4.
			//( 0, 0, 0, -x0,
			//	0, 0, 0, -y0,
			//	0, 0, 0, -z0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[4] - dq_bar[4] * pdot
				);
			p_T_p_alphak[3] += -dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[7] += -dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[11] += -dq[3] * p_dqi_p_alphak;

			// 5. 
			// (0, 0, 0, w0,
			//	0, 0, 0, z0,
			//	0, 0, 0, -y0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[5] - dq_bar[5] * pdot
				);
			p_T_p_alphak[3] += dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[7] += dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[11] += -dq[2] * p_dqi_p_alphak;

			// 6. 
			// (0, 0, 0, -z0,
			//	0, 0, 0, w0,
			//	0, 0, 0, x0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[6] - dq_bar[6] * pdot
				);
			p_T_p_alphak[3] += -dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[7] += dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[11] += dq[1] * p_dqi_p_alphak;

			// 7.
			// (0, 0, 0, y0,
			//	0, 0, 0, -x0,
			//	0, 0, 0, w0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[7] - dq_bar[7] * pdot
				);
			p_T_p_alphak[3] += dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[7] += -dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[11] += dq[0] * p_dqi_p_alphak;

			p_T_p_alphak = Tlw * p_T_p_alphak;
			return p_T_p_alphak;
		}

		__device__ __forceinline__ bool search(int x, int y, Tbx::Point3& vl) const
		{
			float3 vwarp = read_float3_4(vmap_warp(y, x));
			float3 nwarp = read_float3_4(nmap_warp(y, x));

			if (isnan(nwarp.x))
				return false;

			float3 uvd = intr.xyz2uvd(vwarp);
			int2 ukr = make_int2(uvd.x + 0.5, uvd.y + 0.5);

			// we use opengl coordinate, thus world.z should < 0
			if (ukr.x < 0 || ukr.y < 0 || ukr.x >= imgWidth || ukr.y >= imgHeight || vwarp.z >= 0)
				return false;

			float3 vlive = read_float3_4(vmap_live[ukr.y*imgWidth + ukr.x]);
			float3 nlive = read_float3_4(nmap_live[ukr.y*imgWidth + ukr.x]);
			if (isnan(nlive.x))
				return false;

			float dist = norm(vwarp - vlive);
			if (!(dist <= distThres))
				return false;

			float sine = norm(cross(nwarp, nlive));
			if (!(sine < angleThres))
				return false;

			vl = Tbx::Point3(vlive.x, vlive.y, vlive.z);

			return true;
		}

		__device__ __forceinline__ void operator () () const
		{
			const int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
			const int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

			Tbx::Point3 vl;
			bool found_coresp = false;
			if (x < imgWidth && y < imgHeight)
				found_coresp = search(x, y, vl);

			if (found_coresp)
			{
				Tbx::Point3 v(convert(read_float3_4(vmap_cano(y, x))));
				Tbx::Vec3 n(convert(read_float3_4(nmap_cano(y, x))));

				const KnnIdx knn = vmapKnn(y, x);
				Tbx::Dual_quat_cu dq(Tbx::Quat_cu(0, 0, 0, 0), Tbx::Quat_cu(0, 0, 0, 0));
				Tbx::Dual_quat_cu dqk_0;
				float wk[KnnK];
				for (int k = 0; k < KnnK; k++)
				{
					int knnNodeId = knn_k(knn, k);
					if (knnNodeId < nNodes)
					{
						Tbx::Vec3 r, t;
						get_twist(knnNodeId, r, t);
						float4 nodeVw = get_nodesVw(knnNodeId);
						Tbx::Point3 nodesV(convert(read_float3_4(nodeVw)));
						float invNodesW = nodeVw.w;
						Tbx::Dual_quat_cu dqk_k;
						dqk_k.from_twist(r, t);
						// note: we store inv radius as vw.w, thus using * instead of / here
						wk[k] = __expf(-(v - nodesV).dot(v - nodesV)*(2 * invNodesW * invNodesW));
						if (k == 0)
							dqk_0 = dqk_k;
						if (dqk_0.get_non_dual_part().dot(dqk_k.get_non_dual_part()) < 0)
							wk[k] = -wk[k];
						dq = dq + dqk_k * wk[k];
					}
				}

				Tbx::Dual_quat_cu dq_bar = dq;
				float inv_norm_dq_bar = 1.f / dq_bar.get_non_dual_part().norm();
				dq = dq * inv_norm_dq_bar; // normalize

				// the grad energy f
				const float f = data_term_penalty((Tlw*dq.rotate(n)).dot(Tlw*dq.transform(v) - vl));

				// paitial_f_partial_T
				const Tbx::Transfo p_f_p_T = compute_p_f_p_T(n, v, vl, dq);

				for (int knnK = 0; knnK < KnnK; knnK++)
				{
					float p_f_p_alpha[VarPerNode];
					int knnNodeId = knn_k(knn, knnK);
					float wk_k = wk[knnK] * inv_norm_dq_bar * 2;
					if (knnNodeId < nNodes)
					{
						//// comput partial_T_partial_alphak, hard code here.
						Tbx::Dual_quat_cu p_qk_p_alpha;
						Tbx::Transfo p_T_p_alphak;
						Tbx::Vec3 t, r;
						float b, c;
						Tbx::Quat_cu q1;
						get_twist(knnNodeId, r, t);
						{
							float n = r.norm();
							float sin_n, cos_n;
							sincos(n, &sin_n, &cos_n);
							b = n > Tbx::Dual_quat_cu::epsilon() ? sin_n / n : 1;
							c = n > Tbx::Dual_quat_cu::epsilon() ? (cos_n - b) / (n*n) : 0;
							q1 = Tbx::Quat_cu(cos_n*0.5f, r.x*b*0.5f, r.y*b*0.5f, r.z*b*0.5f);
						}

						// alpha0
						p_qk_p_alpha[0] = -r[0] * b;
						p_qk_p_alpha[1] = b + r[0] * r[0] * c;
						p_qk_p_alpha[2] = r[0] * r[1] * c;
						p_qk_p_alpha[3] = r[0] * r[2] * c;
						p_qk_p_alpha = Tbx::Dual_quat_cu::dual_quat_from(p_qk_p_alpha.get_non_dual_part(), t);
						p_T_p_alphak = p_T_p_alphak_func(p_qk_p_alpha, dq_bar, dq,
							inv_norm_dq_bar, wk_k);
						p_f_p_alpha[0] = trace_AtB(p_f_p_T, p_T_p_alphak);

						// alpha1
						p_qk_p_alpha[0] = -r[1] * b;
						p_qk_p_alpha[1] = r[1] * r[0] * c;
						p_qk_p_alpha[2] = b + r[1] * r[1] * c;
						p_qk_p_alpha[3] = r[1] * r[2] * c;
						p_qk_p_alpha = Tbx::Dual_quat_cu::dual_quat_from(p_qk_p_alpha.get_non_dual_part(), t);
						p_T_p_alphak = p_T_p_alphak_func(p_qk_p_alpha, dq_bar, dq,
							inv_norm_dq_bar, wk_k);
						p_f_p_alpha[1] = trace_AtB(p_f_p_T, p_T_p_alphak);

						// alpha2
						p_qk_p_alpha[0] = -r[2] * b;
						p_qk_p_alpha[1] = r[2] * r[0] * c;
						p_qk_p_alpha[2] = r[2] * r[1] * c;
						p_qk_p_alpha[3] = b + r[2] * r[2] * c;
						p_qk_p_alpha = Tbx::Dual_quat_cu::dual_quat_from(p_qk_p_alpha.get_non_dual_part(), t);
						p_T_p_alphak = p_T_p_alphak_func(p_qk_p_alpha, dq_bar, dq,
							inv_norm_dq_bar, wk_k);
						p_f_p_alpha[2] = trace_AtB(p_f_p_T, p_T_p_alphak);

						// alpha3
						p_qk_p_alpha = Tbx::Dual_quat_cu(Tbx::Quat_cu(0, 0, 0, 0),
							Tbx::Quat_cu(-q1[1], q1[0], -q1[3], q1[2]));
						p_T_p_alphak = p_T_p_alphak_func(p_qk_p_alpha, dq_bar, dq,
							inv_norm_dq_bar, wk_k);
						p_f_p_alpha[3] = trace_AtB(p_f_p_T, p_T_p_alphak);

						// alpha4
						p_qk_p_alpha = Tbx::Dual_quat_cu(Tbx::Quat_cu(0, 0, 0, 0),
							Tbx::Quat_cu(-q1[2], q1[3], q1[0], -q1[1]));
						p_T_p_alphak = p_T_p_alphak_func(p_qk_p_alpha, dq_bar, dq,
							inv_norm_dq_bar, wk_k);
						p_f_p_alpha[4] = trace_AtB(p_f_p_T, p_T_p_alphak);

						// alpha5
						p_qk_p_alpha = Tbx::Dual_quat_cu(Tbx::Quat_cu(0, 0, 0, 0),
							Tbx::Quat_cu(-q1[3], -q1[2], q1[1], q1[0]));
						p_T_p_alphak = p_T_p_alphak_func(p_qk_p_alpha, dq_bar, dq,
							inv_norm_dq_bar, wk_k);
						p_f_p_alpha[5] = trace_AtB(p_f_p_T, p_T_p_alphak);

						//// reduce--------------------------------------------------
						int shift = knnNodeId * VarPerNode2;
						int shift_g = knnNodeId * VarPerNode;
						for (int i = 0; i < VarPerNode; ++i)
						{
#pragma unroll
							for (int j = 0; j <= i; ++j)
							{
								atomicAdd(&Hd_[shift + j], p_f_p_alpha[i] * p_f_p_alpha[j]);
#ifdef ENABLE_GPU_DUMP_DEBUG
// debug
if (knnNodeId == 390 && i == 5 && j == 1
	&& debug_buffer_pixel_sum2 && debug_buffer_pixel_val
	)
{
	for (int k = 0; k < VarPerNode; k++)
		debug_buffer_pixel_val[(y*imgWidth + x)*VarPerNode + k] =
		p_f_p_alpha[k];
	debug_buffer_pixel_sum2[y*imgWidth + x] = Hd_[shift + j];
}
#endif
							}
							atomicAdd(&g_[shift_g + i], p_f_p_alpha[i] * f);
							shift += VarPerNode;
						}// end for i
					}// end if knnNodeId < nNodes
				}// end for knnK
			}// end if found corr
		}// end function ()
	};

	__global__ void dataTermCombinedKernel(const DataTermCombined cs)
	{
		cs();
	}

	void GpuGaussNewtonSolver::calcDataTerm()
	{
		DataTermCombined cs;
		cs.angleThres = m_param->fusion_nonRigid_angleThreSin;
		cs.distThres = m_param->fusion_nonRigid_distThre;
		cs.Hd_ = m_Hd;
		cs.g_ = m_g;
		cs.imgHeight = m_vmap_cano->rows();
		cs.imgWidth = m_vmap_cano->cols();
		cs.intr = m_intr;
		cs.nmap_cano = *m_nmap_cano;
		cs.nmap_live = *m_nmap_live;
		cs.nmap_warp = *m_nmap_warp;
		cs.vmap_cano = *m_vmap_cano;
		cs.vmap_live = *m_vmap_live;
		cs.vmap_warp = *m_vmap_warp;
		cs.vmapKnn = m_vmapKnn;
		cs.nNodes = m_numNodes;
		cs.Tlw = m_pWarpField->get_rigidTransform();
		cs.psi_data = m_param->fusion_psi_data;

#ifdef ENABLE_GPU_DUMP_DEBUG
		// debugging
		DeviceArray<float> pixelSum2, pixelVal;
		pixelSum2.create(cs.imgHeight*cs.imgWidth);
		hipMemset(pixelSum2.ptr(), 0, pixelSum2.sizeBytes());
		pixelVal.create(cs.imgHeight*cs.imgWidth*VarPerNode);
		hipMemset(pixelVal.ptr(), 0, pixelVal.sizeBytes());
		cs.debug_buffer_pixel_sum2 = pixelSum2;
		cs.debug_buffer_pixel_val = pixelVal;
#endif

		//////////////////////////////
		dim3 block(CTA_SIZE_X, CTA_SIZE_Y);
		dim3 grid(1, 1, 1);
		grid.x = divUp(cs.imgWidth, block.x);
		grid.y = divUp(cs.imgHeight, block.y);
		dataTermCombinedKernel << <grid, block >> >(cs);
		cudaSafeCall(hipGetLastError(), "dataTermCombinedKernel");

		// debugging
#ifdef ENABLE_GPU_DUMP_DEBUG
		{
			std::vector<float> ps, pv;
			pixelSum2.download(ps);
			pixelVal.download(pv);

			FILE* pFile = fopen("D:/tmp/gpu_pixel.txt", "w");
			for (int i = 0; i < ps.size(); i++)
			{
				fprintf(pFile, "%ef %ef %ef %ef %ef %ef %ef\n",
					pv[i * 6 + 0], pv[i * 6 + 1], pv[i * 6 + 2],
					pv[i * 6 + 3], pv[i * 6 + 4], pv[i * 6 + 5],
					ps[i]);
			}
			fclose(pFile);
		}
#endif
	}
#pragma endregion

#pragma region --define sparse structure

#define ENABLE_GPU_DUMP_DEBUG_B

	__global__ void count_Jr_rows_kernel(int* rctptr, int nMaxNodes)
	{
		int i = threadIdx.x + blockIdx.x*blockDim.x;
		if (i < nMaxNodes)
		{
			WarpField::KnnIdx knn = get_nodesKnn(i);
			int numK = -1;
			for (int k = 0; k < WarpField::KnnK; ++k)
			{
				if (knn_k(knn, k) < nMaxNodes)
					numK = k + 1;
			}

			// each node generate 6*maxK rows
			rctptr[i] = numK * 6;
		}
		if (i == nMaxNodes)
			rctptr[i] = 0;
	}

	__global__ void compute_row_map_kernel(GpuGaussNewtonSolver::JrRow2NodeMapper* row2nodeId, 
		const int* rctptr, int nMaxNodes)
	{
		int iNode = threadIdx.x + blockIdx.x*blockDim.x;
		if (iNode < nMaxNodes)
		{
			int row_b = rctptr[iNode];
			int row_e = rctptr[iNode+1];
			for (int r = row_b; r < row_e; r++)
			{
				GpuGaussNewtonSolver::JrRow2NodeMapper mp;
				mp.nodeId = iNode;
				mp.k = (r - row_b) / 6;
				mp.ixyz = r - 6 * mp.k;
				row2nodeId[r] = mp;
			}
		}
	}

	__global__ void compute_Jr_rowPtr_colIdx_kernel(
		int* rptr, int* rptr_coo, int* colIdx,
		const GpuGaussNewtonSolver::JrRow2NodeMapper* row2nodeId, 
		int nMaxNodes, int nRows)
	{
		enum{
			VarPerNode = GpuGaussNewtonSolver::VarPerNode,
			ColPerRow = VarPerNode * 2
		};
		const int iRow = threadIdx.x + blockIdx.x*blockDim.x;
		if (iRow < nRows)
		{
			const int iNode = row2nodeId[iRow].nodeId;
			if (iNode < nMaxNodes)
			{
				WarpField::KnnIdx knn = get_nodesKnn(iNode);
				int knnNodeId = knn_k(knn, row2nodeId[iRow].k);
				if (knnNodeId < nMaxNodes)
				{
					int col_b = iRow*ColPerRow;
					rptr[iRow] = col_b;

					// each row 2*VerPerNode Cols
					// 1. self
					for (int j = 0; j < VarPerNode; j++, col_b++)
					{
						rptr_coo[col_b] = iRow;
						colIdx[col_b] = iNode*VarPerNode + j;
					}// j
					// 2. neighbor
					for (int j = 0; j < VarPerNode; j++, col_b++)
					{
						rptr_coo[col_b] = iRow;
						colIdx[col_b] = knnNodeId*VarPerNode + j;
					}// j
				}// end if knnNodeId
			}
		}// end if iRow < nRows
		if (iRow == nRows)
			rptr[nRows] = nRows * ColPerRow;
	}

	void GpuGaussNewtonSolver::initSparseStructure()
	{
		// 1. compute Jr structure ==============================================
		// 1.0. decide the total rows we have for each nodes
		{
			dim3 block(CTA_SIZE);
			dim3 grid(divUp(m_numNodes, block.x));
			count_Jr_rows_kernel << <grid, block >> >(m_Jr_RowCounter.ptr(), m_numNodes);
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::initSparseStructure::count_Jr_rows_kernel");
			thrust_wrapper::exclusive_scan(m_Jr_RowCounter.ptr(), m_Jr_RowCounter.ptr(), m_numNodes + 1);
			cudaSafeCall(hipMemcpy(&m_Jrrows, m_Jr_RowCounter.ptr() + m_numNodes,
				sizeof(int), hipMemcpyDeviceToHost), "copy Jr rows to host");
		}

		// 1.1. collect nodes edges info:
		//	each low-level nodes are connected to k higher level nodes
		//	but the connections are not stored for the higher level nodes
		//  thus when processing each node, we add 2*k edges, w.r.t. 2*k*3 rows: each (x,y,z) a row
		//	for each row, there are exactly 2*VarPerNode values
		//	after this step, we can get the CSR/COO structure
		{
			dim3 block(CTA_SIZE);
			dim3 grid(divUp(m_numNodes, block.x));
			compute_row_map_kernel << <grid, block >> >(m_Jr_RowMap2NodeId.ptr(), m_Jr_RowCounter.ptr(), m_numNodes);
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::initSparseStructure::compute_row_map_kernel");
		}
		{
			dim3 block(CTA_SIZE);
			dim3 grid(divUp(m_Jrrows, block.x));
			compute_Jr_rowPtr_colIdx_kernel << <grid, block >> >(m_Jr_RowPtr.ptr(),
				m_Jr_RowPtr_coo.ptr(), m_Jr_ColIdx.ptr(), m_Jr_RowMap2NodeId.ptr(), m_numNodes, m_Jrrows);
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::initSparseStructure::compute_Jr_rowPtr_kernel");
			cudaSafeCall(hipMemcpy(&m_Jrnnzs, m_Jr_RowPtr.ptr() + m_Jrrows,
				sizeof(int), hipMemcpyDeviceToHost), "copy Jr nnz to host");
		}

		// 2. compute Jrt structure ==============================================
		// 2.1. fill (row, col) as (col, row) from Jr and sort.
		hipMemcpy(m_Jrt_RowPtr_coo.ptr(), m_Jr_ColIdx.ptr(), m_Jrnnzs*sizeof(int), hipMemcpyDeviceToDevice);
		hipMemcpy(m_Jrt_ColIdx.ptr(), m_Jr_RowPtr_coo.ptr(), m_Jrnnzs*sizeof(int), hipMemcpyDeviceToDevice);
		modergpu_wrapper::mergesort_by_key(m_Jrt_RowPtr_coo.ptr(), m_Jrt_ColIdx.ptr(), m_Jrnnzs);
		cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::initSparseStructure::mergesort_by_key");

		// 2.2. extract CSR rowptr info.
		if (HIPSPARSE_STATUS_SUCCESS != hipsparseXcoo2csr(m_cuSparseHandle,
			m_Jrt_RowPtr_coo.ptr(), m_Jrnnzs, m_Jrcols,
			m_Jrt_RowPtr.ptr(), HIPSPARSE_INDEX_BASE_ZERO))
			throw std::exception("GpuGaussNewtonSolver::initSparseStructure::hipsparseXcoo2csr failed");

		// 3. compute Jrt*Jr stucture ==============================================
		// quite slow...
		hipsparseSetPointerMode(m_cuSparseHandle, HIPSPARSE_POINTER_MODE_HOST);
		hipsparseXcsrgemmNnz(m_cuSparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			m_Jrcols, m_Jrcols, m_Jrrows, 
			m_Jrt_desc, m_Jrnnzs, m_Jrt_RowPtr.ptr(), m_Jrt_ColIdx.ptr(), 
			m_Jr_desc, m_Jrnnzs, m_Jr_RowPtr.ptr(), m_Jr_ColIdx.ptr(), 
			m_JrtJr_desc, m_JrtJr_RowPtr.ptr(), &m_JrtJr_nnzs);
		if (m_JrtJr_nnzs > m_JrtJr_ColIdx.size())
			throw std::exception("Jr'Jr: size out of range!");
		hipsparseScsrgemm(m_cuSparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			m_Jrcols, m_Jrcols, m_Jrrows,
			m_Jrt_desc, m_Jrnnzs, m_Jrt_val.ptr(), m_Jrt_RowPtr.ptr(), m_Jrt_ColIdx.ptr(),
			m_Jr_desc, m_Jrnnzs, m_Jr_val.ptr(), m_Jr_RowPtr.ptr(), m_Jr_ColIdx.ptr(),
			m_JrtJr_desc, m_JrtJr_val.ptr(), m_JrtJr_RowPtr.ptr(), m_JrtJr_ColIdx.ptr());

#ifdef ENABLE_GPU_DUMP_DEBUG_B
		{
			std::vector<int> host_Jr_rowptr, host_Jr_colIdx, host_Jr_rowptr_coo;
			std::vector<int> host_Jrt_rowptr, host_Jrt_colIdx, host_Jrt_rowptr_coo;
			std::vector<int> host_JrtJr_rowptr, host_JrtJr_colIdx;
			std::vector<float> host_Jr_val, host_Jrt_val, host_JrtJr_val;
			m_Jr_RowPtr.download(host_Jr_rowptr);
			m_Jr_ColIdx.download(host_Jr_colIdx);
			m_Jr_RowPtr_coo.download(host_Jr_rowptr_coo);
			m_Jr_val.download(host_Jr_val);
			m_Jrt_RowPtr.download(host_Jrt_rowptr);
			m_Jrt_ColIdx.download(host_Jrt_colIdx);
			m_Jrt_RowPtr_coo.download(host_Jrt_rowptr_coo);
			m_Jrt_val.download(host_Jrt_val);
			m_JrtJr_RowPtr.download(host_JrtJr_rowptr);
			m_JrtJr_ColIdx.download(host_JrtJr_colIdx);
			m_JrtJr_val.download(host_JrtJr_val);

			FILE* pFile = fopen("D:/tmp/gpu_Jr.txt", "w");
			if (pFile)
			{
				for (int r = 0; r < m_Jrrows; r++)
				{
					int cb = host_Jr_rowptr[r], ce = host_Jr_rowptr[r + 1];
					for (int ic = cb; ic < ce; ic++)
					{
						if (host_Jr_rowptr_coo[ic] != r)
						{
							printf("error: Jr coo not matched: %d %d\n", host_Jr_rowptr_coo[ic], r);
							system("pause");
						}
						fprintf(pFile, "%d %d %f\n", r, host_Jr_colIdx[ic], host_Jr_val[ic]);
					}
				}
				fclose(pFile);
			}

			FILE* pFile1 = fopen("D:/tmp/gpu_Jrt.txt", "w");
			if (pFile1)
			{
				for (int r = 0; r < m_Jrcols; r++)
				{
					int cb = host_Jrt_rowptr[r], ce = host_Jrt_rowptr[r + 1];
					for (int ic = cb; ic < ce; ic++)
					{
						if (ic >= host_Jrt_rowptr_coo.size())
						{
							printf("out of range: [%d] %d > %d\n", r, ic, host_Jrt_rowptr_coo.size());
							system("pause");
						}
						if (host_Jrt_rowptr_coo[ic] != r)
						{
							printf("error: Jrt coo not matched: [%d] %d %d\n", ic, host_Jrt_rowptr_coo[ic], r);
							system("pause");
						}
						fprintf(pFile1, "%d %d %f\n", r, host_Jrt_colIdx[ic], host_Jrt_val[ic]);
					}
				}
				fclose(pFile1);
			}

			FILE* pFile2 = fopen("D:/tmp/gpu_JrtJr.txt", "w");
			if (pFile1)
			{
				for (int r = 0; r < m_Jrcols; r++)
				{
					int cb = host_JrtJr_rowptr[r], ce = host_JrtJr_rowptr[r + 1];
					for (int ic = cb; ic < ce; ic++)
						fprintf(pFile1, "%d %d %f\n", r, host_JrtJr_colIdx[ic], host_JrtJr_val[ic]);
				}
				fclose(pFile1);
			}
		}
#endif
	}

#pragma endregion

#pragma region --define sparse structure
	struct RegTermJacobi
	{
		typedef WarpField::KnnIdx KnnIdx;
		typedef WarpField::IdxType IdxType;
		typedef GpuGaussNewtonSolver::JrRow2NodeMapper Mapper;
		enum
		{
			KnnK = WarpField::KnnK,
			VarPerNode = GpuGaussNewtonSolver::VarPerNode,
			VarPerNode2 = VarPerNode*VarPerNode,
			ColPerRow = VarPerNode * 2
		};

		int nNodes;
		int nRows;
		const Mapper* rows2nodeIds;
		const int* rptr;
		const int* cidx;
		mutable float* vptr;
		mutable float* fptr;

		float psi_reg;
		float lambda;

		__device__ __forceinline__  Tbx::Dual_quat_cu p_qk_p_alpha_func(Tbx::Dual_quat_cu dq, int i)const
		{
			Tbx::Vec3 t, r;
			float b, c, n;
			Tbx::Quat_cu q0(0, 0, 0, 0), q1 = dq.get_non_dual_part();
			switch (i)
			{
			case 0:
				dq.to_twist(r, t);
				n = r.norm();
				if (n > Tbx::Dual_quat_cu::epsilon())
				{
					b = sin(n) / n;
					c = (cos(n) - b) / (n*n);
					q0.coeff0 = -r.x * b;
					q0.coeff1 = b + r.x*r.x*c;
					q0.coeff2 = r.x*r.y*c;
					q0.coeff3 = r.x*r.z*c;
				}
				else
				{
					q0.coeff0 = 0;
					q0.coeff1 = 1;
					q0.coeff2 = 0;
					q0.coeff3 = 0;
				}

				q1.coeff0 = (t.x * q0.coeff1 + t.y * q0.coeff2 + t.z * q0.coeff3) * (-0.5);
				q1.coeff1 = (t.x * q0.coeff0 + t.y * q0.coeff3 - t.z * q0.coeff2) * 0.5;
				q1.coeff2 = (-t.x * q0.coeff3 + t.y * q0.coeff0 + t.z * q0.coeff1) * 0.5;
				q1.coeff3 = (t.x * q0.coeff2 - t.y * q0.coeff1 + t.z * q0.coeff0) * 0.5;
				return Tbx::Dual_quat_cu(q0, q1);
			case 1:
				dq.to_twist(r, t);
				n = r.norm();
				if (n > Tbx::Dual_quat_cu::epsilon())
				{
					b = sin(n) / n;
					c = (cos(n) - b) / (n*n);
					q0.coeff0 = -r.y * b;
					q0.coeff1 = r.y*r.x*c;
					q0.coeff2 = b + r.y*r.y*c;
					q0.coeff3 = r.y*r.z*c;
				}
				else
				{
					q0.coeff0 = 0;
					q0.coeff1 = 0;
					q0.coeff2 = 1;
					q0.coeff3 = 0;
				}

				q1.coeff0 = (t.x * q0.coeff1 + t.y * q0.coeff2 + t.z * q0.coeff3) * (-0.5);
				q1.coeff1 = (t.x * q0.coeff0 + t.y * q0.coeff3 - t.z * q0.coeff2) * 0.5;
				q1.coeff2 = (-t.x * q0.coeff3 + t.y * q0.coeff0 + t.z * q0.coeff1) * 0.5;
				q1.coeff3 = (t.x * q0.coeff2 - t.y * q0.coeff1 + t.z * q0.coeff0) * 0.5;
				return Tbx::Dual_quat_cu(q0, q1);
			case 2:
				dq.to_twist(r, t);
				n = r.norm();
				if (n > Tbx::Dual_quat_cu::epsilon())
				{
					b = sin(n) / n;
					c = (cos(n) - b) / (n*n);

					q0.coeff0 = -r.z * b;
					q0.coeff1 = r.z*r.x*c;
					q0.coeff2 = r.z*r.y*c;
					q0.coeff3 = b + r.z*r.z*c;
				}
				else
				{
					q0.coeff0 = 0;
					q0.coeff1 = 0;
					q0.coeff2 = 0;
					q0.coeff3 = 1;
				}

				q1.coeff0 = (t.x * q0.coeff1 + t.y * q0.coeff2 + t.z * q0.coeff3) * (-0.5);
				q1.coeff1 = (t.x * q0.coeff0 + t.y * q0.coeff3 - t.z * q0.coeff2) * 0.5;
				q1.coeff2 = (-t.x * q0.coeff3 + t.y * q0.coeff0 + t.z * q0.coeff1) * 0.5;
				q1.coeff3 = (t.x * q0.coeff2 - t.y * q0.coeff1 + t.z * q0.coeff0) * 0.5;
				return Tbx::Dual_quat_cu(q0, q1);
			case 3:
				return Tbx::Dual_quat_cu(q0, Tbx::Quat_cu(-q1.coeff1, q1.coeff0, -q1.coeff3, q1.coeff2))*0.5;
			case 4:
				return Tbx::Dual_quat_cu(q0, Tbx::Quat_cu(-q1.coeff2, q1.coeff3, q1.coeff0, -q1.coeff1))*0.5;
			case 5:
				return Tbx::Dual_quat_cu(q0, Tbx::Quat_cu(-q1.coeff3, -q1.coeff2, q1.coeff1, q1.coeff0))*0.5;
			default:
				return Tbx::Dual_quat_cu();
			}
		}

		__device__ __forceinline__  Tbx::Vec3 reg_term_penalty(Tbx::Vec3 f)const
		{
			// the robust Huber penelty gradient
			Tbx::Vec3 df;
			float norm = f.norm();
			if (norm < psi_reg)
				df = f;
			else
			for (int k = 0; k < 3; k++)
				df[k] = sign(f[k])*psi_reg;
			return df;
		}

		__device__ __forceinline__  Tbx::Transfo p_SE3_p_alpha_func(Tbx::Dual_quat_cu dq, int i)const
		{
			Tbx::Transfo T = Tbx::Transfo::empty();
			Tbx::Dual_quat_cu p_dq_p_alphai = p_qk_p_alpha_func(dq, i) * 2.f;

			//// evaluate p_dqi_p_alphak, heavily hard code here
			//// this hard code is crucial to the performance 
			// 0:
			// (0, -z0, y0, x1,
			// z0, 0, -x0, y1,
			//-y0, x0, 0, z1,
			// 0, 0, 0, 0) * 2;
			float p_dqi_p_alphak = p_dq_p_alphai[0];
			T[1] += -dq[3] * p_dqi_p_alphak;
			T[2] += dq[2] * p_dqi_p_alphak;
			T[3] += dq[5] * p_dqi_p_alphak;
			T[4] += dq[3] * p_dqi_p_alphak;
			T[6] += -dq[1] * p_dqi_p_alphak;
			T[7] += dq[6] * p_dqi_p_alphak;
			T[8] += -dq[2] * p_dqi_p_alphak;
			T[9] += dq[1] * p_dqi_p_alphak;
			T[11] += dq[7] * p_dqi_p_alphak;

			// 1
			//( 0, y0, z0, -w1,
			//	y0, -2 * x0, -w0, -z1,
			//	z0, w0, -2 * x0, y1,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[1];
			T[1] += dq[2] * p_dqi_p_alphak;
			T[2] += dq[3] * p_dqi_p_alphak;
			T[3] += -dq[4] * p_dqi_p_alphak;
			T[4] += dq[2] * p_dqi_p_alphak;
			T[5] += -dq[1] * p_dqi_p_alphak * 2;
			T[6] += -dq[0] * p_dqi_p_alphak;
			T[7] += -dq[7] * p_dqi_p_alphak;
			T[8] += dq[3] * p_dqi_p_alphak;
			T[9] += dq[0] * p_dqi_p_alphak;
			T[10] += -dq[1] * p_dqi_p_alphak * 2;
			T[11] += dq[6] * p_dqi_p_alphak;

			// 2.
			// (-2 * y0, x0, w0, z1,
			//	x0, 0, z0, -w1,
			//	-w0, z0, -2 * y0, -x1,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[2];
			T[0] += -dq[2] * p_dqi_p_alphak * 2;
			T[1] += dq[1] * p_dqi_p_alphak;
			T[2] += dq[0] * p_dqi_p_alphak;
			T[3] += dq[7] * p_dqi_p_alphak;
			T[4] += dq[1] * p_dqi_p_alphak;
			T[6] += dq[3] * p_dqi_p_alphak;
			T[7] += -dq[4] * p_dqi_p_alphak;
			T[8] += -dq[0] * p_dqi_p_alphak;
			T[9] += dq[3] * p_dqi_p_alphak;
			T[10] += -dq[2] * p_dqi_p_alphak * 2;
			T[11] += -dq[5] * p_dqi_p_alphak;

			// 3.
			// (-2 * z0, -w0, x0, -y1,
			//	w0, -2 * z0, y0, x1,
			//	x0, y0, 0, -w1,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[3];
			T[0] += -dq[3] * p_dqi_p_alphak * 2;
			T[1] += -dq[0] * p_dqi_p_alphak;
			T[2] += dq[1] * p_dqi_p_alphak;
			T[3] += -dq[6] * p_dqi_p_alphak;
			T[4] += dq[0] * p_dqi_p_alphak;
			T[5] += -dq[3] * p_dqi_p_alphak * 2;
			T[6] += dq[2] * p_dqi_p_alphak;
			T[7] += dq[5] * p_dqi_p_alphak;
			T[8] += dq[1] * p_dqi_p_alphak;
			T[9] += dq[2] * p_dqi_p_alphak;
			T[11] += -dq[4] * p_dqi_p_alphak;

			// 4.
			//( 0, 0, 0, -x0,
			//	0, 0, 0, -y0,
			//	0, 0, 0, -z0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[4];
			T[3] += -dq[1] * p_dqi_p_alphak;
			T[7] += -dq[2] * p_dqi_p_alphak;
			T[11] += -dq[3] * p_dqi_p_alphak;

			// 5. 
			// (0, 0, 0, w0,
			//	0, 0, 0, z0,
			//	0, 0, 0, -y0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[5];
			T[3] += dq[0] * p_dqi_p_alphak;
			T[7] += dq[3] * p_dqi_p_alphak;
			T[11] += -dq[2] * p_dqi_p_alphak;

			// 6. 
			// (0, 0, 0, -z0,
			//	0, 0, 0, w0,
			//	0, 0, 0, x0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[6];
			T[3] += -dq[3] * p_dqi_p_alphak;
			T[7] += dq[0] * p_dqi_p_alphak;
			T[11] += dq[1] * p_dqi_p_alphak;

			// 7.
			// (0, 0, 0, y0,
			//	0, 0, 0, -x0,
			//	0, 0, 0, w0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[7];
			T[3] += dq[2] * p_dqi_p_alphak;
			T[7] += -dq[1] * p_dqi_p_alphak;
			T[11] += dq[0] * p_dqi_p_alphak;

			return T;
		}

		__device__ __forceinline__ void operator () () const
		{
			const int iRow = (threadIdx.x + blockIdx.x * blockDim.x)*6;
		
			if (iRow >= nRows)
				return;

			Mapper mapper = rows2nodeIds[iRow];
			int knnNodeId = knn_k(get_nodesKnn(mapper.nodeId), mapper.k);

			if (knnNodeId >= nNodes)
				return;

			int cooPos = rptr[iRow];

			Tbx::Dual_quat_cu dqi, dqj;
			Tbx::Vec3 ri, ti, rj, tj;
			get_twist(mapper.nodeId, ri, ti);
			get_twist(knnNodeId, rj, tj);
			dqi.from_twist(ri, ti);
			dqj.from_twist(rj, tj);

			float4 nodeVwi = get_nodesVw(mapper.nodeId);
			float4 nodeVwj = get_nodesVw(knnNodeId);
			Tbx::Point3 vi(convert(read_float3_4(nodeVwi)));
			Tbx::Point3 vj(convert(read_float3_4(nodeVwj)));
			float alpha_ij = max(1.f / nodeVwi.w, 1.f / nodeVwj.w);
			float ww = sqrt(lambda * alpha_ij);

			// energy=============================================
			Tbx::Vec3 val = dqi.transform(Tbx::Point3(vj)) - dqj.transform(Tbx::Point3(vj));
			val = reg_term_penalty(val);

			fptr[iRow + 0] = val.x * ww;
			fptr[iRow + 1] = val.y * ww;
			fptr[iRow + 2] = val.z * ww;

			Tbx::Vec3 val1 = dqi.transform(Tbx::Point3(vi)) - dqj.transform(Tbx::Point3(vi));
			val1 = reg_term_penalty(val1);
			fptr[iRow + 3] = val1.x * ww;
			fptr[iRow + 4] = val1.y * ww;
			fptr[iRow + 5] = val1.z * ww;

			// jacobi=============================================
			for (int ialpha = 0; ialpha < VarPerNode; ialpha++)
			{
				Tbx::Transfo p_Ti_p_alpha = p_SE3_p_alpha_func(dqi, ialpha);
				Tbx::Transfo p_Tj_p_alpha = p_SE3_p_alpha_func(dqj, ialpha);

				// partial_psi_partial_alpha
				Tbx::Vec3 p_psi_p_alphai_j = (p_Ti_p_alpha * vj) * ww;
				Tbx::Vec3 p_psi_p_alphaj_j = (p_Tj_p_alpha * vj) * (-ww);
				Tbx::Vec3 p_psi_p_alphai_i = (p_Ti_p_alpha * vi) * ww;
				Tbx::Vec3 p_psi_p_alphaj_i = (p_Tj_p_alpha * vi) * (-ww);

				for (int ixyz = 0; ixyz < 3; ixyz++)
				{
					int pos = cooPos + ixyz*ColPerRow + ialpha;
					vptr[pos] = p_psi_p_alphai_j[ixyz];
					vptr[pos + VarPerNode] = p_psi_p_alphaj_j[ixyz];
					pos += 3 * ColPerRow;
					vptr[pos] = p_psi_p_alphai_i[ixyz];
					vptr[pos + VarPerNode] = p_psi_p_alphaj_i[ixyz];
				}
			}// end for ialpha
		}// end function ()
	};

	__global__ void calcRegTerm_kernel(RegTermJacobi rj)
	{
		rj();
	}

	void GpuGaussNewtonSolver::calcRegTerm()
	{
		RegTermJacobi rj;
		rj.cidx = m_Jr_ColIdx.ptr();
		rj.lambda = m_param->fusion_lambda;
		rj.nNodes = m_numNodes;
		rj.nRows = m_Jrrows;
		rj.psi_reg = m_param->fusion_psi_reg;
		rj.rows2nodeIds = m_Jr_RowMap2NodeId;
		rj.rptr = m_Jr_RowPtr.ptr();
		rj.vptr = m_Jr_val.ptr();
		rj.fptr = m_f_r.ptr();

		dim3 block(CTA_SIZE);
		dim3 grid(divUp(m_Jrrows / 6, block.x));

		calcRegTerm_kernel << <grid, block >> >(rj);
		cudaSafeCall(hipGetLastError(), "calcRegTerm_kernel");

		// 2. compute Jrt ==============================================
		// 2.1. fill (row, col) as (col, row) from Jr and sort.
		hipMemcpy(m_Jrt_RowPtr_coo.ptr(), m_Jr_ColIdx.ptr(), m_Jrnnzs*sizeof(int), hipMemcpyDeviceToDevice);
		hipMemcpy(m_Jrt_val.ptr(), m_Jr_val.ptr(), m_Jrnnzs*sizeof(float), hipMemcpyDeviceToDevice);
		modergpu_wrapper::mergesort_by_key(m_Jrt_RowPtr_coo.ptr(), m_Jrt_val.ptr(), m_Jrnnzs);
		cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::calcRegTerm::mergesort_by_key");

		// 3. compute JrtJr
		hipsparseScsrgemm(m_cuSparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			m_Jrcols, m_Jrcols, m_Jrrows,
			m_Jrt_desc, m_Jrnnzs, m_Jrt_val.ptr(), m_Jrt_RowPtr.ptr(), m_Jrt_ColIdx.ptr(),
			m_Jr_desc, m_Jrnnzs, m_Jr_val.ptr(), m_Jr_RowPtr.ptr(), m_Jr_ColIdx.ptr(),
			m_JrtJr_desc, m_JrtJr_val.ptr(), m_JrtJr_RowPtr.ptr(), m_JrtJr_ColIdx.ptr());

#ifdef ENABLE_GPU_DUMP_DEBUG_B
		{
			std::vector<int> host_Jr_rowptr, host_Jr_colIdx, host_Jr_rowptr_coo;
			std::vector<int> host_Jrt_rowptr, host_Jrt_colIdx, host_Jrt_rowptr_coo;
			std::vector<int> host_JrtJr_rowptr, host_JrtJr_colIdx;
			std::vector<float> host_Jr_val, host_Jrt_val, host_JrtJr_val;
			m_Jr_RowPtr.download(host_Jr_rowptr);
			m_Jr_ColIdx.download(host_Jr_colIdx);
			m_Jr_RowPtr_coo.download(host_Jr_rowptr_coo);
			m_Jr_val.download(host_Jr_val);
			m_Jrt_RowPtr.download(host_Jrt_rowptr);
			m_Jrt_ColIdx.download(host_Jrt_colIdx);
			m_Jrt_RowPtr_coo.download(host_Jrt_rowptr_coo);
			m_Jrt_val.download(host_Jrt_val);
			m_JrtJr_RowPtr.download(host_JrtJr_rowptr);
			m_JrtJr_ColIdx.download(host_JrtJr_colIdx);
			m_JrtJr_val.download(host_JrtJr_val);

			FILE* pFile = fopen("D:/tmp/gpu_Jr.txt", "w");
			if (pFile)
			{
				for (int r = 0; r < m_Jrrows; r++)
				{
					int cb = host_Jr_rowptr[r], ce = host_Jr_rowptr[r + 1];
					for (int ic = cb; ic < ce; ic++)
						fprintf(pFile, "%d %d %f\n", r, host_Jr_colIdx[ic], host_Jr_val[ic]);
				}
				fclose(pFile);
			}

			FILE* pFile1 = fopen("D:/tmp/gpu_Jrt.txt", "w");
			if (pFile1)
			{
				for (int r = 0; r < m_Jrcols; r++)
				{
					int cb = host_Jrt_rowptr[r], ce = host_Jrt_rowptr[r + 1];
					for (int ic = cb; ic < ce; ic++)
						fprintf(pFile1, "%d %d %f\n", r, host_Jrt_colIdx[ic], host_Jrt_val[ic]);
				}
				fclose(pFile1);
			}

			FILE* pFile2 = fopen("D:/tmp/gpu_JrtJr.txt", "w");
			if (pFile1)
			{
				for (int r = 0; r < m_Jrcols; r++)
				{
					int cb = host_JrtJr_rowptr[r], ce = host_JrtJr_rowptr[r + 1];
					for (int ic = cb; ic < ce; ic++)
						fprintf(pFile1, "%d %d %f\n", r, host_JrtJr_colIdx[ic], host_JrtJr_val[ic]);
				}
				fclose(pFile1);
			}
		}
#endif
	}
#pragma endregion
}