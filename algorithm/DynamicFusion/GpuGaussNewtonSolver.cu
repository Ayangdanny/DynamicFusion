#include "hip/hip_runtime.h"
#include "GpuGaussNewtonSolver.h"
#include "device_utils.h"
namespace dfusion
{
	texture<WarpField::KnnIdx, hipTextureType1D, hipReadModeElementType> g_nodesKnnTex;
	texture<float4, hipTextureType1D, hipReadModeElementType> g_nodesVwTex;
	texture<float, hipTextureType1D, hipReadModeElementType> g_twistTex;

	__device__ __forceinline__ float4 get_nodesVw(int i)
	{
		return tex1Dfetch(g_nodesVwTex, i);
	}

	__device__ __forceinline__ WarpField::KnnIdx get_nodesKnn(int i)
	{
		return tex1Dfetch(g_nodesKnnTex, i);
	}

	__device__ __forceinline__ void get_twist(int i, Tbx::Vec3& r, Tbx::Vec3& t)
	{
		int i6 = i * 6;
		r.x = tex1Dfetch(g_twistTex, i6++);
		r.y = tex1Dfetch(g_twistTex, i6++);
		r.z = tex1Dfetch(g_twistTex, i6++);
		t.x = tex1Dfetch(g_twistTex, i6++);
		t.y = tex1Dfetch(g_twistTex, i6++);
		t.z = tex1Dfetch(g_twistTex, i6++);
	}

	// map the lower part to full 6x6 matrix
	__constant__ int g_lower_2_full_6x6[21] = {
		0,
		6, 7,
		12, 13, 14,
		18, 19, 20, 21,
		24, 25, 26, 27, 28,
		30, 31, 32, 33, 34, 35
	};
	__constant__ int g_lfull_2_lower_6x6[6][6] = {
		{ 0, -1, -1, -1, -1, -1 },
		{ 1, 2, -1, -1, -1, -1 },
		{ 3, 4, 5, -1, -1, -1 },
		{ 6, 7, 8, 9, -1, -1 },
		{ 10, 11, 12, 13, 14, -1 },
		{ 15, 16, 17, 18, 19, 20 },
	};

#define D_1_DIV_6 0.166666667

	__device__ __forceinline__ float3 read_float3_4(float4 a)
	{
		return make_float3(a.x, a.y, a.z);
	}

	__device__ __forceinline__ float sqr(float a)
	{
		return a*a;
	}

	__device__ __forceinline__ float pow3(float a)
	{
		return a*a*a;
	}

	__device__ __forceinline__ float sign(float a)
	{
		return a>0 - a<0;
	}

	__device__ __forceinline__ WarpField::IdxType& knn_k(WarpField::KnnIdx& knn, int k)
	{
		return ((WarpField::IdxType*)(&knn))[k];
	}
	__device__ __forceinline__ const WarpField::IdxType& knn_k(const WarpField::KnnIdx& knn, int k)
	{
		return ((WarpField::IdxType*)(&knn))[k];
	}

#pragma region --bind textures
	void GpuGaussNewtonSolver::bindTextures()
	{
		if (1)
		{
			size_t offset;
			hipChannelFormatDesc desc = hipCreateChannelDesc<WarpField::KnnIdx>();
			hipBindTexture(&offset, &g_nodesKnnTex, m_nodesKnn.ptr(), &desc, 
				m_nodesKnn.size() * sizeof(WarpField::KnnIdx));
			if (offset != 0)
				throw std::exception("GpuGaussNewtonSolver::bindTextures(): non-zero-offset error!");
		}
		if (1)
		{
			size_t offset;
			hipChannelFormatDesc desc = hipCreateChannelDesc<float4>();
			hipBindTexture(&offset, &g_nodesVwTex, m_nodesVw.ptr(), &desc, 
				m_nodesVw.size() * sizeof(float4));
			if (offset != 0)
				throw std::exception("GpuGaussNewtonSolver::bindTextures(): non-zero-offset error!");
		}
		if (1)
		{
			size_t offset;
			hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
			hipBindTexture(&offset, &g_twistTex, m_twist.ptr(), &desc,
				m_twist.size() * sizeof(float));
			if (offset != 0)
				throw std::exception("GpuGaussNewtonSolver::bindTextures(): non-zero-offset error!");
		}
	}

	void GpuGaussNewtonSolver::unBindTextures()
	{
		hipUnbindTexture(g_twistTex);
		hipUnbindTexture(g_nodesVwTex);
		hipUnbindTexture(g_nodesKnnTex);
	}
#pragma endregion

#pragma region --calc data term

	struct DataTermCombined
	{
		typedef WarpField::KnnIdx KnnIdx;
		typedef WarpField::IdxType IdxType;
		enum
		{
			CTA_SIZE_X = GpuGaussNewtonSolver::CTA_SIZE_X,
			CTA_SIZE_Y = GpuGaussNewtonSolver::CTA_SIZE_Y,
			CTA_SIZE = CTA_SIZE_X * CTA_SIZE_Y,
			KnnK = WarpField::KnnK,
			VarPerNode = GpuGaussNewtonSolver::VarPerNode,
			VarPerNode2 = VarPerNode*VarPerNode,
			LowerPartNum = GpuGaussNewtonSolver::LowerPartNum,
		};

		PtrStep<float4> vmap_live;
		PtrStep<float4> nmap_live;
		PtrStep<float4> vmap_warp;
		PtrStep<float4> nmap_warp;
		PtrStep<float4> vmap_cano;
		PtrStep<float4> nmap_cano;
		PtrStep<KnnIdx> vmapKnn;
		float* Hd_;
		float* g_;

		Intr intr;
		Tbx::Transfo Tlw;

		int imgWidth;
		int imgHeight;
		int nNodes;

		float distThres;
		float angleThres;
		float psi_reg;
		float psi_data;

		__device__ __forceinline__ float data_term_energy(float f)const
		{
			return psi_data*psi_data * D_1_DIV_6 * (1 - pow3(max(0.f, 1.f - sqr(f / psi_data))));

			//// the robust Tukey penelty gradient
			//if (abs(f) <= psi_data)
			//	return psi_data*psi_data * D_1_DIV_6 * (1 - pow3(1 - sqr(f / psi_data)));
			//else
			//	return psi_data*psi_data * D_1_DIV_6;
		}

		__device__ __forceinline__ float data_term_penalty(float f)const
		{
			return f * sqr(max(0.f, 1.f - sqr(f / psi_data)));
			//// the robust Tukey penelty gradient
			//if (abs(f) <= psi_data)
			//	return f * sqr(1 - sqr(f / psi_data));
			//else
			//	return 0;
		}

		__device__ __forceinline__ Tbx::Transfo outer_product(Tbx::Vec3 n, Tbx::Point3 v)const
		{
			return Tbx::Transfo(
				n.x*v.x, n.x*v.y, n.x*v.z, n.x,
				n.y*v.x, n.y*v.y, n.y*v.z, n.y,
				n.z*v.x, n.z*v.y, n.z*v.z, n.z,
				0, 0, 0, 0
				);
		}

		__device__ __forceinline__ Tbx::Dual_quat_cu p_qk_p_alpha_func(Tbx::Dual_quat_cu dq, int i)const
		{
			Tbx::Quat_cu q0(0, 0, 0, 0), q1 = dq.get_non_dual_part();

			if (i < 3)
			{
				Tbx::Vec3 t, r;
				float b, c, n;
				dq.to_twist(r, t);
				n = r.norm();
				b = sin(n) / n;
				c = (cos(n) - b) / (n*n);
				if (n > Tbx::Dual_quat_cu::epsilon())
				{
					q0[0] = -r[i] * b;
					q0[1] = r[i] * r[0] * c;
					q0[2] = r[i] * r[1] * c;
					q0[3] = r[i] * r[2] * c;
					q0[i+1] += b;
				}
				else
				{
					q0[i+1] = 1;
				}

				q1.coeff0 = (t.x * q0.coeff1 + t.y * q0.coeff2 + t.z * q0.coeff3) * (-0.5);
				q1.coeff1 = (t.x * q0.coeff0 + t.y * q0.coeff3 - t.z * q0.coeff2) * 0.5;
				q1.coeff2 = (-t.x * q0.coeff3 + t.y * q0.coeff0 + t.z * q0.coeff1) * 0.5;
				q1.coeff3 = (t.x * q0.coeff2 - t.y * q0.coeff1 + t.z * q0.coeff0) * 0.5;
				return Tbx::Dual_quat_cu(q0, q1);
			}
			else
			{
				switch (i)
				{
				case 3:
					return Tbx::Dual_quat_cu(q0, Tbx::Quat_cu(-q1.coeff1, q1.coeff0, -q1.coeff3, q1.coeff2)*0.5f);
				case 4:
					return Tbx::Dual_quat_cu(q0, Tbx::Quat_cu(-q1.coeff2, q1.coeff3, q1.coeff0, -q1.coeff1)*0.5f);
				case 5:
					return Tbx::Dual_quat_cu(q0, Tbx::Quat_cu(-q1.coeff3, -q1.coeff2, q1.coeff1, q1.coeff0)*0.5f);
				default:
					return Tbx::Dual_quat_cu::identity();
				}
			}
		}

		__device__ __forceinline__ float trace_AtB(Tbx::Transfo A, Tbx::Transfo B)const
		{
			float sum = 0;
			for (int i = 0; i < 16; i++)
				sum += A[i] * B[i];
			return sum;
		}

		__device__ __forceinline__ Tbx::Transfo p_SE3_p_dq_func(Tbx::Dual_quat_cu dq, int i)const
		{
			Tbx::Quat_cu q0 = dq.get_non_dual_part();
			Tbx::Quat_cu q1 = dq.get_dual_part();
			float x0 = q0.i(), y0 = q0.j(), z0 = q0.k(), w0 = q0.w();
			float x1 = q1.i(), y1 = q1.j(), z1 = q1.k(), w1 = q1.w();
			switch (i)
			{
			case 0:
				return Tbx::Transfo(
					0, -z0, y0, x1,
					z0, 0, -x0, y1,
					-y0, x0, 0, z1,
					0, 0, 0, 0) * 2;
			case 1:
				return Tbx::Transfo(
					0, y0, z0, -w1,
					y0, -2 * x0, -w0, -z1,
					z0, w0, -2 * x0, y1,
					0, 0, 0, 0) * 2;
			case 2:
				return Tbx::Transfo(
					-2 * y0, x0, w0, z1,
					x0, 0, z0, -w1,
					-w0, z0, -2 * y0, -x1,
					0, 0, 0, 0) * 2;
			case 3:
				return Tbx::Transfo(
					-2 * z0, -w0, x0, -y1,
					w0, -2 * z0, y0, x1,
					x0, y0, 0, -w1,
					0, 0, 0, 0) * 2;
			case 4:
				return Tbx::Transfo(
					0, 0, 0, -x0,
					0, 0, 0, -y0,
					0, 0, 0, -z0,
					0, 0, 0, 0) * 2;
			case 5:
				return Tbx::Transfo(
					0, 0, 0, w0,
					0, 0, 0, z0,
					0, 0, 0, -y0,
					0, 0, 0, 0) * 2;
			case 6:
				return Tbx::Transfo(
					0, 0, 0, -z0,
					0, 0, 0, w0,
					0, 0, 0, x0,
					0, 0, 0, 0) * 2;
			case 7:
				return Tbx::Transfo(
					0, 0, 0, y0,
					0, 0, 0, -x0,
					0, 0, 0, w0,
					0, 0, 0, 0) * 2;
			default:
				printf("index out of range");
				return Tbx::Transfo::identity();
			}
		}

		__device__ __forceinline__ Tbx::Transfo p_SE3_p_alpha_func(Tbx::Dual_quat_cu dq, int i)const
		{
			Tbx::Transfo T = Tbx::Transfo::empty();
			Tbx::Dual_quat_cu p_dq_p_alphai = p_qk_p_alpha_func(dq, i);
			for (int j = 0; j < 8; j++)
				T = T + p_SE3_p_dq_func(dq, j)*p_dq_p_alphai[j];
			return T;
		}

		__device__ __forceinline__ Tbx::Transfo compute_p_f_p_T(const Tbx::Vec3& n,
			const Tbx::Point3& v, const Tbx::Point3& vl, const Tbx::Dual_quat_cu& dq)const
		{
			//Tbx::Transfo T = Tlw*dq.to_transformation_after_normalize();
			//Tbx::Transfo nvt = outer_product(n, v);
			//Tbx::Transfo vlnt = outer_product(n, vl).transpose();
			//Tbx::Transfo p_f_p_T = T*(nvt + nvt.transpose()) - vlnt;
			Tbx::Vec3 Tn = Tlw*dq.rotate(n);
			Tbx::Point3 Tv(Tlw*dq.transform(v) - vl);
			return Tbx::Transfo(
				Tn.x*v.x + n.x*Tv.x, Tn.x*v.y + n.y*Tv.x, Tn.x*v.z + n.z*Tv.x, Tn.x,
				Tn.y*v.x + n.x*Tv.y, Tn.y*v.y + n.y*Tv.y, Tn.y*v.z + n.z*Tv.y, Tn.y,
				Tn.z*v.x + n.x*Tv.z, Tn.z*v.y + n.y*Tv.z, Tn.z*v.z + n.z*Tv.z, Tn.z,
				n.x, n.y, n.z, 0
				);
		}

		__device__ __forceinline__ bool search(int x, int y, Tbx::Point3& vl) const
		{
			float3 vwarp = read_float3_4(vmap_warp(y, x));
			float3 nwarp = read_float3_4(nmap_warp(y, x));

			if (isnan(nwarp.x))
				return false;

			float3 uvd = intr.xyz2uvd(vwarp);
			int2 ukr = make_int2(uvd.x + 0.5, uvd.y + 0.5);

			// we use opengl coordinate, thus world.z should < 0
			if (ukr.x < 0 || ukr.y < 0 || ukr.x >= imgWidth || ukr.y >= imgHeight || vwarp.z >= 0)
				return false;

			float3 vlive = read_float3_4(vmap_live[ukr.y*imgWidth + ukr.x]);
			float3 nlive = read_float3_4(nmap_live[ukr.y*imgWidth + ukr.x]);
			if (isnan(nlive.x))
				return false;

			float dist = norm(vwarp - vlive);
			if (!(dist <= distThres))
				return false;

			float sine = norm(cross(nwarp, nlive));
			if (!(sine <= angleThres))
				return false;

			vl = Tbx::Point3(vlive.x, vlive.y, vlive.z);

			return true;
		}

		__device__ __forceinline__ void operator () () const
		{
			const int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
			const int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

			Tbx::Point3 vl;
			bool found_coresp = false;
			if (x < imgWidth && y < imgHeight)
				found_coresp = search(x, y, vl);

			if (found_coresp)
			{
				Tbx::Point3 v(convert(read_float3_4(vmap_cano(y, x))));
				Tbx::Vec3 n(convert(read_float3_4(nmap_cano(y, x))));

				const KnnIdx knn = vmapKnn(y, x);
				Tbx::Dual_quat_cu dq(Tbx::Quat_cu(0, 0, 0, 0), Tbx::Quat_cu(0, 0, 0, 0));
				Tbx::Dual_quat_cu dqk[KnnK];
				float wk[KnnK];
				for (int k = 0; k < KnnK; k++)
				{
					int knnNodeId = knn_k(knn, k);
					if (knnNodeId < nNodes)
					{
						Tbx::Vec3 r, t;
						get_twist(knnNodeId, r, t);
						float4 nodeVw = get_nodesVw(knnNodeId);
						Tbx::Point3 nodesV(convert(read_float3_4(nodeVw)));
						float invNodesW = nodeVw.w;
						dqk[k].from_twist(r, t);
						// note: we store inv radius as vw.w, thus using * instead of / here
						wk[k] = __expf(-(v - nodesV).dot(v - nodesV)*(2 * invNodesW * invNodesW));
						if (dqk[0].get_non_dual_part().dot(dqk[k].get_non_dual_part()) < 0)
							wk[k] = -wk[k];
						dq = dq + dqk[k] * wk[k];
					}
				}

				Tbx::Dual_quat_cu dq_bar = dq;
				float inv_norm_dq_bar = 1.f / dq_bar.get_non_dual_part().norm();
				dq = dq * inv_norm_dq_bar; // normalize

				v = Tlw*dq.transform(v);
				n = Tlw*dq.rotate(n);

				// the grad energy f
				const float f = data_term_penalty(n.dot(v - vl));

				// paitial_f_partial_T
				const Tbx::Transfo p_f_p_T = compute_p_f_p_T(n, v, vl, dq);

				for (int knnK = 0; knnK < KnnK; knnK++)
				{
					float p_f_p_alpha[VarPerNode];
					int knnNodeId = knn_k(knn, knnK);
					float wk_k = wk[knnK] * inv_norm_dq_bar * 2;
					if (knnNodeId < nNodes)
					{
						// partial_T_partial_alphak
						for (int ialpha = 0; ialpha < 6; ialpha++)
						{
							Tbx::Transfo p_T_p_alphak = Tbx::Transfo::empty();
							Tbx::Dual_quat_cu p_qk_p_alpha = p_qk_p_alpha_func(dqk[knnK], ialpha);
							float pdot = dq_bar.get_non_dual_part().dot(p_qk_p_alpha.get_non_dual_part())
								* sqr(inv_norm_dq_bar);
	
							//// evaluate p_dqi_p_alphak, heavily hard code here
							//// this hard code is crucial to the performance 
							// 0:
							// (0, -z0, y0, x1,
							// z0, 0, -x0, y1,
							//-y0, x0, 0, z1,
							// 0, 0, 0, 0) * 2;
							float p_dqi_p_alphak = wk_k * (
								p_qk_p_alpha[0] - dq_bar[0] *  pdot
								);
							p_T_p_alphak[1] += -dq[3] * p_dqi_p_alphak;
							p_T_p_alphak[2] += dq[2] * p_dqi_p_alphak;
							p_T_p_alphak[3] += dq[5] * p_dqi_p_alphak;
							p_T_p_alphak[4] += dq[3] * p_dqi_p_alphak;
							p_T_p_alphak[6] += -dq[1] * p_dqi_p_alphak;
							p_T_p_alphak[7] += dq[6] * p_dqi_p_alphak;
							p_T_p_alphak[8] += -dq[2] * p_dqi_p_alphak;
							p_T_p_alphak[9] += dq[1] * p_dqi_p_alphak;
							p_T_p_alphak[11] += dq[7] * p_dqi_p_alphak;

							// 1
							//( 0, y0, z0, -w1,
							//	y0, -2 * x0, -w0, -z1,
							//	z0, w0, -2 * x0, y1,
							//	0, 0, 0, 0) * 2;
							p_dqi_p_alphak = wk_k * (
								p_qk_p_alpha[1] - dq_bar[1] * pdot
								);
							p_T_p_alphak[1] += dq[2] * p_dqi_p_alphak;
							p_T_p_alphak[2] += dq[3] * p_dqi_p_alphak;
							p_T_p_alphak[3] += -dq[4] * p_dqi_p_alphak;
							p_T_p_alphak[4] += dq[2] * p_dqi_p_alphak;
							p_T_p_alphak[5] += -dq[1] * p_dqi_p_alphak * 2;
							p_T_p_alphak[6] += -dq[0] * p_dqi_p_alphak;
							p_T_p_alphak[7] += -dq[7] * p_dqi_p_alphak;
							p_T_p_alphak[8] += dq[3] * p_dqi_p_alphak;
							p_T_p_alphak[9] += dq[0] * p_dqi_p_alphak;
							p_T_p_alphak[10] += -dq[1] * p_dqi_p_alphak * 2;
							p_T_p_alphak[11] += dq[6] * p_dqi_p_alphak;

							// 2.
							// (-2 * y0, x0, w0, z1,
							//	x0, 0, z0, -w1,
							//	-w0, z0, -2 * y0, -x1,
							//	0, 0, 0, 0) * 2;
							p_dqi_p_alphak = wk_k * (
								p_qk_p_alpha[2] - dq_bar[2] * pdot
								);
							p_T_p_alphak[0] += -dq[2] * p_dqi_p_alphak * 2;
							p_T_p_alphak[1] += dq[1] * p_dqi_p_alphak;
							p_T_p_alphak[2] += dq[0] * p_dqi_p_alphak;
							p_T_p_alphak[3] += dq[7] * p_dqi_p_alphak;
							p_T_p_alphak[4] += dq[1] * p_dqi_p_alphak;
							p_T_p_alphak[6] += dq[3] * p_dqi_p_alphak;
							p_T_p_alphak[7] += -dq[4] * p_dqi_p_alphak;
							p_T_p_alphak[8] += -dq[0] * p_dqi_p_alphak;
							p_T_p_alphak[9] += dq[3] * p_dqi_p_alphak;
							p_T_p_alphak[10] += -dq[2] * p_dqi_p_alphak * 2;
							p_T_p_alphak[11] += -dq[5] * p_dqi_p_alphak;

							// 3.
							// (-2 * z0, -w0, x0, -y1,
							//	w0, -2 * z0, y0, x1,
							//	x0, y0, 0, -w1,
							//	0, 0, 0, 0) * 2;
							p_dqi_p_alphak = wk_k * (
								p_qk_p_alpha[3] - dq_bar[3] * pdot
								);
							p_T_p_alphak[0] += -dq[3] * p_dqi_p_alphak * 2;
							p_T_p_alphak[1] += -dq[0] * p_dqi_p_alphak;
							p_T_p_alphak[2] += dq[1] * p_dqi_p_alphak;
							p_T_p_alphak[3] += -dq[6] * p_dqi_p_alphak;
							p_T_p_alphak[4] += dq[0] * p_dqi_p_alphak;
							p_T_p_alphak[5] += -dq[3] * p_dqi_p_alphak * 2;
							p_T_p_alphak[6] += dq[2] * p_dqi_p_alphak;
							p_T_p_alphak[7] += dq[5] * p_dqi_p_alphak;
							p_T_p_alphak[8] += dq[1] * p_dqi_p_alphak;
							p_T_p_alphak[9] += dq[2] * p_dqi_p_alphak;
							p_T_p_alphak[11] += -dq[4] * p_dqi_p_alphak;

							// 4.
							//( 0, 0, 0, -x0,
							//	0, 0, 0, -y0,
							//	0, 0, 0, -z0,
							//	0, 0, 0, 0) * 2;
							p_dqi_p_alphak = wk_k * (
								p_qk_p_alpha[4] - dq_bar[4] * pdot
								) * 2;
							p_T_p_alphak[3] += -dq[1] * p_dqi_p_alphak;
							p_T_p_alphak[7] += -dq[2] * p_dqi_p_alphak;
							p_T_p_alphak[11] += -dq[3] * p_dqi_p_alphak;

							// 5. 
							// (0, 0, 0, w0,
							//	0, 0, 0, z0,
							//	0, 0, 0, -y0,
							//	0, 0, 0, 0) * 2;
							p_dqi_p_alphak = wk_k * (
								p_qk_p_alpha[5] - dq_bar[5] * pdot
								);
							p_T_p_alphak[3] += dq[0] * p_dqi_p_alphak;
							p_T_p_alphak[7] += dq[3] * p_dqi_p_alphak;
							p_T_p_alphak[11] += -dq[2] * p_dqi_p_alphak;

							// 6. 
							// (0, 0, 0, -z0,
							//	0, 0, 0, w0,
							//	0, 0, 0, x0,
							//	0, 0, 0, 0) * 2;
							p_dqi_p_alphak = wk_k * (
								p_qk_p_alpha[6] - dq_bar[6] * pdot
								);
							p_T_p_alphak[3] += -dq[3] * p_dqi_p_alphak;
							p_T_p_alphak[7] += dq[0] * p_dqi_p_alphak;
							p_T_p_alphak[11] += dq[1] * p_dqi_p_alphak;

							// 7.
							// (0, 0, 0, y0,
							//	0, 0, 0, -x0,
							//	0, 0, 0, w0,
							//	0, 0, 0, 0) * 2;
							p_dqi_p_alphak = wk_k * (
								p_qk_p_alpha[7] - dq_bar[7] * pdot
								);
							p_T_p_alphak[3] += -dq[2] * p_dqi_p_alphak;
							p_T_p_alphak[7] += -dq[1] * p_dqi_p_alphak;
							p_T_p_alphak[11] += dq[0] * p_dqi_p_alphak;

							p_f_p_alpha[ialpha] = trace_AtB(p_f_p_T, Tlw * p_T_p_alphak);
						}// end for ialpha

						// reduce
						int shift = knnNodeId * VarPerNode2;
						int shift_g = knnNodeId * VarPerNode;
						for (int i = 0; i < VarPerNode; ++i)
						{
							#pragma unroll
							for (int j = 0; j <= i; ++j)
								atomicAdd(&Hd_[shift + j], p_f_p_alpha[i] * p_f_p_alpha[j]);
							atomicAdd(&g_[shift_g + i], p_f_p_alpha[i] * f);
							shift += VarPerNode;
						}// end for i
					}// end if knnNodeId < nNodes
				}// end for knnK
			}// end if found corr
		}// end function ()
	};

	__global__ void dataTermCombinedKernel(const DataTermCombined cs)
	{
		cs();
	}

	void GpuGaussNewtonSolver::calcDataTerm()
	{
		DataTermCombined cs;
		cs.angleThres = m_param->fusion_nonRigid_angleThreSin;
		cs.distThres = m_param->fusion_nonRigid_distThre;
		cs.Hd_ = m_Hd;
		cs.g_ = m_g;
		cs.imgHeight = m_vmap_cano->rows();
		cs.imgWidth = m_vmap_cano->cols();
		cs.intr = m_intr;
		cs.nmap_cano = *m_nmap_cano;
		cs.nmap_live = *m_nmap_live;
		cs.nmap_warp = *m_nmap_warp;
		cs.vmap_cano = *m_vmap_cano;
		cs.vmap_live = *m_vmap_live;
		cs.vmap_warp = *m_vmap_warp;
		cs.vmapKnn = m_vmapKnn;
		cs.nNodes = m_numNodes;
		cs.Tlw = m_pWarpField->get_rigidTransform();
		cs.psi_data = m_param->fusion_psi_data;
		cs.psi_reg = m_param->fusion_psi_reg;

		//////////////////////////////
		dim3 block(CTA_SIZE_X, CTA_SIZE_Y);
		dim3 grid(1, 1, 1);
		grid.x = divUp(cs.imgWidth, block.x);
		grid.y = divUp(cs.imgHeight, block.y);
		dataTermCombinedKernel << <grid, block >> >(cs);
		cudaSafeCall(hipGetLastError(), "dataTermCombinedKernel");
	}
#pragma endregion
}