#include "hip/hip_runtime.h"
#include "GpuGaussNewtonSolver.h"
#include "device_utils.h"
namespace dfusion
{
	texture<WarpField::KnnIdx, hipTextureType1D, hipReadModeElementType> g_nodesKnnTex;
	texture<float4, hipTextureType1D, hipReadModeElementType> g_nodesVwTex;
	texture<float, hipTextureType1D, hipReadModeElementType> g_twistTex;

	__device__ __forceinline__ float4 get_nodesVw(int i)
	{
		return tex1Dfetch(g_nodesVwTex, i);
	}

	__device__ __forceinline__ WarpField::KnnIdx get_nodesKnn(int i)
	{
		return tex1Dfetch(g_nodesKnnTex, i);
	}

	__device__ __forceinline__ void get_twist(int i, Tbx::Vec3& r, Tbx::Vec3& t)
	{
		int i6 = i * 6;
		r.x = tex1Dfetch(g_twistTex, i6++);
		r.y = tex1Dfetch(g_twistTex, i6++);
		r.z = tex1Dfetch(g_twistTex, i6++);
		t.x = tex1Dfetch(g_twistTex, i6++);
		t.y = tex1Dfetch(g_twistTex, i6++);
		t.z = tex1Dfetch(g_twistTex, i6++);
	}

	// map the lower part to full 6x6 matrix
	__constant__ int g_lower_2_full_6x6[21] = {
		0,
		6, 7,
		12, 13, 14,
		18, 19, 20, 21,
		24, 25, 26, 27, 28,
		30, 31, 32, 33, 34, 35
	};
	__constant__ int g_lfull_2_lower_6x6[6][6] = {
		{ 0, -1, -1, -1, -1, -1 },
		{ 1, 2, -1, -1, -1, -1 },
		{ 3, 4, 5, -1, -1, -1 },
		{ 6, 7, 8, 9, -1, -1 },
		{ 10, 11, 12, 13, 14, -1 },
		{ 15, 16, 17, 18, 19, 20 },
	};

#define D_1_DIV_6 0.166666667

	__device__ __forceinline__ float3 read_float3_4(float4 a)
	{
		return make_float3(a.x, a.y, a.z);
	}

	__device__ __forceinline__ float sqr(float a)
	{
		return a*a;
	}

	__device__ __forceinline__ float pow3(float a)
	{
		return a*a*a;
	}

	__device__ __forceinline__ float sign(float a)
	{
		return a>0 - a<0;
	}

	__device__ __forceinline__ WarpField::IdxType& knn_k(WarpField::KnnIdx& knn, int k)
	{
		return ((WarpField::IdxType*)(&knn))[k];
	}

#pragma region --bind textures
	void GpuGaussNewtonSolver::bindTextures()
	{
		if (1)
		{
			size_t offset;
			hipChannelFormatDesc desc = hipCreateChannelDesc<WarpField::KnnIdx>();
			hipBindTexture(&offset, &g_nodesKnnTex, m_nodesKnn.ptr(), &desc, 
				m_nodesKnn.size() * sizeof(WarpField::KnnIdx));
			if (offset != 0)
				throw std::exception("GpuGaussNewtonSolver::bindTextures(): non-zero-offset error!");
		}
		if (1)
		{
			size_t offset;
			hipChannelFormatDesc desc = hipCreateChannelDesc<float4>();
			hipBindTexture(&offset, &g_nodesVwTex, m_nodesVw.ptr(), &desc, 
				m_nodesVw.size() * sizeof(float4));
			if (offset != 0)
				throw std::exception("GpuGaussNewtonSolver::bindTextures(): non-zero-offset error!");
		}
		if (1)
		{
			size_t offset;
			hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
			hipBindTexture(&offset, &g_twistTex, m_twist.ptr(), &desc,
				m_twist.size() * sizeof(float));
			if (offset != 0)
				throw std::exception("GpuGaussNewtonSolver::bindTextures(): non-zero-offset error!");
		}
	}

	void GpuGaussNewtonSolver::unBindTextures()
	{
		hipUnbindTexture(g_twistTex);
		hipUnbindTexture(g_nodesVwTex);
		hipUnbindTexture(g_nodesKnnTex);
	}
#pragma endregion

#pragma region --calc data term
	struct DataTermCombined
	{
		typedef WarpField::KnnIdx KnnIdx;
		typedef WarpField::IdxType IdxType;
		enum
		{
			CTA_SIZE_X = GpuGaussNewtonSolver::CTA_SIZE_X,
			CTA_SIZE_Y = GpuGaussNewtonSolver::CTA_SIZE_Y,
			CTA_SIZE = CTA_SIZE_X * CTA_SIZE_Y,
			KnnK = WarpField::KnnK,
			VarPerNode = GpuGaussNewtonSolver::VarPerNode,
			VarPerNode2 = VarPerNode*VarPerNode,
			LowerPartNum = GpuGaussNewtonSolver::LowerPartNum,
		};

		PtrStep<float4> vmap_live;
		PtrStep<float4> nmap_live;
		PtrStep<float4> vmap_warp;
		PtrStep<float4> nmap_warp;
		PtrStep<float4> vmap_cano;
		PtrStep<float4> nmap_cano;
		PtrStep<KnnIdx> vmapKnn;
		float* Hd_;
		float* g_;

		Intr intr;
		Tbx::Transfo Tlw;

		int imgWidth;
		int imgHeight;
		int nNodes;

		float distThres;
		float angleThres;
		float psi_reg;
		float psi_data;

		__device__ __forceinline__ float data_term_energy(float f)const
		{
			return psi_data*psi_data * D_1_DIV_6 * (1 - pow3(max(0.f, 1.f - sqr(f / psi_data))));

			//// the robust Tukey penelty gradient
			//if (abs(f) <= psi_data)
			//	return psi_data*psi_data * D_1_DIV_6 * (1 - pow3(1 - sqr(f / psi_data)));
			//else
			//	return psi_data*psi_data * D_1_DIV_6;
		}

		__device__ __forceinline__ float data_term_penalty(float f)const
		{
			return f * sqr(max(0.f, 1.f - sqr(f / psi_data)));
			//// the robust Tukey penelty gradient
			//if (abs(f) <= psi_data)
			//	return f * sqr(1 - sqr(f / psi_data));
			//else
			//	return 0;
		}

		__device__ __forceinline__ Tbx::Transfo outer_product(Tbx::Vec3 n, Tbx::Point3 v)const
		{
			return Tbx::Transfo(
				n.x*v.x, n.x*v.y, n.x*v.z, n.x,
				n.y*v.x, n.y*v.y, n.y*v.z, n.y,
				n.z*v.x, n.z*v.y, n.z*v.z, n.z,
				0, 0, 0, 0
				);
		}

		__device__ __forceinline__ Tbx::Dual_quat_cu p_qk_p_alpha_func(Tbx::Dual_quat_cu dq, int i)const
		{
			Tbx::Vec3 t, r;
			float b, c, n;
			Tbx::Quat_cu q0(0, 0, 0, 0), q1 = dq.get_non_dual_part();
			switch (i)
			{
			case 0:
				dq.to_twist(r, t);
				n = r.norm();
				if (n > Tbx::Dual_quat_cu::epsilon())
				{
					b = sin(n) / n;
					c = (cos(n) - b) / (n*n);
					q0.coeff0 = -r.x * b;
					q0.coeff1 = b + r.x*r.x*c;
					q0.coeff2 = r.x*r.y*c;
					q0.coeff3 = r.x*r.z*c;
				}
				else
				{
					q0.coeff0 = 0;
					q0.coeff1 = 1;
					q0.coeff2 = 0;
					q0.coeff3 = 0;
				}

				q1.coeff0 = (t.x * q0.coeff1 + t.y * q0.coeff2 + t.z * q0.coeff3) * (-0.5);
				q1.coeff1 = (t.x * q0.coeff0 + t.y * q0.coeff3 - t.z * q0.coeff2) * 0.5;
				q1.coeff2 = (-t.x * q0.coeff3 + t.y * q0.coeff0 + t.z * q0.coeff1) * 0.5;
				q1.coeff3 = (t.x * q0.coeff2 - t.y * q0.coeff1 + t.z * q0.coeff0) * 0.5;
				return Tbx::Dual_quat_cu(q0, q1);
			case 1:
				dq.to_twist(r, t);
				n = r.norm();
				if (n > Tbx::Dual_quat_cu::epsilon())
				{
					b = sin(n) / n;
					c = (cos(n) - b) / (n*n);
					q0.coeff0 = -r.y * b;
					q0.coeff1 = r.y*r.x*c;
					q0.coeff2 = b + r.y*r.y*c;
					q0.coeff3 = r.y*r.z*c;
				}
				else
				{
					q0.coeff0 = 0;
					q0.coeff1 = 0;
					q0.coeff2 = 1;
					q0.coeff3 = 0;
				}

				q1.coeff0 = (t.x * q0.coeff1 + t.y * q0.coeff2 + t.z * q0.coeff3) * (-0.5);
				q1.coeff1 = (t.x * q0.coeff0 + t.y * q0.coeff3 - t.z * q0.coeff2) * 0.5;
				q1.coeff2 = (-t.x * q0.coeff3 + t.y * q0.coeff0 + t.z * q0.coeff1) * 0.5;
				q1.coeff3 = (t.x * q0.coeff2 - t.y * q0.coeff1 + t.z * q0.coeff0) * 0.5;
				return Tbx::Dual_quat_cu(q0, q1);
			case 2:
				dq.to_twist(r, t);
				n = r.norm();
				if (n > Tbx::Dual_quat_cu::epsilon())
				{
					b = sin(n) / n;
					c = (cos(n) - b) / (n*n);

					q0.coeff0 = -r.z * b;
					q0.coeff1 = r.z*r.x*c;
					q0.coeff2 = r.z*r.y*c;
					q0.coeff3 = b + r.z*r.z*c;
				}
				else
				{
					q0.coeff0 = 0;
					q0.coeff1 = 0;
					q0.coeff2 = 0;
					q0.coeff3 = 1;
				}

				q1.coeff0 = (t.x * q0.coeff1 + t.y * q0.coeff2 + t.z * q0.coeff3) * (-0.5);
				q1.coeff1 = (t.x * q0.coeff0 + t.y * q0.coeff3 - t.z * q0.coeff2) * 0.5;
				q1.coeff2 = (-t.x * q0.coeff3 + t.y * q0.coeff0 + t.z * q0.coeff1) * 0.5;
				q1.coeff3 = (t.x * q0.coeff2 - t.y * q0.coeff1 + t.z * q0.coeff0) * 0.5;
				return Tbx::Dual_quat_cu(q0, q1);
			case 3:
				return Tbx::Dual_quat_cu(q0, Tbx::Quat_cu(-q1.coeff1, q1.coeff0, -q1.coeff3, q1.coeff2))*0.5;
			case 4:
				return Tbx::Dual_quat_cu(q0, Tbx::Quat_cu(-q1.coeff2, q1.coeff3, q1.coeff0, -q1.coeff1))*0.5;
			case 5:
				return Tbx::Dual_quat_cu(q0, Tbx::Quat_cu(-q1.coeff3, -q1.coeff2, q1.coeff1, q1.coeff0))*0.5;
			default:
				printf("p_qk_p_alpha_func: out of range");
				return Tbx::Dual_quat_cu::identity();
			}
		}

		__device__ __forceinline__ float trace_AtB(Tbx::Transfo A, Tbx::Transfo B)const
		{
			float sum = 0;
			for (int i = 0; i < 16; i++)
				sum += A[i] * B[i];
			return sum;
		}

		__device__ __forceinline__ Tbx::Transfo p_SE3_p_dq_func(Tbx::Dual_quat_cu dq, int i)const
		{
			Tbx::Quat_cu q0 = dq.get_non_dual_part();
			Tbx::Quat_cu q1 = dq.get_dual_part();
			float x0 = q0.i(), y0 = q0.j(), z0 = q0.k(), w0 = q0.w();
			float x1 = q1.i(), y1 = q1.j(), z1 = q1.k(), w1 = q1.w();
			switch (i)
			{
			case 0:
				return Tbx::Transfo(
					0, -z0, y0, x1,
					z0, 0, -x0, y1,
					-y0, x0, 0, z1,
					0, 0, 0, 0) * 2;
			case 1:
				return Tbx::Transfo(
					0, y0, z0, -w1,
					y0, -2 * x0, -w0, -z1,
					z0, w0, -2 * x0, y1,
					0, 0, 0, 0) * 2;
			case 2:
				return Tbx::Transfo(
					-2 * y0, x0, w0, z1,
					x0, 0, z0, -w1,
					-w0, z0, -2 * y0, -x1,
					0, 0, 0, 0) * 2;
			case 3:
				return Tbx::Transfo(
					-2 * z0, -w0, x0, -y1,
					w0, -2 * z0, y0, x1,
					x0, y0, 0, -w1,
					0, 0, 0, 0) * 2;
			case 4:
				return Tbx::Transfo(
					0, 0, 0, -x0,
					0, 0, 0, -y0,
					0, 0, 0, -z0,
					0, 0, 0, 0) * 2;
			case 5:
				return Tbx::Transfo(
					0, 0, 0, w0,
					0, 0, 0, z0,
					0, 0, 0, -y0,
					0, 0, 0, 0) * 2;
			case 6:
				return Tbx::Transfo(
					0, 0, 0, -z0,
					0, 0, 0, w0,
					0, 0, 0, x0,
					0, 0, 0, 0) * 2;
			case 7:
				return Tbx::Transfo(
					0, 0, 0, y0,
					0, 0, 0, -x0,
					0, 0, 0, w0,
					0, 0, 0, 0) * 2;
			default:
				printf("index out of range");
				return Tbx::Transfo::identity();
			}
		}

		__device__ __forceinline__ Tbx::Transfo p_SE3_p_alpha_func(Tbx::Dual_quat_cu dq, int i)const
		{
			Tbx::Transfo T = Tbx::Transfo::empty();
			Tbx::Dual_quat_cu p_dq_p_alphai = p_qk_p_alpha_func(dq, i);
			for (int j = 0; j < 8; j++)
				T = T + p_SE3_p_dq_func(dq, j)*p_dq_p_alphai[j];
			return T;
		}

		__device__ __forceinline__ Tbx::Transfo compute_p_f_p_T(const Tbx::Vec3& n,
			const Tbx::Point3& v, const Tbx::Point3& vl, const Tbx::Dual_quat_cu& dq)const
		{
			//Tbx::Transfo T = Tlw*dq.to_transformation_after_normalize();
			//Tbx::Transfo nvt = outer_product(n, v);
			//Tbx::Transfo vlnt = outer_product(n, vl).transpose();
			//Tbx::Transfo p_f_p_T = T*(nvt + nvt.transpose()) - vlnt;
			Tbx::Vec3 Tn = Tlw*dq.rotate(n);
			Tbx::Point3 Tv(Tlw*dq.transform(v) - vl);
			return Tbx::Transfo(
				Tn.x*v.x + n.x*Tv.x, Tn.x*v.y + n.y*Tv.x, Tn.x*v.z + n.z*Tv.x, Tn.x,
				Tn.y*v.x + n.x*Tv.y, Tn.y*v.y + n.y*Tv.y, Tn.y*v.z + n.z*Tv.y, Tn.y,
				Tn.z*v.x + n.x*Tv.z, Tn.z*v.y + n.y*Tv.z, Tn.z*v.z + n.z*Tv.z, Tn.z,
				n.x, n.y, n.z, 0
				);
		}

		__device__ __forceinline__ bool search(int x, int y, Tbx::Point3& vl) const
		{
			float3 vwarp = read_float3_4(vmap_warp(y, x));
			float3 nwarp = read_float3_4(nmap_warp(y, x));

			if (isnan(nwarp.x))
				return false;

			float3 uvd = intr.xyz2uvd(vwarp);
			int2 ukr = make_int2(uvd.x + 0.5, uvd.y + 0.5);

			// we use opengl coordinate, thus world.z should < 0
			if (ukr.x < 0 || ukr.y < 0 || ukr.x >= imgWidth || ukr.y >= imgHeight || vwarp.z >= 0)
				return false;

			float3 vlive = read_float3_4(vmap_live[ukr.y*imgWidth + ukr.x]);
			float3 nlive = read_float3_4(nmap_live[ukr.y*imgWidth + ukr.x]);
			if (isnan(nlive.x))
				return false;

			float dist = norm(vwarp - vlive);
			if (!(dist <= distThres))
				return false;

			float sine = norm(cross(nwarp, nlive));
			if (!(sine <= angleThres))
				return false;

			vl = Tbx::Point3(vlive.x, vlive.y, vlive.z);

			return true;
		}

		__device__ __forceinline__ void operator () () const
		{
			int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
			int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

			Tbx::Point3 vl;
			bool found_coresp = false;
			if (x < imgWidth && y < imgHeight)
				found_coresp = search(x, y, vl);

			KnnIdx knn = make_ushort4(0,0,0,0);
			if (found_coresp)
			{
				Tbx::Point3 v(convert(read_float3_4(vmap_cano(y, x))));
				Tbx::Vec3 n(convert(read_float3_4(nmap_cano(y, x))));

				knn = vmapKnn(y, x);
				Tbx::Dual_quat_cu dq(Tbx::Quat_cu(0, 0, 0, 0), Tbx::Quat_cu(0, 0, 0, 0));
				Tbx::Dual_quat_cu dqk[KnnK];
				float wk[KnnK];
				for (int k = 0; k < KnnK; k++)
				{
					int knnNodeId = knn_k(knn, k);
					if (knnNodeId < nNodes)
					{
						Tbx::Vec3 r, t;
						get_twist(knnNodeId, r, t);
						float4 nodeVw = get_nodesVw(knnNodeId);
						Tbx::Point3 nodesV(convert(read_float3_4(nodeVw)));
						float invNodesW = nodeVw.w;
						dqk[k].from_twist(r, t);
						// note: we store inv radius as vw.w, thus using * instead of / here
						wk[k] = __expf(-(v - nodesV).dot(v - nodesV)*(2 * invNodesW * invNodesW));
						if (dqk[0].get_non_dual_part().dot(dqk[k].get_non_dual_part()) < 0)
							wk[k] = -wk[k];
						dq = dq + dqk[k] * wk[k];
					}
				}

				Tbx::Dual_quat_cu dq_bar = dq;
				float inv_norm_dq_bar = 1.f / dq_bar.get_non_dual_part().norm();
				float inv_norm_dq_bar3 = inv_norm_dq_bar*inv_norm_dq_bar*inv_norm_dq_bar;
				dq = dq * inv_norm_dq_bar; // normalize

				v = Tlw*dq.transform(v);
				n = Tlw*dq.rotate(n);

				// the grad energy f
				float f = data_term_penalty(n.dot(v - vl));

				// paitial_f_partial_T
				Tbx::Transfo p_f_p_T = compute_p_f_p_T(n, v, vl, dq);

				for (int knnK = 0; knnK < KnnK; knnK++)
				{
					float p_f_p_alpha[VarPerNode];
					int knnNodeId = knn_k(knn, knnK);
					if (knnNodeId < nNodes)
					{
						// partial_T_partial_alphak
						for (int ialpha = 0; ialpha < 6; ialpha++)
						{
							Tbx::Transfo p_T_p_alphak = Tbx::Transfo::empty();
							Tbx::Dual_quat_cu p_qk_p_alpha = p_qk_p_alpha_func(dqk[knnK], ialpha);
							for (int idq = 0; idq < 8; idq++)
							{
								// partial_SE3_partial_dqi
								Tbx::Transfo p_SE3_p_dqi = p_SE3_p_dq_func(dq, idq);
								float dq_bar_i = dq_bar[idq];

								// partial_dqi_partial_alphak
								float p_dqi_p_alphak = 0;
								for (int j = 0; j < 8; j++)
								{
									// partial_dqi_partial_qkj
									float dq_bar_j = dq_bar[j];
									float p_dqi_p_qkj = wk[knnK] * inv_norm_dq_bar * (idq == j);
									if (j < 4)
										p_dqi_p_qkj -= wk[knnK] * inv_norm_dq_bar3*dq_bar_i*dq_bar_j;

									// partial_qkj_partial_alphak
									float p_qkj_p_alphak = p_qk_p_alpha[j];

									p_dqi_p_alphak += p_dqi_p_qkj * p_qkj_p_alphak;
								}// end for j

								p_T_p_alphak += p_SE3_p_dqi * p_dqi_p_alphak;
							}// end for idq
							p_T_p_alphak = Tlw * p_T_p_alphak;

							p_f_p_alpha[ialpha] = trace_AtB(p_f_p_T, p_T_p_alphak);
						}// end for ialpha

						// reduce
						int shift = knnNodeId * VarPerNode2;
						int shift_g = knnNodeId * VarPerNode;
						for (int i = 0; i < VarPerNode; ++i)
						{
							#pragma unroll
							for (int j = 0; j <= i; ++j)
								atomicAdd(&Hd_[shift + j], p_f_p_alpha[i] * p_f_p_alpha[j]);
							atomicAdd(&g_[shift_g + i], p_f_p_alpha[i] * f);
							shift += VarPerNode;
						}// end for i
					}// end if knnNodeId < nNodes
				}// end for knnK
			}// end if found corr
		}// end function ()
	};

	__global__ void dataTermCombinedKernel(const DataTermCombined cs)
	{
		cs();
	}

	void GpuGaussNewtonSolver::calcDataTerm()
	{
		DataTermCombined cs;
		cs.angleThres = m_param->fusion_nonRigid_angleThreSin;
		cs.distThres = m_param->fusion_nonRigid_distThre;
		cs.Hd_ = m_Hd;
		cs.g_ = m_g;
		cs.imgHeight = m_vmap_cano->rows();
		cs.imgWidth = m_vmap_cano->cols();
		cs.intr = m_intr;
		cs.nmap_cano = *m_nmap_cano;
		cs.nmap_live = *m_nmap_live;
		cs.nmap_warp = *m_nmap_warp;
		cs.vmap_cano = *m_vmap_cano;
		cs.vmap_live = *m_vmap_live;
		cs.vmap_warp = *m_vmap_warp;
		cs.vmapKnn = m_vmapKnn;
		cs.nNodes = m_numNodes;
		cs.Tlw = m_pWarpField->get_rigidTransform();
		cs.psi_data = m_param->fusion_psi_data;
		cs.psi_reg = m_param->fusion_psi_reg;

		//////////////////////////////
		dim3 block(CTA_SIZE_X, CTA_SIZE_Y);
		dim3 grid(1, 1, 1);
		grid.x = divUp(cs.imgWidth, block.x);
		grid.y = divUp(cs.imgHeight, block.y);
		dataTermCombinedKernel << <grid, block >> >(cs);
		cudaSafeCall(hipGetLastError(), "dataTermCombinedKernel");
	}
#pragma endregion
}