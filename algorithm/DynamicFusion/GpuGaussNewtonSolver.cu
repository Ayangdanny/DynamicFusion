#include "hip/hip_runtime.h"
#include "GpuGaussNewtonSolver.h"
#include "device_utils.h"
#include "cudpp\thrust_wrapper.h"
#include "cudpp\ModerGpuWrapper.h"
#include <iostream>
#include "GpuCholeSky.h"
namespace dfusion
{
//#define DEFINE_USE_HALF_GRAPH_EDGE
//#define CALC_DATA_TERM_NUMERIC
//#define CALC_REG_TERM_NUMERIC
//#define DEBUG_ASSIGN_10M_TO_NO_CORR
//#define DEBUG_ASSIGN_BIG_ENERGY_TO_NO_CORR

#ifdef DEFINE_USE_HALF_GRAPH_EDGE
	enum{RowPerNode_RegTerm = 3};
#else
	enum{ RowPerNode_RegTerm = 6 };
#endif
//#define USE_L2_NORM_DATA_TERM
//#define USE_L2_NORM_REG_TERM
#define CHECK(a, msg){if(!(a)) throw std::exception(msg);} 
#define CHECK_LE(a, b){if((a) > (b)) {std::cout << "" << #a << "(" << a << ")<=" << #b << "(" << b << ")";throw std::exception(" ###error!");}} 

	texture<WarpField::KnnIdx, hipTextureType1D, hipReadModeElementType> g_nodesKnnTex;
	texture<float4, hipTextureType1D, hipReadModeElementType> g_nodesVwTex;
	texture<float, hipTextureType1D, hipReadModeElementType> g_twistTex;
	texture<float, hipTextureType1D, hipReadModeElementType> g_JrtValTex;
	texture<int, hipTextureType1D, hipReadModeElementType> g_JrtCidxTex;
	texture<float, hipTextureType1D, hipReadModeElementType> g_BtValTex;
	texture<int, hipTextureType1D, hipReadModeElementType> g_BtCidxTex;
	texture<float, hipTextureType1D, hipReadModeElementType> g_BtLtinvValTex;
	texture<float, hipTextureType1D, hipReadModeElementType> g_HdLinvTex;

	__device__ __forceinline__ float4 get_nodesVw(int i)
	{
		return tex1Dfetch(g_nodesVwTex, i);
	}

	__device__ __forceinline__ WarpField::KnnIdx get_nodesKnn(int i)
	{
		return tex1Dfetch(g_nodesKnnTex, i);
	}

	__device__ __forceinline__ void get_twist(int i, Tbx::Vec3& r, Tbx::Vec3& t)
	{
		int i6 = i * 6;
		r.x = tex1Dfetch(g_twistTex, i6++);
		r.y = tex1Dfetch(g_twistTex, i6++);
		r.z = tex1Dfetch(g_twistTex, i6++);
		t.x = tex1Dfetch(g_twistTex, i6++);
		t.y = tex1Dfetch(g_twistTex, i6++);
		t.z = tex1Dfetch(g_twistTex, i6++);
	}

	__device__ __forceinline__ float get_JrtVal(int i)
	{
		return tex1Dfetch(g_JrtValTex, i);
	}
	__device__ __forceinline__ int get_JrtCidx(int i)
	{
		return tex1Dfetch(g_JrtCidxTex, i);
	}

	__device__ __forceinline__ float get_BtVal(int i)
	{
		return tex1Dfetch(g_BtValTex, i);
	}
	__device__ __forceinline__ int get_BtCidx(int i)
	{
		return tex1Dfetch(g_BtCidxTex, i);
	}

	__device__ __forceinline__ float get_HdLinv(int i)
	{
		return tex1Dfetch(g_HdLinvTex, i);
	}

	__device__ __forceinline__ float get_BtLtinvVal(int i)
	{
		return tex1Dfetch(g_BtLtinvValTex, i);
	}
	__device__ __forceinline__ int get_BtLtinvCidx(int i)
	{
		return tex1Dfetch(g_BtCidxTex, i);
	}

	// map the lower part to full 6x6 matrix
	__constant__ int g_lower_2_full_6x6[21] = {
		0,
		6, 7,
		12, 13, 14,
		18, 19, 20, 21,
		24, 25, 26, 27, 28,
		30, 31, 32, 33, 34, 35
	};
	__constant__ int g_lower_2_rowShift_6x6[21] = {
		0,
		1, 1,
		2, 2, 2,
		3, 3, 3, 3,
		4, 4, 4, 4, 4,
		5, 5, 5, 5, 5, 5
	};
	__constant__ int g_lower_2_colShift_6x6[21] = {
		0,
		0, 1,
		0, 1, 2,
		0, 1, 2, 3,
		0, 1, 2, 3, 4,
		0, 1, 2, 3, 4, 5
	};
	__constant__ int g_lfull_2_lower_6x6[6][6] = {
		{ 0, -1, -1, -1, -1, -1 },
		{ 1, 2, -1, -1, -1, -1 },
		{ 3, 4, 5, -1, -1, -1 },
		{ 6, 7, 8, 9, -1, -1 },
		{ 10, 11, 12, 13, 14, -1 },
		{ 15, 16, 17, 18, 19, 20 },
	};

#define D_1_DIV_6 0.166666667

	__device__ __forceinline__ float3 read_float3_4(float4 a)
	{
		return make_float3(a.x, a.y, a.z);
	}

	__device__ __forceinline__ float sqr(float a)
	{
		return a*a;
	}

	__device__ __forceinline__ float pow3(float a)
	{
		return a*a*a;
	}

	__device__ __forceinline__ float sign(float a)
	{
		return (a>0.f) - (a<0.f);
	}

	__device__ __forceinline__ WarpField::IdxType& knn_k(WarpField::KnnIdx& knn, int k)
	{
		return ((WarpField::IdxType*)(&knn))[k];
	}
	__device__ __forceinline__ const WarpField::IdxType& knn_k(const WarpField::KnnIdx& knn, int k)
	{
		return ((WarpField::IdxType*)(&knn))[k];
	}

	__device__ __forceinline__ void sort_knn(WarpField::KnnIdx& knn)
	{
		for (int i = 1; i < WarpField::KnnK; i++)
		{
			WarpField::IdxType x = knn_k(knn,i);
			int	j = i;
			while (j > 0 && knn_k(knn, j - 1) > x)
			{
				knn_k(knn, j) = knn_k(knn, j - 1);
				j = j - 1;
			}
			knn_k(knn, j) = x;
		}
	}

#pragma region --bind textures
	void GpuGaussNewtonSolver::bindTextures()
	{
		if (1)
		{
			size_t offset;
			hipChannelFormatDesc desc = hipCreateChannelDesc<WarpField::KnnIdx>();
			hipBindTexture(&offset, &g_nodesKnnTex, m_nodesKnn.ptr(), &desc,
				m_nodesKnn.size() * sizeof(WarpField::KnnIdx));
			if (offset != 0)
				throw std::exception("GpuGaussNewtonSolver::bindTextures(): non-zero-offset error1!");
		}
		if (1)
		{
			size_t offset;
			hipChannelFormatDesc desc = hipCreateChannelDesc<float4>();
			hipBindTexture(&offset, &g_nodesVwTex, m_nodesVw.ptr(), &desc,
				m_nodesVw.size() * sizeof(float4));
			if (offset != 0)
				throw std::exception("GpuGaussNewtonSolver::bindTextures(): non-zero-offset error2!");
		}
		if (1)
		{
			size_t offset;
			hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
			hipBindTexture(&offset, &g_twistTex, m_twist.ptr(), &desc,
				m_twist.size() * sizeof(float));
			if (offset != 0)
				throw std::exception("GpuGaussNewtonSolver::bindTextures(): non-zero-offset error3!");
		}
		if (1)
		{
			size_t offset;
			hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
			hipBindTexture(&offset, &g_JrtValTex, m_Jrt_val.ptr(), &desc,
				m_Jrt_val.size() * sizeof(float));
			if (offset != 0)
				throw std::exception("GpuGaussNewtonSolver::bindTextures(): non-zero-offset error4!");
		}
		if (1)
		{
			size_t offset;
			hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
			hipBindTexture(&offset, &g_JrtCidxTex, m_Jrt_ColIdx.ptr(), &desc,
				m_Jrt_ColIdx.size() * sizeof(int));
			if (offset != 0)
				throw std::exception("GpuGaussNewtonSolver::bindTextures(): non-zero-offset error5!");
		}
		if (1)
		{
			size_t offset;
			hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
			hipBindTexture(&offset, &g_BtLtinvValTex, m_Bt_Ltinv_val.ptr(), &desc,
				m_Bt_Ltinv_val.size() * sizeof(float));
			if (offset != 0)
				throw std::exception("GpuGaussNewtonSolver::bindTextures(): non-zero-offset error6!");
		}
		if (1)
		{
			size_t offset;
			hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
			hipBindTexture(&offset, &g_BtCidxTex, m_Bt_ColIdx.ptr(), &desc,
				m_Bt_ColIdx.size() * sizeof(int));
			if (offset != 0)
				throw std::exception("GpuGaussNewtonSolver::bindTextures(): non-zero-offset error7!");
		}
		if (1)
		{
			size_t offset;
			hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
			hipBindTexture(&offset, &g_BtValTex, m_Bt_val.ptr(), &desc,
				m_Bt_val.size() * sizeof(float));
			if (offset != 0)
				throw std::exception("GpuGaussNewtonSolver::bindTextures(): non-zero-offset error8!");
		}
		if (1)
		{
			size_t offset;
			hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
			hipBindTexture(&offset, &g_HdLinvTex, m_Hd_Linv.ptr(), &desc,
				m_Hd_Linv.size() * sizeof(float));
			if (offset != 0)
				throw std::exception("GpuGaussNewtonSolver::bindTextures(): non-zero-offset error9!");
		}
	}

	void GpuGaussNewtonSolver::unBindTextures()
	{
		hipUnbindTexture(g_twistTex);
		hipUnbindTexture(g_nodesVwTex);
		hipUnbindTexture(g_nodesKnnTex);
		hipUnbindTexture(g_JrtValTex);
		hipUnbindTexture(g_JrtCidxTex);
		hipUnbindTexture(g_BtValTex);
		hipUnbindTexture(g_BtCidxTex);
		hipUnbindTexture(g_BtLtinvValTex);
		hipUnbindTexture(g_HdLinvTex);
	}
#pragma endregion

#pragma region --calc data term

//#define ENABLE_GPU_DUMP_DEBUG

	//__device__ float g_totalEnergy;

	struct DataTermCombined
	{
		typedef WarpField::KnnIdx KnnIdx;
		typedef WarpField::IdxType IdxType;
		enum
		{
			CTA_SIZE_X = GpuGaussNewtonSolver::CTA_SIZE_X,
			CTA_SIZE_Y = GpuGaussNewtonSolver::CTA_SIZE_Y,
			CTA_SIZE = CTA_SIZE_X * CTA_SIZE_Y,
			KnnK = WarpField::KnnK,
			VarPerNode = GpuGaussNewtonSolver::VarPerNode,
			VarPerNode2 = VarPerNode*VarPerNode,
			LowerPartNum = GpuGaussNewtonSolver::LowerPartNum,
		};

		PtrStep<float4> vmap_live;
		PtrStep<float4> nmap_live;
		PtrStep<float4> vmap_warp;
		PtrStep<float4> nmap_warp;
		PtrStep<float4> vmap_cano;
		PtrStep<float4> nmap_cano;
		PtrStep<KnnIdx> vmapKnn;
		float* Hd_;
		float* g_;

		Intr intr;
		Tbx::Transfo Tlw_inv;
		Tbx::Transfo Tlw;

		int imgWidth;
		int imgHeight;
		int nNodes;

		float distThres;
		float angleThres;
		float psi_data;

		float* totalEnergy;


#ifdef ENABLE_GPU_DUMP_DEBUG
		// for debug
		float* debug_buffer_pixel_sum2;
		float* debug_buffer_pixel_val;
#endif

		__device__ __forceinline__ float data_term_energy(float f)const
		{
#ifdef USE_L2_NORM_DATA_TERM
			return 0.5f*f*f;
#else
			// the robust Tukey penelty gradient
			if (abs(f) <= psi_data)
				return psi_data*psi_data / 6.f *(1 - pow(1 - sqr(f / psi_data), 3));
			else
				return psi_data*psi_data / 6.f;
#endif
		}

		__device__ __forceinline__ float data_term_penalty(float f)const
		{
#ifdef USE_L2_NORM_DATA_TERM
			return f;
#else
			return f * sqr(max(0.f, 1.f - sqr(f / psi_data)));
			//// the robust Tukey penelty gradient
			//if (abs(f) <= psi_data)
			//	return f * sqr(1 - sqr(f / psi_data));
			//else
			//	return 0;
#endif
		}

		__device__ __forceinline__ float trace_AtB(Tbx::Transfo A, Tbx::Transfo B)const
		{
			float sum = 0;
			for (int i = 0; i < 16; i++)
				sum += A[i] * B[i];
			return sum;
		}

		__device__ __forceinline__ Tbx::Transfo compute_p_f_p_T(const Tbx::Vec3& n,
			const Tbx::Point3& v, const Tbx::Point3& vl, const Tbx::Dual_quat_cu& dq)const
		{
			//Tbx::Transfo T = Tlw*dq.to_transformation_after_normalize();
			//Tbx::Transfo nvt = outer_product(n, v);
			//Tbx::Transfo vlnt = outer_product(n, vl).transpose();
			//Tbx::Transfo p_f_p_T = T*(nvt + nvt.transpose()) - vlnt;
			Tbx::Vec3 Tn = dq.rotate(n);
			Tbx::Point3 Tv(dq.transform(v) - vl);
			return Tbx::Transfo(
				Tn.x*v.x + n.x*Tv.x, Tn.x*v.y + n.y*Tv.x, Tn.x*v.z + n.z*Tv.x, Tn.x,
				Tn.y*v.x + n.x*Tv.y, Tn.y*v.y + n.y*Tv.y, Tn.y*v.z + n.z*Tv.y, Tn.y,
				Tn.z*v.x + n.x*Tv.z, Tn.z*v.y + n.y*Tv.z, Tn.z*v.z + n.z*Tv.z, Tn.z,
				n.x, n.y, n.z, 0
				);
		}

		__device__ __forceinline__ Tbx::Transfo p_T_p_alphak_func(const Tbx::Dual_quat_cu& p_qk_p_alpha,
			const Tbx::Dual_quat_cu& dq_bar, const Tbx::Dual_quat_cu& dq, float inv_norm_dq_bar, float wk_k)const
		{
			Tbx::Transfo p_T_p_alphak = Tbx::Transfo::empty();

			float pdot = dq_bar.get_non_dual_part().dot(p_qk_p_alpha.get_non_dual_part())
				* sqr(inv_norm_dq_bar);

			//// evaluate p_dqi_p_alphak, heavily hard code here
			//// this hard code is crucial to the performance 
			// 0:
			// (0, -z0, y0, x1,
			// z0, 0, -x0, y1,
			//-y0, x0, 0, z1,
			// 0, 0, 0, 0) * 2;
			float p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[0] - dq_bar[0] * pdot
				);
			p_T_p_alphak[1] += -dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[2] += dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[3] += dq[5] * p_dqi_p_alphak;
			p_T_p_alphak[4] += dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[6] += -dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[7] += dq[6] * p_dqi_p_alphak;
			p_T_p_alphak[8] += -dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[9] += dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[11] += dq[7] * p_dqi_p_alphak;

			// 1
			//( 0, y0, z0, -w1,
			//	y0, -2 * x0, -w0, -z1,
			//	z0, w0, -2 * x0, y1,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[1] - dq_bar[1] * pdot
				);
			p_T_p_alphak[1] += dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[2] += dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[3] += -dq[4] * p_dqi_p_alphak;
			p_T_p_alphak[4] += dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[5] += -dq[1] * p_dqi_p_alphak * 2;
			p_T_p_alphak[6] += -dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[7] += -dq[7] * p_dqi_p_alphak;
			p_T_p_alphak[8] += dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[9] += dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[10] += -dq[1] * p_dqi_p_alphak * 2;
			p_T_p_alphak[11] += dq[6] * p_dqi_p_alphak;

			// 2.
			// (-2 * y0, x0, w0, z1,
			//	x0, 0, z0, -w1,
			//	-w0, z0, -2 * y0, -x1,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[2] - dq_bar[2] * pdot
				);
			p_T_p_alphak[0] += -dq[2] * p_dqi_p_alphak * 2;
			p_T_p_alphak[1] += dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[2] += dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[3] += dq[7] * p_dqi_p_alphak;
			p_T_p_alphak[4] += dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[6] += dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[7] += -dq[4] * p_dqi_p_alphak;
			p_T_p_alphak[8] += -dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[9] += dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[10] += -dq[2] * p_dqi_p_alphak * 2;
			p_T_p_alphak[11] += -dq[5] * p_dqi_p_alphak;

			// 3.
			// (-2 * z0, -w0, x0, -y1,
			//	w0, -2 * z0, y0, x1,
			//	x0, y0, 0, -w1,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[3] - dq_bar[3] * pdot
				);
			p_T_p_alphak[0] += -dq[3] * p_dqi_p_alphak * 2;
			p_T_p_alphak[1] += -dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[2] += dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[3] += -dq[6] * p_dqi_p_alphak;
			p_T_p_alphak[4] += dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[5] += -dq[3] * p_dqi_p_alphak * 2;
			p_T_p_alphak[6] += dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[7] += dq[5] * p_dqi_p_alphak;
			p_T_p_alphak[8] += dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[9] += dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[11] += -dq[4] * p_dqi_p_alphak;

			// 4.
			//( 0, 0, 0, -x0,
			//	0, 0, 0, -y0,
			//	0, 0, 0, -z0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[4] - dq_bar[4] * pdot
				);
			p_T_p_alphak[3] += -dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[7] += -dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[11] += -dq[3] * p_dqi_p_alphak;

			// 5. 
			// (0, 0, 0, w0,
			//	0, 0, 0, z0,
			//	0, 0, 0, -y0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[5] - dq_bar[5] * pdot
				);
			p_T_p_alphak[3] += dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[7] += dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[11] += -dq[2] * p_dqi_p_alphak;

			// 6. 
			// (0, 0, 0, -z0,
			//	0, 0, 0, w0,
			//	0, 0, 0, x0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[6] - dq_bar[6] * pdot
				);
			p_T_p_alphak[3] += -dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[7] += dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[11] += dq[1] * p_dqi_p_alphak;

			// 7.
			// (0, 0, 0, y0,
			//	0, 0, 0, -x0,
			//	0, 0, 0, w0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[7] - dq_bar[7] * pdot
				);
			p_T_p_alphak[3] += dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[7] += -dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[11] += dq[0] * p_dqi_p_alphak;

			return p_T_p_alphak;
		}

		__device__ __forceinline__ bool search(int x, int y, Tbx::Point3& vl) const
		{
			float3 vwarp = read_float3_4(vmap_warp(y, x));
			float3 nwarp = read_float3_4(nmap_warp(y, x));

			return search(vwarp, nwarp, vl);
		}

		__device__ __forceinline__ bool search(float3 vwarp, float3 nwarp, Tbx::Point3& vl) const
		{
			if (isnan(nwarp.x) || isnan(vwarp.x))
				return false;

			float3 uvd = intr.xyz2uvd(vwarp);
			int2 ukr = make_int2(__float2int_rn(uvd.x), __float2int_rn(uvd.y));

			// we use opengl coordinate, thus world.z should < 0
			if (ukr.x < 0 || ukr.y < 0 || ukr.x >= imgWidth || ukr.y >= imgHeight || vwarp.z >= 0)
				return false;

			float3 vlive = read_float3_4(vmap_live[ukr.y*imgWidth + ukr.x]);
			float3 nlive = read_float3_4(nmap_live[ukr.y*imgWidth + ukr.x]);
			if (isnan(nlive.x) || isnan(vlive.x))
				return false;

#ifndef DEBUG_ASSIGN_10M_TO_NO_CORR
			float dist = norm(vwarp - vlive);
			if (!(dist <= distThres))
				return false;

			float sine = norm(cross(nwarp, nlive));
			if (!(sine < angleThres))
				return false;
#endif

			vl = Tbx::Point3(vlive.x, vlive.y, vlive.z);

			return true;
		}

		template<int maxK>
		__device__ __forceinline__ void calc_dataterm () const
		{
			const int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
			const int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

			Tbx::Point3 vl;
			bool found_coresp = false;
			if (x < imgWidth && y < imgHeight)
				found_coresp = search(x, y, vl);

			vl = Tlw_inv * vl;

			if (found_coresp)
			{
				Tbx::Point3 v(convert(read_float3_4(vmap_cano(y, x))));
				Tbx::Vec3 n(convert(read_float3_4(nmap_cano(y, x))));

				const KnnIdx knn = vmapKnn(y, x);
				Tbx::Dual_quat_cu dq(Tbx::Quat_cu(0, 0, 0, 0), Tbx::Quat_cu(0, 0, 0, 0));
				Tbx::Dual_quat_cu dqk_0;
				float wk[maxK];

				// dqk_0
				{
					Tbx::Vec3 r, t;
					get_twist(knn_k(knn, 0), r, t);
					float4 nodeVw = get_nodesVw(knn_k(knn, 0));
					Tbx::Vec3 nodesV(convert(read_float3_4(nodeVw)) - v);
					dqk_0.from_twist(r, t);
					float expIn = nodesV.dot(nodesV) * nodeVw.w * nodeVw.w;
					wk[0] = __expf(-0.5f * expIn);
					dq = dq + dqk_0 * wk[0];
				}

				// other dqk_k
#pragma unroll
				for (int k = 1; k < maxK; k++)
				{
					int knnNodeId = knn_k(knn, k);
					
					Tbx::Vec3 r, t;
					get_twist(knnNodeId, r, t);
					float4 nodeVw = get_nodesVw(knnNodeId);
					Tbx::Vec3 nodesV(convert(read_float3_4(nodeVw))-v);
					Tbx::Dual_quat_cu dqk_k;
					dqk_k.from_twist(r, t);
					wk[k] = __expf(-0.5f * nodesV.dot(nodesV) * nodeVw.w * nodeVw.w)
						 *sign(dqk_0.get_non_dual_part().dot(dqk_k.get_non_dual_part()));
					dq = dq + dqk_k * wk[k];		
				}

				Tbx::Dual_quat_cu dq_bar = dq;
				float norm_dq_bar = dq_bar.norm();
				if (norm_dq_bar < Tbx::Dual_quat_cu::epsilon())
					return;
				float inv_norm_dq_bar = 1.f / norm_dq_bar;

				dq = dq * inv_norm_dq_bar; // normalize

				// the grad energy f
				const float f = data_term_penalty(dq.rotate(n).dot(dq.transform(v) - vl));

				// paitial_f_partial_T
				const Tbx::Transfo p_f_p_T = compute_p_f_p_T(n, v, vl, dq);

				for (int knnK = 0; knnK < maxK; knnK++)
				{
					float p_f_p_alpha[VarPerNode];
					int knnNodeId = knn_k(knn, knnK);
					float wk_k = wk[knnK] * inv_norm_dq_bar * 2;
					
					//// comput partial_T_partial_alphak, hard code here.
					Tbx::Dual_quat_cu p_qk_p_alpha;
					Tbx::Transfo p_T_p_alphak;
					Tbx::Vec3 t, r;
					float b, c;
					Tbx::Quat_cu q1;
					get_twist(knnNodeId, r, t);
					{
						float n = r.norm();
						float sin_n, cos_n;
						sincos(n, &sin_n, &cos_n);
						b = n > Tbx::Dual_quat_cu::epsilon() ? sin_n / n : 1;
						c = n > Tbx::Dual_quat_cu::epsilon() ? (cos_n - b) / (n*n) : 0;
						q1 = Tbx::Quat_cu(cos_n*0.5f, r.x*b*0.5f, r.y*b*0.5f, r.z*b*0.5f);
					}

					// alpha0
					p_qk_p_alpha[0] = -r[0] * b;
					p_qk_p_alpha[1] = b + r[0] * r[0] * c;
					p_qk_p_alpha[2] = r[0] * r[1] * c;
					p_qk_p_alpha[3] = r[0] * r[2] * c;
					p_qk_p_alpha = Tbx::Dual_quat_cu::dual_quat_from(p_qk_p_alpha.get_non_dual_part(), t);
					p_T_p_alphak = p_T_p_alphak_func(p_qk_p_alpha, dq_bar, dq,
						inv_norm_dq_bar, wk_k);
					p_f_p_alpha[0] = trace_AtB(p_f_p_T, p_T_p_alphak);

					// alpha1
					p_qk_p_alpha[0] = -r[1] * b;
					p_qk_p_alpha[1] = r[1] * r[0] * c;
					p_qk_p_alpha[2] = b + r[1] * r[1] * c;
					p_qk_p_alpha[3] = r[1] * r[2] * c;
					p_qk_p_alpha = Tbx::Dual_quat_cu::dual_quat_from(p_qk_p_alpha.get_non_dual_part(), t);
					p_T_p_alphak = p_T_p_alphak_func(p_qk_p_alpha, dq_bar, dq,
						inv_norm_dq_bar, wk_k);
					p_f_p_alpha[1] = trace_AtB(p_f_p_T, p_T_p_alphak);

					// alpha2
					p_qk_p_alpha[0] = -r[2] * b;
					p_qk_p_alpha[1] = r[2] * r[0] * c;
					p_qk_p_alpha[2] = r[2] * r[1] * c;
					p_qk_p_alpha[3] = b + r[2] * r[2] * c;
					p_qk_p_alpha = Tbx::Dual_quat_cu::dual_quat_from(p_qk_p_alpha.get_non_dual_part(), t);
					p_T_p_alphak = p_T_p_alphak_func(p_qk_p_alpha, dq_bar, dq,
						inv_norm_dq_bar, wk_k);
					p_f_p_alpha[2] = trace_AtB(p_f_p_T, p_T_p_alphak);

					// alpha3
					p_qk_p_alpha = Tbx::Dual_quat_cu(Tbx::Quat_cu(0, 0, 0, 0),
						Tbx::Quat_cu(-q1[1], q1[0], -q1[3], q1[2]));
					p_T_p_alphak = p_T_p_alphak_func(p_qk_p_alpha, dq_bar, dq,
						inv_norm_dq_bar, wk_k);
					p_f_p_alpha[3] = trace_AtB(p_f_p_T, p_T_p_alphak);

					// alpha4
					p_qk_p_alpha = Tbx::Dual_quat_cu(Tbx::Quat_cu(0, 0, 0, 0),
						Tbx::Quat_cu(-q1[2], q1[3], q1[0], -q1[1]));
					p_T_p_alphak = p_T_p_alphak_func(p_qk_p_alpha, dq_bar, dq,
						inv_norm_dq_bar, wk_k);
					p_f_p_alpha[4] = trace_AtB(p_f_p_T, p_T_p_alphak);

					// alpha5
					p_qk_p_alpha = Tbx::Dual_quat_cu(Tbx::Quat_cu(0, 0, 0, 0),
						Tbx::Quat_cu(-q1[3], -q1[2], q1[1], q1[0]));
					p_T_p_alphak = p_T_p_alphak_func(p_qk_p_alpha, dq_bar, dq,
						inv_norm_dq_bar, wk_k);
					p_f_p_alpha[5] = trace_AtB(p_f_p_T, p_T_p_alphak);

					//// reduce--------------------------------------------------
					int shift = knnNodeId * VarPerNode2;
					int shift_g = knnNodeId * VarPerNode;
					for (int i = 0; i < VarPerNode; ++i)
					{
#pragma unroll
						for (int j = 0; j <= i; ++j)
						{
							atomicAdd(&Hd_[shift + j], p_f_p_alpha[i] * p_f_p_alpha[j]);
#ifdef ENABLE_GPU_DUMP_DEBUG
// debug
if (knnNodeId == 390 && i == 5 && j == 1
&& debug_buffer_pixel_sum2 && debug_buffer_pixel_val
)
{
for (int k = 0; k < VarPerNode; k++)
	debug_buffer_pixel_val[(y*imgWidth + x)*VarPerNode + k] =
	p_f_p_alpha[k];
debug_buffer_pixel_sum2[y*imgWidth + x] = Hd_[shift + j];
}
#endif
						}
						atomicAdd(&g_[shift_g + i], p_f_p_alpha[i] * f);
						shift += VarPerNode;
					}// end for i					
				}// end for knnK
			}// end if found corr
		}// end function ()

		template<int maxK>
		__device__ __forceinline__ Tbx::Dual_quat_cu calc_pixel_dq(WarpField::KnnIdx knn, 
			Tbx::Point3 v, float* wk)const
		{
			Tbx::Dual_quat_cu dqk_0;
			Tbx::Dual_quat_cu dq(Tbx::Quat_cu(0,0,0,0), Tbx::Quat_cu(0,0,0,0));
			// dqk_0
			{
				Tbx::Vec3 r, t;
				get_twist(knn_k(knn, 0), r, t);
				float4 nodeVw = get_nodesVw(knn_k(knn, 0));
				Tbx::Vec3 nodesV(convert(read_float3_4(nodeVw)) - v);
				dqk_0.from_twist(r, t);
				wk[0] = __expf(-0.5f * nodesV.dot(nodesV) * nodeVw.w * nodeVw.w);
				dq += dqk_0 * wk[0];
			}

			// other dqk_k
#pragma unroll
			for (int k = 1; k < maxK; k++)
			{
				float4 nodeVw = get_nodesVw(knn_k(knn, k));
				Tbx::Vec3 nodesV(convert(read_float3_4(nodeVw)) - v);
				Tbx::Dual_quat_cu dqk_k;
				Tbx::Vec3 r, t;
				get_twist(knn_k(knn, k), r, t);
				dqk_k.from_twist(r, t);
				wk[k] = __expf(-0.5f * nodesV.dot(nodesV) * nodeVw.w * nodeVw.w)
					*sign(dqk_0.get_non_dual_part().dot(dqk_k.get_non_dual_part()));
				dq += dqk_k * wk[k];
			}
			return dq;
		}

		__device__ __forceinline__ void exchange_ri_k(WarpField::KnnIdx knn, 
			const float* wk, int k, int i, Tbx::Dual_quat_cu& dq, float& inc)const
		{
			Tbx::Vec3 r, t;
			get_twist(knn_k(knn, k), r, t);
			Tbx::Dual_quat_cu old_dqk, new_dqk;
			old_dqk.from_twist(r, t);
			inc = get_numeric_inc(r[i]);
			r[i] += inc;
			new_dqk.from_twist(r, t);
			dq -= old_dqk * wk[k];
			dq += new_dqk * wk[k] * sign(old_dqk.get_non_dual_part().dot(new_dqk.get_non_dual_part()));
		}
		__device__ __forceinline__ void exchange_ti_k(WarpField::KnnIdx knn,
			const float* wk, int k, int i, Tbx::Dual_quat_cu& dq, float& inc)const
		{
			Tbx::Vec3 r, t;
			get_twist(knn_k(knn, k), r, t);
			Tbx::Dual_quat_cu old_dqk, new_dqk;
			old_dqk.from_twist(r, t);
			inc = get_numeric_inc(t[i]);
			t[i] += inc;
			new_dqk.from_twist(r, t);
			dq -= old_dqk * wk[k];
			dq += new_dqk * wk[k] * sign(old_dqk.get_non_dual_part().dot(new_dqk.get_non_dual_part()));
		}

		__device__ __forceinline__ float get_numeric_inc(float v) const
		{
			return max( 1e-5f, v* 1e-3f);
		}

		template<int maxK>
		__device__ __forceinline__ void calc_dataterm_numeric() const
		{
			const int x = threadIdx.x + blockIdx.x * blockDim.x;
			const int y = threadIdx.y + blockIdx.y * blockDim.y;
			if (x >= imgWidth || y >= imgHeight)
				return;

			const KnnIdx knn = vmapKnn(y, x);
			Tbx::Point3 v(convert(read_float3_4(vmap_cano(y, x))));
			Tbx::Vec3 n(convert(read_float3_4(nmap_cano(y, x))));

			if (isnan(n.x) || isnan(v.x))
				return;

			// 1. get all nodes params
			// 2. compute function=================================================
			float wk[maxK];
			Tbx::Dual_quat_cu dq = calc_pixel_dq<maxK>(knn, v, wk);
			float norm_dq = dq.norm();
			if (norm_dq < Tbx::Dual_quat_cu::epsilon())
				return;
			Tbx::Dual_quat_cu dq_not_normalized = dq;
			dq = dq * (1.f / norm_dq); // normalize

			// find corr
			Tbx::Vec3 nwarp = Tlw*dq.rotate(n);
			Tbx::Point3 vwarp = Tlw*dq.transform(v);
			Tbx::Point3 vl;
			//bool corr_found = search(convert(vwarp), convert(nwarp), vl);
			bool corr_found = search(x, y, vl);
			if (!corr_found)
				return;

			// the grad energy
			const float f = nwarp.dot(vwarp - vl);
			const float psi_f = data_term_penalty(f);

			// 3. compute jacobi
			for (int knnK = 0; knnK < maxK; knnK++)
			{
				float df[6];

				// 3.0 p_r[0:2]
				for (int i = 0; i < 3; i++)
				{
					float inc;
					Tbx::Dual_quat_cu dq1 = dq_not_normalized;
					exchange_ri_k(knn, wk, knnK, i, dq1, inc);
					dq1 *= (1.f / dq1.norm());
					nwarp = Tlw*dq1.rotate(n);
					vwarp = Tlw*dq1.transform(v);

					Tbx::Point3 vl1 = vl;
					//corr_found = search(convert(vwarp), convert(nwarp), vl1);
					//if (!corr_found)
					//	return;

					float f1 = nwarp.dot(vwarp - vl1);
					df[i] = (f1 - f) / inc;
				}// i=0:3

				// 3.1 p_t[0:2]
				for (int i = 0; i < 3; i++)
				{
					float inc;
					Tbx::Dual_quat_cu dq1 = dq_not_normalized;
					exchange_ti_k(knn, wk, knnK, i, dq1, inc);
					dq1 *= (1.f / dq1.norm());
					nwarp = Tlw*dq1.rotate(n);
					vwarp = Tlw*dq1.transform(v);

					Tbx::Point3 vl1 = vl;
					//corr_found = search(convert(vwarp), convert(nwarp), vl1);
					//if (!corr_found)
					//	return;

					float f1 = nwarp.dot(vwarp - vl1);
					df[i+3] = (f1 - f) / inc;
				}// i=0:3

				//// reduce--------------------------------------------------
				int shift = knn_k(knn, knnK) * VarPerNode2;
				int shift_g = knn_k(knn, knnK) * VarPerNode;
				for (int i = 0; i < VarPerNode; ++i)
				{
#pragma unroll
					for (int j = 0; j <= i; ++j)
						atomicAdd(&Hd_[shift + j], df[i] * df[j]);
					atomicAdd(&g_[shift_g + i], df[i] * psi_f);
					shift += VarPerNode;
				}// end for i
			}// end for knnK
		}// end function ()

		template<int maxK>
		__device__ __forceinline__ void calcTotalEnergy()const
		{
			const int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
			const int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

			Tbx::Point3 vl;
			bool found_coresp = false;
			if (x < imgWidth && y < imgHeight)
				found_coresp = search(x, y, vl);

			if (found_coresp)
			{
				Tbx::Point3 v(convert(read_float3_4(vmap_cano(y, x))));
				Tbx::Vec3 n(convert(read_float3_4(nmap_cano(y, x))));

				const KnnIdx knn = vmapKnn(y, x);
				Tbx::Dual_quat_cu dq(Tbx::Quat_cu(0, 0, 0, 0), Tbx::Quat_cu(0, 0, 0, 0));
				Tbx::Dual_quat_cu dqk_0;
				float wk[maxK];
				// dqk_0
				{
					Tbx::Vec3 r, t;
					get_twist(knn_k(knn, 0), r, t);
					float4 nodeVw = get_nodesVw(knn_k(knn, 0));
					Tbx::Vec3 nodesV(convert(read_float3_4(nodeVw)) - v);
					dqk_0.from_twist(r, t);
					float expIn = nodesV.dot(nodesV) * nodeVw.w * nodeVw.w;
					wk[0] = __expf(-0.5f * expIn);
					dq = dq + dqk_0 * wk[0];
				}

				// other dqk_k
#pragma unroll
				for (int k = 1; k < maxK; k++)
				{
					int knnNodeId = knn_k(knn, k);

					Tbx::Vec3 r, t;
					get_twist(knnNodeId, r, t);
					float4 nodeVw = get_nodesVw(knnNodeId);
					Tbx::Vec3 nodesV(convert(read_float3_4(nodeVw)) - v);
					Tbx::Dual_quat_cu dqk_k;
					dqk_k.from_twist(r, t);
					wk[k] = __expf(-0.5f * nodesV.dot(nodesV) * nodeVw.w * nodeVw.w)
						*sign(dqk_0.get_non_dual_part().dot(dqk_k.get_non_dual_part()));
					dq = dq + dqk_k * wk[k];
				}

				float norm_dq = dq.norm();
				if (norm_dq < Tbx::Dual_quat_cu::epsilon())
					return;
				dq = dq * (1.f / norm_dq); // normalize

				// the grad energy f
				const float f = data_term_energy((Tlw*dq.rotate(n)).dot(Tlw*dq.transform(v) - vl));
				//atomicAdd(totalEnergy, f);
				totalEnergy[y*imgWidth + x] = f;
			}//end if find corr
#ifdef DEBUG_ASSIGN_BIG_ENERGY_TO_NO_CORR
			else // debug: add constant penalty
			{
				totalEnergy[y*imgWidth + x] = data_term_energy(psi_data);
			}
#endif
		}
	};

	template<int maxK>
	__global__ void dataTermCombinedKernel(const DataTermCombined cs)
	{
#ifdef CALC_DATA_TERM_NUMERIC
		cs.calc_dataterm_numeric<maxK>();
#else
		cs.calc_dataterm<maxK>();
#endif
	}

	void GpuGaussNewtonSolver::calcDataTerm()
	{
		DataTermCombined cs;
		cs.angleThres = m_param->fusion_nonRigid_angleThreSin;
		cs.distThres = m_param->fusion_nonRigid_distThre;
		cs.Hd_ = m_Hd;
		cs.g_ = m_g;
		cs.imgHeight = m_vmap_cano->rows();
		cs.imgWidth = m_vmap_cano->cols();
		cs.intr = m_intr;
		cs.nmap_cano = *m_nmap_cano;
		cs.nmap_live = *m_nmap_live;
		cs.nmap_warp = *m_nmap_warp;
		cs.vmap_cano = *m_vmap_cano;
		cs.vmap_live = *m_vmap_live;
		cs.vmap_warp = *m_vmap_warp;
		cs.vmapKnn = m_vmapKnn;
		cs.nNodes = m_numNodes;
		cs.Tlw = m_pWarpField->get_rigidTransform();
		cs.Tlw_inv = m_pWarpField->get_rigidTransform().fast_invert();
		cs.psi_data = m_param->fusion_psi_data;

#ifdef ENABLE_GPU_DUMP_DEBUG
		// debugging
		DeviceArray<float> pixelSum2, pixelVal;
		pixelSum2.create(cs.imgHeight*cs.imgWidth);
		hipMemset(pixelSum2.ptr(), 0, pixelSum2.sizeBytes());
		pixelVal.create(cs.imgHeight*cs.imgWidth*VarPerNode);
		hipMemset(pixelVal.ptr(), 0, pixelVal.sizeBytes());
		cs.debug_buffer_pixel_sum2 = pixelSum2;
		cs.debug_buffer_pixel_val = pixelVal;
#endif

		//////////////////////////////
		dim3 block(CTA_SIZE_X, CTA_SIZE_Y);
		dim3 grid(1, 1, 1);
		grid.x = divUp(cs.imgWidth, block.x);
		grid.y = divUp(cs.imgHeight, block.y);
		int maxK = min(WarpField::KnnK, m_pWarpField->getNumNodesInLevel(0));
		switch (maxK)
		{
		case 1:
			dataTermCombinedKernel<1> << <grid, block >> >(cs);
			break;
		case 2:
			dataTermCombinedKernel<2> << <grid, block >> >(cs);
			break;
		case 3:
			dataTermCombinedKernel<3> << <grid, block >> >(cs);
			break;
		case 4:
			dataTermCombinedKernel<4> << <grid, block >> >(cs);
			break;
		default:
			throw std::exception("non supported KnnK in data term calc!");
		}
		cudaSafeCall(hipGetLastError(), "dataTermCombinedKernel");

		// debugging
#ifdef ENABLE_GPU_DUMP_DEBUG
		{
			std::vector<float> ps, pv;
			pixelSum2.download(ps);
			pixelVal.download(pv);

			FILE* pFile = fopen("D:/tmp/gpu_pixel.txt", "w");
			for (int i = 0; i < ps.size(); i++)
			{
				fprintf(pFile, "%ef %ef %ef %ef %ef %ef %ef\n",
					pv[i * 6 + 0], pv[i * 6 + 1], pv[i * 6 + 2],
					pv[i * 6 + 3], pv[i * 6 + 4], pv[i * 6 + 5],
					ps[i]);
			}
			fclose(pFile);
		}
#endif
	}

	template<int maxK>
	__global__ void calcDataTermTotalEnergyKernel(const DataTermCombined cs)
	{
		cs.calcTotalEnergy<maxK>();
	}

#pragma endregion

#pragma region --define sparse structure
	__global__ void count_Jr_rows_kernel(int* rctptr, int nMaxNodes)
	{
		int i = threadIdx.x + blockIdx.x*blockDim.x;
		if (i >= nMaxNodes)
			return;
	
		WarpField::KnnIdx knn = get_nodesKnn(i);
		int numK = -1;
		for (int k = 0; k < WarpField::KnnK; ++k)
		{
			if (knn_k(knn, k) < nMaxNodes)
				numK = k;
		}

		// each node generate 6*maxK rows
		rctptr[i] = (numK + 1) * RowPerNode_RegTerm;
		
		if (i == 0)
			rctptr[nMaxNodes] = 0;
	}

	__global__ void compute_row_map_kernel(GpuGaussNewtonSolver::JrRow2NodeMapper* row2nodeId, 
		const int* rctptr, int nMaxNodes)
	{
		int iNode = threadIdx.x + blockIdx.x*blockDim.x;
		if (iNode < nMaxNodes)
		{
			int row_b = rctptr[iNode];
			int row_e = rctptr[iNode+1];
			for (int r = row_b; r < row_e; r++)
			{
				GpuGaussNewtonSolver::JrRow2NodeMapper mp;
				mp.nodeId = iNode;
				mp.k = (r - row_b) / RowPerNode_RegTerm;
				mp.ixyz = r - RowPerNode_RegTerm * mp.k;
				row2nodeId[r] = mp;
			}
		}
	}

	__global__ void compute_Jr_rowPtr_colIdx_kernel(
		int* rptr, int* rptr_coo, int* colIdx,
		const GpuGaussNewtonSolver::JrRow2NodeMapper* row2nodeId, 
		int nMaxNodes, int nRows)
	{
		enum{
			VarPerNode = GpuGaussNewtonSolver::VarPerNode,
			ColPerRow = VarPerNode * 2
		};
		const int iRow = threadIdx.x + blockIdx.x*blockDim.x;
		if (iRow >= nRows)
			return;

		const int iNode = row2nodeId[iRow].nodeId;
		if (iNode < nMaxNodes)
		{
			WarpField::KnnIdx knn = get_nodesKnn(iNode);
			int knnNodeId = knn_k(knn, row2nodeId[iRow].k);
			if (knnNodeId < nMaxNodes)
			{
				int col_b = iRow*ColPerRow;
				rptr[iRow] = col_b;

				// each row 2*VerPerNode Cols
				// 1. self
				for (int j = 0; j < VarPerNode; j++, col_b++)
				{
					rptr_coo[col_b] = iRow;
					colIdx[col_b] = iNode*VarPerNode + j;
				}// j
				// 2. neighbor
				for (int j = 0; j < VarPerNode; j++, col_b++)
				{
					rptr_coo[col_b] = iRow;
					colIdx[col_b] = knnNodeId*VarPerNode + j;
				}// j
			}// end if knnNodeId
		}

		// the 1st thread also write the last value
		if (iRow == 0)
			rptr[nRows] = nRows * ColPerRow;
	}

	__global__ void calc_B_cidx_kernel(int* B_rptr_coo, int* B_cidx, 
		const int* B_rptr, int nRows, int nMaxNodes, int nLv0Nodes)
	{
		int iRow = threadIdx.x + blockIdx.x*blockDim.x;
		if (iRow < nRows)
		{
			int iNode = iRow / GpuGaussNewtonSolver::VarPerNode;

			WarpField::KnnIdx knn = get_nodesKnn(iNode);
			int col_b = B_rptr[iRow];
			for (int k = 0; k < WarpField::KnnK; ++k)
			{
				int knnNodeId = knn_k(knn, k);
				if (knnNodeId < nMaxNodes)
				{
					// 2. neighbor
					for (int j = 0; j < GpuGaussNewtonSolver::VarPerNode; j++, col_b++)
					{
						B_rptr_coo[col_b] = iRow;
						B_cidx[col_b] = (knnNodeId-nLv0Nodes)*GpuGaussNewtonSolver::VarPerNode + j;
					}// j
				}
			}
		}
	}

	void GpuGaussNewtonSolver::initSparseStructure()
	{
		// 1. compute Jr structure ==============================================
		// 1.0. decide the total rows we have for each nodes
		{
			dim3 block(CTA_SIZE);
			dim3 grid(divUp(m_numNodes, block.x));
			count_Jr_rows_kernel << <grid, block >> >(m_Jr_RowCounter.ptr(), m_numNodes);
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::initSparseStructure::count_Jr_rows_kernel");
			thrust_wrapper::exclusive_scan(m_Jr_RowCounter.ptr(), m_Jr_RowCounter.ptr(), m_numNodes + 1);
			cudaSafeCall(hipMemcpy(&m_Jrrows, m_Jr_RowCounter.ptr() + m_numNodes,
				sizeof(int), hipMemcpyDeviceToHost), "copy Jr rows to host");
		}

		// 1.1. collect nodes edges info:
		//	each low-level nodes are connected to k higher level nodes
		//	but the connections are not stored for the higher level nodes
		//  thus when processing each node, we add 2*k edges, w.r.t. 2*k*3 rows: each (x,y,z) a row
		//	for each row, there are exactly 2*VarPerNode values
		//	after this step, we can get the CSR/COO structure
		if (m_Jrrows > 0)
		{
			dim3 block(CTA_SIZE);
			dim3 grid(divUp(m_numNodes, block.x));
			compute_row_map_kernel << <grid, block >> >(m_Jr_RowMap2NodeId.ptr(), m_Jr_RowCounter.ptr(), m_numNodes);
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::initSparseStructure::compute_row_map_kernel");
		}
		if (m_Jrrows > 0)
		{
			CHECK_LE(m_Jrrows + 1, m_Jr_RowPtr.size());
			CHECK_LE(m_Jrcols + 1, m_Jrt_RowPtr.size());
			dim3 block(CTA_SIZE);
			dim3 grid(divUp(m_Jrrows, block.x));
			compute_Jr_rowPtr_colIdx_kernel << <grid, block >> >(m_Jr_RowPtr.ptr(),
				m_Jr_RowPtr_coo.ptr(), m_Jr_ColIdx.ptr(), m_Jr_RowMap2NodeId.ptr(), m_numNodes, m_Jrrows);
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::initSparseStructure::compute_Jr_rowPtr_kernel");
			cudaSafeCall(hipMemcpy(&m_Jrnnzs, m_Jr_RowPtr.ptr() + m_Jrrows,
				sizeof(int), hipMemcpyDeviceToHost), "copy Jr nnz to host");
			CHECK_LE(m_Jrnnzs, m_Jr_RowPtr_coo.size());
			CHECK_LE(0, m_Jrnnzs);
		}

		// 2. compute Jrt structure ==============================================
		// 2.1. fill (row, col) as (col, row) from Jr and sort.
		if (m_Jrrows > 0)
		{
			hipMemcpy(m_Jrt_RowPtr_coo.ptr(), m_Jr_ColIdx.ptr(), m_Jrnnzs*sizeof(int), hipMemcpyDeviceToDevice);
			hipMemcpy(m_Jrt_ColIdx.ptr(), m_Jr_RowPtr_coo.ptr(), m_Jrnnzs*sizeof(int), hipMemcpyDeviceToDevice);
			// !!!NOTE: we must use mergesort here, it can guarentees the order of values of the same key
			modergpu_wrapper::mergesort_by_key(m_Jrt_RowPtr_coo.ptr(), m_Jrt_ColIdx.ptr(), m_Jrnnzs);
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::initSparseStructure::mergesort_by_key1");

			// 2.2. extract CSR rowptr info.
			if (HIPSPARSE_STATUS_SUCCESS != hipsparseXcoo2csr(m_cuSparseHandle,
				m_Jrt_RowPtr_coo.ptr(), m_Jrnnzs, m_Jrcols,
				m_Jrt_RowPtr.ptr(), HIPSPARSE_INDEX_BASE_ZERO))
				throw std::exception("GpuGaussNewtonSolver::initSparseStructure::hipsparseXcoo2csr failed");
		}

		// 3. compute B structure ==============================================
		// 3.1 the row ptr of B is the same with the first L0 rows of Jrt.
		if (m_Brows > 0)
		{
			CHECK_LE(m_Brows, m_B_RowPtr.size());
			CHECK_LE(m_Bcols, m_Bt_RowPtr.size());
			hipMemcpy(m_B_RowPtr.ptr(), m_Jrt_RowPtr.ptr(), (m_Brows + 1)*sizeof(int), hipMemcpyDeviceToDevice);
			cudaSafeCall(hipMemcpy(&m_Bnnzs, m_B_RowPtr.ptr() + m_Brows,
				sizeof(int), hipMemcpyDeviceToHost), "copy B nnz to host");
			CHECK_LE(m_Bnnzs, m_B_RowPtr_coo.size());
		}
		
		// 3.2 the col-idx of B
		if (m_Brows > 0)
		{
			dim3 block(CTA_SIZE);
			dim3 grid(divUp(m_Brows, block.x));
			calc_B_cidx_kernel << <grid, block >> >(m_B_RowPtr_coo.ptr(),
				m_B_ColIdx.ptr(), m_B_RowPtr.ptr(), m_Brows, m_numNodes, m_numLv0Nodes);
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::initSparseStructure::calc_B_cidx_kernel");
		}

		// 3.3 sort to compute Bt
		if (m_Bnnzs > 0)
		{
			hipMemcpy(m_Bt_RowPtr_coo.ptr(), m_B_ColIdx.ptr(), m_Bnnzs*sizeof(int), hipMemcpyDeviceToDevice);
			hipMemcpy(m_Bt_ColIdx.ptr(), m_B_RowPtr_coo.ptr(), m_Bnnzs*sizeof(int), hipMemcpyDeviceToDevice);
			modergpu_wrapper::mergesort_by_key(m_Bt_RowPtr_coo.ptr(), m_Bt_ColIdx.ptr(), m_Bnnzs);
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::initSparseStructure::mergesort_by_key2");
			if (HIPSPARSE_STATUS_SUCCESS != hipsparseXcoo2csr(m_cuSparseHandle,
				m_Bt_RowPtr_coo.ptr(), m_Bnnzs, m_Bcols,
				m_Bt_RowPtr.ptr(), HIPSPARSE_INDEX_BASE_ZERO))
				throw std::exception("GpuGaussNewtonSolver::initSparseStructure::hipsparseXcoo2csr failed");
		}
	}

#pragma endregion

#pragma region --calc reg term
	struct RegTermJacobi
	{
		typedef WarpField::KnnIdx KnnIdx;
		typedef WarpField::IdxType IdxType;
		typedef GpuGaussNewtonSolver::JrRow2NodeMapper Mapper;
		enum
		{
			KnnK = WarpField::KnnK,
			VarPerNode = GpuGaussNewtonSolver::VarPerNode,
			VarPerNode2 = VarPerNode*VarPerNode,
			ColPerRow = VarPerNode * 2
		};

		int nNodes;
		int nRows;
		const Mapper* rows2nodeIds;
		const int* rptr;
		const int* cidx;
		mutable float* vptr;
		mutable float* fptr;

		float psi_reg;
		float lambda;

		float* totalEnergy;

		__device__ __forceinline__  Tbx::Dual_quat_cu p_qk_p_alpha_func(Tbx::Dual_quat_cu dq, int i)const
		{
			Tbx::Vec3 t, r;
			float b, c, n;
			Tbx::Quat_cu q0(0, 0, 0, 0), q1 = dq.get_non_dual_part();
			switch (i)
			{
			case 0:
				dq.to_twist(r, t);
				n = r.norm();
				if (n > Tbx::Dual_quat_cu::epsilon())
				{
					b = sin(n) / n;
					c = (cos(n) - b) / (n*n);
					q0.coeff0 = -r.x * b;
					q0.coeff1 = b + r.x*r.x*c;
					q0.coeff2 = r.x*r.y*c;
					q0.coeff3 = r.x*r.z*c;
				}
				else
				{
					q0.coeff0 = 0;
					q0.coeff1 = 1;
					q0.coeff2 = 0;
					q0.coeff3 = 0;
				}

				q1.coeff0 = (t.x * q0.coeff1 + t.y * q0.coeff2 + t.z * q0.coeff3) * (-0.5);
				q1.coeff1 = (t.x * q0.coeff0 + t.y * q0.coeff3 - t.z * q0.coeff2) * 0.5;
				q1.coeff2 = (-t.x * q0.coeff3 + t.y * q0.coeff0 + t.z * q0.coeff1) * 0.5;
				q1.coeff3 = (t.x * q0.coeff2 - t.y * q0.coeff1 + t.z * q0.coeff0) * 0.5;
				return Tbx::Dual_quat_cu(q0, q1);
			case 1:
				dq.to_twist(r, t);
				n = r.norm();
				if (n > Tbx::Dual_quat_cu::epsilon())
				{
					b = sin(n) / n;
					c = (cos(n) - b) / (n*n);
					q0.coeff0 = -r.y * b;
					q0.coeff1 = r.y*r.x*c;
					q0.coeff2 = b + r.y*r.y*c;
					q0.coeff3 = r.y*r.z*c;
				}
				else
				{
					q0.coeff0 = 0;
					q0.coeff1 = 0;
					q0.coeff2 = 1;
					q0.coeff3 = 0;
				}

				q1.coeff0 = (t.x * q0.coeff1 + t.y * q0.coeff2 + t.z * q0.coeff3) * (-0.5);
				q1.coeff1 = (t.x * q0.coeff0 + t.y * q0.coeff3 - t.z * q0.coeff2) * 0.5;
				q1.coeff2 = (-t.x * q0.coeff3 + t.y * q0.coeff0 + t.z * q0.coeff1) * 0.5;
				q1.coeff3 = (t.x * q0.coeff2 - t.y * q0.coeff1 + t.z * q0.coeff0) * 0.5;
				return Tbx::Dual_quat_cu(q0, q1);
			case 2:
				dq.to_twist(r, t);
				n = r.norm();
				if (n > Tbx::Dual_quat_cu::epsilon())
				{
					b = sin(n) / n;
					c = (cos(n) - b) / (n*n);

					q0.coeff0 = -r.z * b;
					q0.coeff1 = r.z*r.x*c;
					q0.coeff2 = r.z*r.y*c;
					q0.coeff3 = b + r.z*r.z*c;
				}
				else
				{
					q0.coeff0 = 0;
					q0.coeff1 = 0;
					q0.coeff2 = 0;
					q0.coeff3 = 1;
				}

				q1.coeff0 = (t.x * q0.coeff1 + t.y * q0.coeff2 + t.z * q0.coeff3) * (-0.5);
				q1.coeff1 = (t.x * q0.coeff0 + t.y * q0.coeff3 - t.z * q0.coeff2) * 0.5;
				q1.coeff2 = (-t.x * q0.coeff3 + t.y * q0.coeff0 + t.z * q0.coeff1) * 0.5;
				q1.coeff3 = (t.x * q0.coeff2 - t.y * q0.coeff1 + t.z * q0.coeff0) * 0.5;
				return Tbx::Dual_quat_cu(q0, q1);
			case 3:
				return Tbx::Dual_quat_cu(q0, Tbx::Quat_cu(-q1.coeff1, q1.coeff0, -q1.coeff3, q1.coeff2))*0.5;
			case 4:
				return Tbx::Dual_quat_cu(q0, Tbx::Quat_cu(-q1.coeff2, q1.coeff3, q1.coeff0, -q1.coeff1))*0.5;
			case 5:
				return Tbx::Dual_quat_cu(q0, Tbx::Quat_cu(-q1.coeff3, -q1.coeff2, q1.coeff1, q1.coeff0))*0.5;
			default:
				return Tbx::Dual_quat_cu();
			}
		}

		__device__ __forceinline__  float reg_term_energy(Tbx::Vec3 f)const
		{
#ifdef USE_L2_NORM_REG_TERM
			return 0.5f*f.dot(f);
#else
			// the robust Huber penelty gradient
			float s = 0;
			float norm = f.norm();
			if (norm < psi_reg)
				s = norm * norm * 0.5f;
			else
				s = psi_reg*(norm - psi_reg*0.5f);
			return s;
#endif
		}

		__device__ __forceinline__  Tbx::Vec3 reg_term_penalty(Tbx::Vec3 f)const
		{
#ifdef USE_L2_NORM_REG_TERM
			return f;
#else
			// the robust Huber penelty gradient
			Tbx::Vec3 df;
			float norm = f.norm();
			if (norm < psi_reg)
				df = f;
			else
			for (int k = 0; k < 3; k++)
				df[k] = f[k]*psi_reg / norm;
			return df;
#endif
		}

		__device__ __forceinline__  Tbx::Transfo p_SE3_p_alpha_func(Tbx::Dual_quat_cu dq, int i)const
		{
			Tbx::Transfo T = Tbx::Transfo::empty();
			Tbx::Dual_quat_cu p_dq_p_alphai = p_qk_p_alpha_func(dq, i) * 2.f;

			//// evaluate p_dqi_p_alphak, heavily hard code here
			//// this hard code is crucial to the performance 
			// 0:
			// (0, -z0, y0, x1,
			// z0, 0, -x0, y1,
			//-y0, x0, 0, z1,
			// 0, 0, 0, 0) * 2;
			float p_dqi_p_alphak = p_dq_p_alphai[0];
			T[1] += -dq[3] * p_dqi_p_alphak;
			T[2] += dq[2] * p_dqi_p_alphak;
			T[3] += dq[5] * p_dqi_p_alphak;
			T[4] += dq[3] * p_dqi_p_alphak;
			T[6] += -dq[1] * p_dqi_p_alphak;
			T[7] += dq[6] * p_dqi_p_alphak;
			T[8] += -dq[2] * p_dqi_p_alphak;
			T[9] += dq[1] * p_dqi_p_alphak;
			T[11] += dq[7] * p_dqi_p_alphak;

			// 1
			//( 0, y0, z0, -w1,
			//	y0, -2 * x0, -w0, -z1,
			//	z0, w0, -2 * x0, y1,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[1];
			T[1] += dq[2] * p_dqi_p_alphak;
			T[2] += dq[3] * p_dqi_p_alphak;
			T[3] += -dq[4] * p_dqi_p_alphak;
			T[4] += dq[2] * p_dqi_p_alphak;
			T[5] += -dq[1] * p_dqi_p_alphak * 2;
			T[6] += -dq[0] * p_dqi_p_alphak;
			T[7] += -dq[7] * p_dqi_p_alphak;
			T[8] += dq[3] * p_dqi_p_alphak;
			T[9] += dq[0] * p_dqi_p_alphak;
			T[10] += -dq[1] * p_dqi_p_alphak * 2;
			T[11] += dq[6] * p_dqi_p_alphak;

			// 2.
			// (-2 * y0, x0, w0, z1,
			//	x0, 0, z0, -w1,
			//	-w0, z0, -2 * y0, -x1,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[2];
			T[0] += -dq[2] * p_dqi_p_alphak * 2;
			T[1] += dq[1] * p_dqi_p_alphak;
			T[2] += dq[0] * p_dqi_p_alphak;
			T[3] += dq[7] * p_dqi_p_alphak;
			T[4] += dq[1] * p_dqi_p_alphak;
			T[6] += dq[3] * p_dqi_p_alphak;
			T[7] += -dq[4] * p_dqi_p_alphak;
			T[8] += -dq[0] * p_dqi_p_alphak;
			T[9] += dq[3] * p_dqi_p_alphak;
			T[10] += -dq[2] * p_dqi_p_alphak * 2;
			T[11] += -dq[5] * p_dqi_p_alphak;

			// 3.
			// (-2 * z0, -w0, x0, -y1,
			//	w0, -2 * z0, y0, x1,
			//	x0, y0, 0, -w1,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[3];
			T[0] += -dq[3] * p_dqi_p_alphak * 2;
			T[1] += -dq[0] * p_dqi_p_alphak;
			T[2] += dq[1] * p_dqi_p_alphak;
			T[3] += -dq[6] * p_dqi_p_alphak;
			T[4] += dq[0] * p_dqi_p_alphak;
			T[5] += -dq[3] * p_dqi_p_alphak * 2;
			T[6] += dq[2] * p_dqi_p_alphak;
			T[7] += dq[5] * p_dqi_p_alphak;
			T[8] += dq[1] * p_dqi_p_alphak;
			T[9] += dq[2] * p_dqi_p_alphak;
			T[11] += -dq[4] * p_dqi_p_alphak;

			// 4.
			//( 0, 0, 0, -x0,
			//	0, 0, 0, -y0,
			//	0, 0, 0, -z0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[4];
			T[3] += -dq[1] * p_dqi_p_alphak;
			T[7] += -dq[2] * p_dqi_p_alphak;
			T[11] += -dq[3] * p_dqi_p_alphak;

			// 5. 
			// (0, 0, 0, w0,
			//	0, 0, 0, z0,
			//	0, 0, 0, -y0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[5];
			T[3] += dq[0] * p_dqi_p_alphak;
			T[7] += dq[3] * p_dqi_p_alphak;
			T[11] += -dq[2] * p_dqi_p_alphak;

			// 6. 
			// (0, 0, 0, -z0,
			//	0, 0, 0, w0,
			//	0, 0, 0, x0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[6];
			T[3] += -dq[3] * p_dqi_p_alphak;
			T[7] += dq[0] * p_dqi_p_alphak;
			T[11] += dq[1] * p_dqi_p_alphak;

			// 7.
			// (0, 0, 0, y0,
			//	0, 0, 0, -x0,
			//	0, 0, 0, w0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[7];
			T[3] += dq[2] * p_dqi_p_alphak;
			T[7] += -dq[1] * p_dqi_p_alphak;
			T[11] += dq[0] * p_dqi_p_alphak;

			return T;
		}

		__device__ __forceinline__ void operator () () const
		{
			const int iRow = (threadIdx.x + blockIdx.x * blockDim.x)*RowPerNode_RegTerm;
		
			if (iRow >= nRows)
				return;

			Mapper mapper = rows2nodeIds[iRow];
			int knnNodeId = knn_k(get_nodesKnn(mapper.nodeId), mapper.k);

			if (knnNodeId >= nNodes)
				return;

			int cooPos = rptr[iRow];

			Tbx::Dual_quat_cu dqi, dqj;
			Tbx::Vec3 ri, ti, rj, tj;
			get_twist(mapper.nodeId, ri, ti);
			get_twist(knnNodeId, rj, tj);
			dqi.from_twist(ri, ti);
			dqj.from_twist(rj, tj);

			float4 nodeVwi = get_nodesVw(mapper.nodeId);
			float4 nodeVwj = get_nodesVw(knnNodeId);
			Tbx::Point3 vi(convert(read_float3_4(nodeVwi)));
			Tbx::Point3 vj(convert(read_float3_4(nodeVwj)));
			float alpha_ij = max(1.f / nodeVwi.w, 1.f / nodeVwj.w);
			float ww = sqrt(lambda * alpha_ij);

			//if (isinf(nodeVwj.w))
			//	printf("inf found: %d %d %f %f %f %f\n", mapper.nodeId, knnNodeId, 
			//	nodeVwj.w, 1.f / nodeVwj.w, alpha_ij, ww);

			// debug
			//float ww2 = ww*ww;
			//ww = 1;
			//ww *= ww;

			// energy=============================================
			Tbx::Vec3 val = dqi.transform(Tbx::Point3(vj)) - dqj.transform(Tbx::Point3(vj));
			val = reg_term_penalty(val);

			fptr[iRow + 0] = val.x * ww;
			fptr[iRow + 1] = val.y * ww;
			fptr[iRow + 2] = val.z * ww;

#ifndef DEFINE_USE_HALF_GRAPH_EDGE
			Tbx::Vec3 val1 = dqj.transform(Tbx::Point3(vi)) - dqi.transform(Tbx::Point3(vi));
			val1 = reg_term_penalty(val1);
			fptr[iRow + 3] = val1.x * ww;
			fptr[iRow + 4] = val1.y * ww;
			fptr[iRow + 5] = val1.z * ww;
#endif

			// debug
			//ww = 1;
			//ww = ww2;

			// jacobi=============================================
			for (int ialpha = 0; ialpha < VarPerNode; ialpha++)
			{
				Tbx::Transfo p_Ti_p_alpha = p_SE3_p_alpha_func(dqi, ialpha);
				Tbx::Transfo p_Tj_p_alpha = p_SE3_p_alpha_func(dqj, ialpha);

				// partial_psi_partial_alpha
				Tbx::Vec3 p_psi_p_alphai_j = (p_Ti_p_alpha * vj) * ww;
				Tbx::Vec3 p_psi_p_alphaj_j = (p_Tj_p_alpha * vj) * (-ww);
#ifndef DEFINE_USE_HALF_GRAPH_EDGE
				Tbx::Vec3 p_psi_p_alphai_i = (p_Ti_p_alpha * vi) * (-ww);
				Tbx::Vec3 p_psi_p_alphaj_i = (p_Tj_p_alpha * vi) * ww;
#endif

				for (int ixyz = 0; ixyz < 3; ixyz++)
				{
					int pos = cooPos + ixyz*ColPerRow + ialpha;
					vptr[pos] = p_psi_p_alphai_j[ixyz];
					vptr[pos + VarPerNode] = p_psi_p_alphaj_j[ixyz];
#ifndef DEFINE_USE_HALF_GRAPH_EDGE
					pos += 3 * ColPerRow;
					vptr[pos] = p_psi_p_alphai_i[ixyz];
					vptr[pos + VarPerNode] = p_psi_p_alphaj_i[ixyz];
#endif
				}
			}// end for ialpha
		}// end function ()

		__device__ __forceinline__ float get_numeric_inc(float v) const
		{
			return max(1e-5f, v* 1e-3f);
		}

		__device__ __forceinline__ void calc_reg_numeric () const
		{
			const int iRow = (threadIdx.x + blockIdx.x * blockDim.x)*RowPerNode_RegTerm;

			if (iRow >= nRows)
				return;

			Mapper mapper = rows2nodeIds[iRow];
			int knnNodeId = knn_k(get_nodesKnn(mapper.nodeId), mapper.k);

			if (knnNodeId >= nNodes)
				return;

			int cooPos = rptr[iRow];

			Tbx::Dual_quat_cu dqi, dqj;
			Tbx::Vec3 ri, ti, rj, tj;
			get_twist(mapper.nodeId, ri, ti);
			get_twist(knnNodeId, rj, tj);
			dqi.from_twist(ri, ti);
			dqj.from_twist(rj, tj);

			float4 nodeVwi = get_nodesVw(mapper.nodeId);
			float4 nodeVwj = get_nodesVw(knnNodeId);
			Tbx::Point3 vi(convert(read_float3_4(nodeVwi)));
			Tbx::Point3 vj(convert(read_float3_4(nodeVwj)));
			float alpha_ij = max(1.f / nodeVwi.w, 1.f / nodeVwj.w);
			float ww = sqrt(lambda * alpha_ij);

			// energy=============================================
			Tbx::Vec3 val_j = dqi.transform(Tbx::Point3(vj)) - dqj.transform(Tbx::Point3(vj));
			Tbx::Vec3 psi_val_j = reg_term_penalty(val_j);

			fptr[iRow + 0] = psi_val_j.x * ww;
			fptr[iRow + 1] = psi_val_j.y * ww;
			fptr[iRow + 2] = psi_val_j.z * ww;

#ifndef DEFINE_USE_HALF_GRAPH_EDGE
			Tbx::Vec3 val_i = dqj.transform(Tbx::Point3(vi)) - dqi.transform(Tbx::Point3(vi));
			Tbx::Vec3 psi_val_i = reg_term_penalty(val_i);
			fptr[iRow + 3] = psi_val_i.x * ww;
			fptr[iRow + 4] = psi_val_i.y * ww;
			fptr[iRow + 5] = psi_val_i.z * ww;
#endif

			// jacobi=============================================
			for (int ialpha = 0; ialpha < 3; ialpha++)
			{
				float inci = get_numeric_inc(ri[ialpha]);
				ri[ialpha] += inci;
				dqi.from_twist(ri, ti);
				Tbx::Vec3 val_j_inci = dqi.transform(Tbx::Point3(vj)) - dqj.transform(Tbx::Point3(vj));
				Tbx::Vec3 val_i_inci = dqj.transform(Tbx::Point3(vi)) - dqi.transform(Tbx::Point3(vi));
				ri[ialpha] -= inci;
				dqi.from_twist(ri, ti);

				float incj = get_numeric_inc(rj[ialpha]);
				rj[ialpha] += incj;
				dqj.from_twist(rj, tj);
				Tbx::Vec3 val_j_incj = dqi.transform(Tbx::Point3(vj)) - dqj.transform(Tbx::Point3(vj));
				Tbx::Vec3 val_i_incj = dqj.transform(Tbx::Point3(vi)) - dqi.transform(Tbx::Point3(vi));
				rj[ialpha] -= incj;
				dqj.from_twist(rj, tj);

				for (int ixyz = 0; ixyz < 3; ixyz++)
				{
					int pos = cooPos + ixyz*ColPerRow + ialpha;
					vptr[pos] = ww * (val_j_inci[ixyz] - val_j[ixyz]) / inci;
					vptr[pos + VarPerNode] = ww * (val_j_incj[ixyz] - val_j[ixyz]) / incj;
#ifndef DEFINE_USE_HALF_GRAPH_EDGE
					pos += 3 * ColPerRow;
					vptr[pos] = ww * (val_i_inci[ixyz] - val_i[ixyz]) / inci;
					vptr[pos + VarPerNode] = ww * (val_i_incj[ixyz] - val_i[ixyz]) / incj;
#endif
				}
			}// end for ialpha
			for (int ialpha = 0; ialpha < 3; ialpha++)
			{
				float inci = get_numeric_inc(ti[ialpha]);
				ti[ialpha] += inci;
				dqi.from_twist(ri, ti);
				Tbx::Vec3 val_j_inci = dqi.transform(Tbx::Point3(vj)) - dqj.transform(Tbx::Point3(vj));
				Tbx::Vec3 val_i_inci = dqj.transform(Tbx::Point3(vi)) - dqi.transform(Tbx::Point3(vi));
				ti[ialpha] -= inci;
				dqi.from_twist(ri, ti);

				float incj = get_numeric_inc(tj[ialpha]);
				tj[ialpha] += incj;
				dqj.from_twist(rj, tj);
				Tbx::Vec3 val_j_incj = dqi.transform(Tbx::Point3(vj)) - dqj.transform(Tbx::Point3(vj));
				Tbx::Vec3 val_i_incj = dqj.transform(Tbx::Point3(vi)) - dqi.transform(Tbx::Point3(vi));
				tj[ialpha] -= incj;
				dqj.from_twist(rj, tj);

				for (int ixyz = 0; ixyz < 3; ixyz++)
				{
					int pos = cooPos + ixyz*ColPerRow + ialpha + 3;
					vptr[pos] = ww * (val_j_inci[ixyz] - val_j[ixyz]) / inci;
					vptr[pos + VarPerNode] = ww * (val_j_incj[ixyz] - val_j[ixyz]) / incj;
#ifndef DEFINE_USE_HALF_GRAPH_EDGE
					pos += 3 * ColPerRow;
					vptr[pos] = ww * (val_i_inci[ixyz] - val_i[ixyz]) / inci;
					vptr[pos + VarPerNode] = ww * (val_i_incj[ixyz] - val_i[ixyz]) / incj;
#endif
				}
			}// end for ialpha
		}// end function ()

		__device__ __forceinline__ void calcTotalEnergy () const
		{
			const int iNode = threadIdx.x + blockIdx.x * blockDim.x;
			const int iRow = iNode * RowPerNode_RegTerm;

			if (iRow >= nRows)
				return;

			Mapper mapper = rows2nodeIds[iRow];
			int knnNodeId = knn_k(get_nodesKnn(mapper.nodeId), mapper.k);

			if (knnNodeId >= nNodes)
				return;

			Tbx::Dual_quat_cu dqi, dqj;
			Tbx::Vec3 ri, ti, rj, tj;
			get_twist(mapper.nodeId, ri, ti);
			get_twist(knnNodeId, rj, tj);
			dqi.from_twist(ri, ti);
			dqj.from_twist(rj, tj);

			float4 nodeVwi = get_nodesVw(mapper.nodeId);
			float4 nodeVwj = get_nodesVw(knnNodeId);
			Tbx::Point3 vi(convert(read_float3_4(nodeVwi)));
			Tbx::Point3 vj(convert(read_float3_4(nodeVwj)));
			float alpha_ij = max(1.f / nodeVwi.w, 1.f / nodeVwj.w);
			float ww2 = lambda * alpha_ij;

			// energy=============================================
			Tbx::Vec3 val = dqi.transform(Tbx::Point3(vj)) - dqj.transform(Tbx::Point3(vj));
			float eg = ww2 * reg_term_energy(val);
#ifndef DEFINE_USE_HALF_GRAPH_EDGE
			Tbx::Vec3 val1 = dqi.transform(Tbx::Point3(vi)) - dqj.transform(Tbx::Point3(vi));
			eg += ww2 * reg_term_energy(val1);
#endif

			//atomicAdd(totalEnergy, eg);
			totalEnergy[iNode] = eg;
		}
	};

	__global__ void calcRegTerm_kernel(RegTermJacobi rj)
	{
#ifdef CALC_REG_TERM_NUMERIC
		rj.calc_reg_numeric();
#else
		rj();
#endif
	}
	__global__ void calcRegTermTotalEnergy_kernel(RegTermJacobi rj)
	{
		rj.calcTotalEnergy();
	}

	void GpuGaussNewtonSolver::calcRegTerm()
	{
		if (m_Jrrows > 0)
		{
			CHECK_LE(1, m_Jrrows);
			RegTermJacobi rj;
			rj.cidx = m_Jr_ColIdx.ptr();
			rj.lambda = m_param->fusion_lambda;
			rj.nNodes = m_numNodes;
			rj.nRows = m_Jrrows;
			rj.psi_reg = m_param->fusion_psi_reg;
			rj.rows2nodeIds = m_Jr_RowMap2NodeId;
			rj.rptr = m_Jr_RowPtr.ptr();
			rj.vptr = m_Jr_val.ptr();
			rj.fptr = m_f_r.ptr();

			dim3 block(CTA_SIZE);
			dim3 grid(divUp(m_Jrrows / RowPerNode_RegTerm, block.x));

			calcRegTerm_kernel << <grid, block >> >(rj);
			cudaSafeCall(hipGetLastError(), "calcRegTerm_kernel");

			// 2. compute Jrt ==============================================
			// 2.1. fill (row, col) as (col, row) from Jr and sort.
			hipMemcpy(m_Jrt_RowPtr_coo.ptr(), m_Jr_ColIdx.ptr(), m_Jrnnzs*sizeof(int), hipMemcpyDeviceToDevice);
			hipMemcpy(m_Jrt_val.ptr(), m_Jr_val.ptr(), m_Jrnnzs*sizeof(float), hipMemcpyDeviceToDevice);
			modergpu_wrapper::mergesort_by_key(m_Jrt_RowPtr_coo.ptr(), m_Jrt_val.ptr(), m_Jrnnzs);
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::calcRegTerm::mergesort_by_key");
		}
	}
#pragma endregion

#pragma region --calc Hessian
#define ENABLE_GPU_DUMP_DEBUG_H
	__global__ void calcJr0tJr0_add_to_Hd_kernel(float* Hd, int nLv0Nodes, 
		const int* Jrt_rptr, float diag_eps)
	{
		enum
		{
			VarPerNode = GpuGaussNewtonSolver::VarPerNode,
			VarPerNode2 = VarPerNode*VarPerNode,
			LowerPartNum = GpuGaussNewtonSolver::LowerPartNum
		};

		int tid = threadIdx.x + blockIdx.x * blockDim.x;
		int iNode = tid / LowerPartNum;
		if (iNode >= nLv0Nodes)
			return;
		int eleLowerShift = tid - iNode*LowerPartNum;
		int rowShift = g_lower_2_rowShift_6x6[eleLowerShift];
		int colShift = g_lower_2_colShift_6x6[eleLowerShift];
		int row0 = iNode*VarPerNode;

		const int row0_begin = Jrt_rptr[row0 + rowShift];
		const int row_len = Jrt_rptr[row0 + rowShift + 1] - row0_begin;
		const int row1_begin = Jrt_rptr[row0 + colShift];

		float sum = diag_eps * (rowShift == colShift);
		for (int i = 0; i < row_len; i++)
			sum += get_JrtVal(row1_begin + i) * get_JrtVal(row0_begin + i);

		Hd[iNode * VarPerNode2 + rowShift*VarPerNode+colShift] += sum;
	}
	
	__global__ void fill_Hd_upper_kernel(float* Hd, int nLv0Nodes)
	{
		enum
		{
			VarPerNode = GpuGaussNewtonSolver::VarPerNode,
			VarPerNode2 = VarPerNode*VarPerNode,
			LowerPartNum = GpuGaussNewtonSolver::LowerPartNum
		};

		int tid = threadIdx.x + blockIdx.x * blockDim.x;
		int iNode = tid / LowerPartNum;
		if (iNode >= nLv0Nodes)
			return;
		int eleLowerShift = tid - iNode*LowerPartNum;
		int rowShift = g_lower_2_rowShift_6x6[eleLowerShift];
		int colShift = g_lower_2_colShift_6x6[eleLowerShift];
		
		Hd[iNode * VarPerNode2 + colShift * VarPerNode + rowShift] = 
			Hd[iNode * VarPerNode2 + rowShift * VarPerNode + colShift];
	}

	__global__ void calcB_kernel(
		float* B_val, const int* B_rptr_coo, const int* B_cidx, 
		int nBrows, int Bnnz, const int* Jrt_rptr)
	{
		enum{VarPerNode = GpuGaussNewtonSolver::VarPerNode};

		int tid = threadIdx.x + blockIdx.x * blockDim.x;
		if (tid >= Bnnz)
			return;

		int iBrow = B_rptr_coo[tid];
		int iBcol = B_cidx[tid];

		int Jr0t_cb = Jrt_rptr[iBrow];
		int Jr0t_ce = Jrt_rptr[iBrow + 1];

		int Jr1_rb = Jrt_rptr[iBcol + nBrows];
		int Jr1_re = Jrt_rptr[iBcol + nBrows + 1];

		float sum = 0.f;
		for (int i0 = Jr0t_cb, i1 = Jr1_rb; i0 < Jr0t_ce && i1 < Jr1_re; )
		{
			int Jr0t_c = get_JrtCidx(i0);
			int Jr1_r = get_JrtCidx(i1);
			if (Jr0t_c == Jr1_r)
			{
				for (int k = 0; k < VarPerNode; k++)
					sum += get_JrtVal(i0 + k) * get_JrtVal(i1 + k);
				i0 += VarPerNode;
				i1 += VarPerNode;
			}

			i0 += (Jr0t_c < Jr1_r) * VarPerNode;
			i1 += (Jr0t_c > Jr1_r) * VarPerNode;
		}// i

		B_val[tid] = sum;
	}

	__global__ void calcHr_kernel(float* Hr, const int* Jrt_rptr,
		int HrRowsCols, int nBrows, float diag_eps)
	{
		int tid = threadIdx.x + blockIdx.x * blockDim.x;

		if (tid >= (HrRowsCols + 1)*HrRowsCols / 2)
			return;

		// y is the triangular number
		int y = floor(-0.5 + sqrt(0.25 + 2 * tid));
		int triangularNumber = y * (y + 1) / 2;
		// x should <= y
		int x = tid - triangularNumber;

		int Jrt13_ib = Jrt_rptr[y + nBrows];
		int Jrt13_ie = Jrt_rptr[y + nBrows + 1];
		int Jrt13_jb = Jrt_rptr[x + nBrows];
		int Jrt13_je = Jrt_rptr[x + nBrows + 1];

		float sum = diag_eps * (x == y);
		for (int i = Jrt13_ib, j = Jrt13_jb; i < Jrt13_ie && j < Jrt13_je;)
		{
			int ci = get_JrtCidx(i);
			int cj = get_JrtCidx(j);
			if (ci == cj)
			{
				float s = 0.f;
				for (int k = 0; k < GpuGaussNewtonSolver::VarPerNode; k++)
					s += get_JrtVal(i + k) * get_JrtVal(j + k);
				sum += s;
				i += GpuGaussNewtonSolver::VarPerNode;
				j += GpuGaussNewtonSolver::VarPerNode;
			}

			i += (ci < cj) * GpuGaussNewtonSolver::VarPerNode;
			j += (ci > cj) * GpuGaussNewtonSolver::VarPerNode;
		}// i

		Hr[y*HrRowsCols + x] = Hr[x*HrRowsCols + y] = sum;
	}

	void GpuGaussNewtonSolver::calcHessian()
	{
		// 1. compute Jr0'Jr0 and accumulate into Hd
		if (m_Jrrows > 0)
		{
			dim3 block(CTA_SIZE);
			dim3 grid(divUp(m_numLv0Nodes*LowerPartNum, block.x));
			calcJr0tJr0_add_to_Hd_kernel << <grid, block >> >(m_Hd, m_numLv0Nodes, 
				m_Jrt_RowPtr.ptr(), m_param->fusion_GaussNewton_diag_regTerm);
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::calcHessian::calcJr0tJr0_add_to_Hd_kernel");
		}

		// 1.1 fill the upper tri part of Hd
		// previously, we only calculate the lower triangular pert of Hd;
		// now that the computation of Hd is ready, we fill the mission upper part
		if (m_numLv0Nodes > 0)
		{
			dim3 block(CTA_SIZE);
			dim3 grid(divUp(m_numLv0Nodes*LowerPartNum, block.x));
			fill_Hd_upper_kernel << <grid, block >> >(m_Hd, m_numLv0Nodes);
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::calcHessian::fill_Hd_upper_kernel");
		}

		// 2. compute B = Jr0'Jr1
		if (m_Bnnzs > 0)
		{
			dim3 block(CTA_SIZE);
			dim3 grid(divUp(m_Bnnzs, block.x));
			calcB_kernel << <grid, block >> >(m_B_val.ptr(), m_B_RowPtr_coo.ptr(), 
				m_B_ColIdx.ptr(), m_Brows, m_Bnnzs, m_Jrt_RowPtr.ptr());
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::calcHessian::calcB_kernel");
		}

		// 3. compute Bt
		if (m_Bnnzs > 0)
		{
			hipMemcpy(m_Bt_RowPtr_coo.ptr(), m_B_ColIdx.ptr(), m_Bnnzs*sizeof(int), hipMemcpyDeviceToDevice);
			hipMemcpy(m_Bt_val.ptr(), m_B_val.ptr(), m_Bnnzs*sizeof(float), hipMemcpyDeviceToDevice);
			modergpu_wrapper::mergesort_by_key(m_Bt_RowPtr_coo.ptr(), m_Bt_val.ptr(), m_Bnnzs);
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::calcHessian::mergesort_by_key");
		}

		// 4. compute Hr
		CHECK_LE(m_HrRowsCols*m_HrRowsCols, m_Hr.size());
		if (m_HrRowsCols > 0)
		{
			dim3 block(CTA_SIZE);
			dim3 grid(divUp(m_HrRowsCols*(m_HrRowsCols+1)/2, block.x));
			calcHr_kernel << <grid, block >> >(m_Hr.ptr(), m_Jrt_RowPtr.ptr(),
				m_HrRowsCols, m_Brows, m_param->fusion_GaussNewton_diag_regTerm);
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::calcHessian::calcHr_kernel");
		}

		// 5. compute g = -(g + Jr'*fr)
		if (m_Jrrows > 0)
		{
			float alpha = -1.f;
			float beta = -1.f;
			if (HIPSPARSE_STATUS_SUCCESS != hipsparseScsrmv(
				m_cuSparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m_Jrcols,
				m_Jrrows, m_Jrnnzs, &alpha, m_Jrt_desc, m_Jrt_val.ptr(), m_Jrt_RowPtr.ptr(),
				m_Jrt_ColIdx.ptr(), m_f_r.ptr(), &beta, m_g.ptr()))
				throw std::exception("GpuGaussNewtonSolver::calcHessian::hipsparseScsrmv failed!\n");
		}
	}
#pragma endregion

#pragma region --block solve
	__global__ void calcBtLtinv_kernel(float* BtLtinv, const int* Bt_rptr, 
		const int* Bt_rptr_coo, int nLv0Nodes, int nnz)
	{
		enum{ VarPerNode = GpuGaussNewtonSolver::VarPerNode };
		int tid = threadIdx.x + blockIdx.x * blockDim.x;
		if (tid >= nnz)
			return;

		int row = Bt_rptr_coo[tid];
		int col = get_BtLtinvCidx(tid);
		int iNodeCol = col / VarPerNode;
		int cshift = col - iNodeCol * VarPerNode;

		float sum = 0.f;
		int Hd_row_b = iNodeCol * VarPerNode;
		int Bt_b = Bt_rptr[row] / VarPerNode;
		int Bt_e = Bt_rptr[row + 1] / VarPerNode;
		int Bt_col_b = -1;

		// binary search Hd_row_b in the range col of [Bt_b, Bt_e]
		while (Bt_b < Bt_e)
		{
			int imid = ((Bt_b + Bt_e) >> 1);
			Bt_col_b = get_BtCidx(imid*VarPerNode);
			if (Bt_col_b < Hd_row_b)
				Bt_b = imid + 1;
			else
				Bt_e = imid;
		}
		Bt_b *= VarPerNode;
		Bt_e *= VarPerNode;
		Bt_col_b = get_BtCidx(Bt_b);
		if (Bt_col_b == Hd_row_b && Bt_b == Bt_e)
		{
			Hd_row_b = (Hd_row_b + cshift) * VarPerNode;
			for (int k = 0; k <= cshift; k++)
				sum += get_BtVal(Bt_b + k) * get_HdLinv(Hd_row_b + k);
		}

		// write the result
		BtLtinv[tid] = sum;
	}

	__global__ void calcQ_kernel(float* Q, const float* Hr,
		const int* Bt_rptr, int HrRowsCols, int nBrows)
	{
		int tid = threadIdx.x + blockIdx.x * blockDim.x;

		if (tid >= (HrRowsCols + 1)*HrRowsCols / 2)
			return;

		// y is the triangular number
		int y = floor(-0.5 + sqrt(0.25 + 2 * tid));
		int triangularNumber = y * (y + 1) / 2;
		// x should <= y
		int x = tid - triangularNumber;

		int Bt_ib = Bt_rptr[y];
		int Bt_ie = Bt_rptr[y + 1];
		int Bt_jb = Bt_rptr[x];
		int Bt_je = Bt_rptr[x + 1];

		float sum = 0.f;
		for (int i = Bt_ib, j = Bt_jb; i < Bt_ie && j < Bt_je;)
		{
			int ci = get_BtLtinvCidx(i);
			int cj = get_BtLtinvCidx(j);
			if (ci == cj)
			{
				float s = 0.f;
				for (int k = 0; k < GpuGaussNewtonSolver::VarPerNode; k++)
					s += get_BtLtinvVal(i + k) * get_BtLtinvVal(j + k);
				sum += s;
				i += GpuGaussNewtonSolver::VarPerNode;
				j += GpuGaussNewtonSolver::VarPerNode;
			}

			i += (ci < cj) * GpuGaussNewtonSolver::VarPerNode;
			j += (ci > cj) * GpuGaussNewtonSolver::VarPerNode;
		}// i

		Q[y*HrRowsCols + x] = Q[x*HrRowsCols + y] = Hr[y*HrRowsCols + x] - sum;
	}

	// vec_out = alpha * Linv * vec_in + beta * vec_out
	__global__ void calc_Hd_Linv_x_vec_kernel(float* vec_out, const float* vec_in, int nRows,
		float alpha = 1.f, float beta = 0.f)
	{
		int iRow = threadIdx.x + blockIdx.x*blockDim.x;
		if (iRow >= nRows)
			return;
		int iNode = iRow / GpuGaussNewtonSolver::VarPerNode;
		int rshift = iRow - iNode * GpuGaussNewtonSolver::VarPerNode;
		int iPos = iRow * GpuGaussNewtonSolver::VarPerNode + rshift;

		float sum = 0.f;
		for (int k = -rshift; k <= 0; k++)
			sum += get_HdLinv(iPos + k) * vec_in[iRow + k];

		vec_out[iRow] = alpha * sum + beta * vec_out[iRow];
	}

	// vec_out = alpha * Ltinv * vec_in + beta * vec_out
	__global__ void calc_Hd_Ltinv_x_vec_kernel(float* vec_out, const float* vec_in, int nRows,
		float alpha = 1.f, float beta = 0.f)
	{
		int iRow = threadIdx.x + blockIdx.x*blockDim.x;
		if (iRow >= nRows)
			return;
		int iNode = iRow / GpuGaussNewtonSolver::VarPerNode;
		int rshift = iRow - iNode * GpuGaussNewtonSolver::VarPerNode;
		int iPos = iRow * GpuGaussNewtonSolver::VarPerNode + rshift;

		float sum = 0.f;
		for (int k = 0; k < GpuGaussNewtonSolver::VarPerNode-rshift; k++)
			sum += get_HdLinv(iPos + k*GpuGaussNewtonSolver::VarPerNode) * vec_in[iRow + k];

		vec_out[iRow] = alpha * sum + beta * vec_out[iRow];
	}

	void GpuGaussNewtonSolver::blockSolve()
	{
		hipsparseStatus_t cuSparseStatus;
		CHECK_LE(m_numLv0Nodes*VarPerNode*VarPerNode, m_Hd_Linv.size());
		CHECK_LE(m_numLv0Nodes*VarPerNode*VarPerNode, m_Hd_LLtinv.size());

		// 1. batch LLt the diag blocks Hd==================================================

		// 1.0. copy Hd to Linv buffer
		cudaSafeCall(hipMemcpy(m_Hd_Linv.ptr(), m_Hd.ptr(), m_numLv0Nodes*VarPerNode
			*VarPerNode*m_Hd.elem_size, hipMemcpyDeviceToDevice), 
			"GpuGaussNewtonSolver::blockSolve::copy Hd to Hd_L");

		// 1.1 Hd = L*L'
		gpu_cholesky::single_thread_cholesky_batched(m_Hd_Linv.ptr(), VarPerNode,
			VarPerNode*VarPerNode, m_numLv0Nodes);
		checkNan(m_Hd_Linv, m_numLv0Nodes*VarPerNode*VarPerNode, "Hd_L");

		// 1.2 inv(L)
		gpu_cholesky::single_thread_tril_inv_batched(m_Hd_Linv.ptr(), VarPerNode,
			VarPerNode*VarPerNode, m_numLv0Nodes);
		checkNan(m_Hd_Linv, m_numLv0Nodes*VarPerNode*VarPerNode, "Hd_Linv");

		// 1.3 inv(L*L') = inv(L')*inv(L) = inv(L)'*inv(L)
		gpu_cholesky::single_thread_LtL_batched(
			m_Hd_LLtinv.ptr(), VarPerNode*VarPerNode, m_Hd_Linv.ptr(), 
			VarPerNode*VarPerNode, VarPerNode, m_numLv0Nodes);
		checkNan(m_Hd_LLtinv, m_numLv0Nodes*VarPerNode*VarPerNode, "Hd_LLtinv");

		// 2. compute Q = Hr - Bt * inv(Hd) * B ======================================
		CHECK_LE(m_HrRowsCols*m_HrRowsCols, m_Q.size());
		// 2.1 compute Bt*Ltinv
		if (m_HrRowsCols > 0)
		{
			dim3 block(CTA_SIZE);
			dim3 grid(divUp(m_Bnnzs, block.x));
			calcBtLtinv_kernel << <grid, block >> >(m_Bt_Ltinv_val.ptr(),
				m_Bt_RowPtr.ptr(), m_Bt_RowPtr_coo.ptr(), m_numLv0Nodes, m_Bnnzs);
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::calcHessian::calcBtLtinv_kernel");
		}

		// 2.2 compute Q
		if (m_HrRowsCols > 0)
		{
			dim3 block(CTA_SIZE);
			dim3 grid(divUp(m_HrRowsCols*(m_HrRowsCols+1)/2, block.x));
			calcQ_kernel << <grid, block >> >(m_Q.ptr(), m_Hr.ptr(), m_Bt_RowPtr.ptr(),
				m_HrRowsCols, m_Brows);
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::calcHessian::calcQ_kernel");

			// kept Q before factorize, for debug
			if (m_Q_kept.size() == m_Q.size())
				hipMemcpy(m_Q_kept.ptr(), m_Q.ptr(),
					m_HrRowsCols*m_HrRowsCols*m_Q.elem_size,
					hipMemcpyDeviceToDevice);
			checkNan(m_Q, m_HrRowsCols*m_HrRowsCols, "Q");
		}

		// 3. llt decompostion of Q ==================================================
		// 3.1 decide the working space of the solver
		if (m_HrRowsCols > 0)
		{
			int lwork = 0;
			hipsolverDnSpotrf_bufferSize(m_cuSolverHandle, HIPBLAS_FILL_MODE_LOWER,
				m_HrRowsCols, m_Q.ptr(), m_HrRowsCols, &lwork);
			if (lwork > m_cuSolverWorkSpace.size())
			{
				// we store dev info in the last element
				m_cuSolverWorkSpace.create(lwork * 1.5 + 1);
				printf("hipsolverDnSpotrf_bufferSize: %d\n", lwork);
			}

			// 3.2 Cholesky decomposition
			// before this step, m_Q is calculated as filled as symmetric matrix
			// note that cublas uses column majored storage, thus after this step
			// the matrix m_Q should be viewed as column-majored matrix
			hipsolverStatus_t fst = hipsolverDnSpotrf(m_cuSolverHandle, HIPBLAS_FILL_MODE_LOWER, m_HrRowsCols,
				m_Q.ptr(), m_HrRowsCols, m_cuSolverWorkSpace.ptr(), lwork,
				(int*)m_cuSolverWorkSpace.ptr() + m_cuSolverWorkSpace.size() - 1);
			if (HIPSOLVER_STATUS_SUCCESS != fst)
			{
				printf("hipsolverDnSpotrf failed: status: %d\n", fst);
				throw std::exception();
			}
			checkNan(m_Q, m_HrRowsCols*m_HrRowsCols, "Q1");
		}
		// 4. solve H*h = g =============================================================
		const int sz = m_Jrcols;
		const int sz0 = m_Brows;
		const int sz1 = sz - sz0;
		CHECK_LE(sz, m_u.size());
		CHECK_LE(sz, m_h.size());
		CHECK_LE(sz, m_g.size());
		CHECK_LE(sz, m_tmpvec.size());

		// 4.1 let H = LL', first we solve for L*u=g;
		// 4.1.1 u(0:sz0-1) = HdLinv*g(0:sz0-1)
		if (sz0 > 0)
		{
			dim3 block(CTA_SIZE);
			dim3 grid(divUp(sz0, block.x));
			calc_Hd_Linv_x_vec_kernel << <grid, block >> >(m_u.ptr(),
				m_g.ptr(), sz0);
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::calcHessian::calc_Hd_Linv_x_vec_kernel");
		}
	
		// 4.1.2 u(sz0:sz-1) = LQinv*(g(sz0:sz-1) - Bt*HdLtinv*HdLinv*g(0:sz0-1))
		if (sz1 > 0)
		{
			// tmpvec = HdLtinv*HdLinv*g(0:sz0-1)
			dim3 block(CTA_SIZE);
			dim3 grid(divUp(sz0, block.x));
			calc_Hd_Ltinv_x_vec_kernel << <grid, block >> >(m_tmpvec.ptr(),
				m_u.ptr(), sz0);
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::calcHessian::calc_Hd_Ltinv_x_vec_kernel");

			// u(sz0:sz-1) = g(sz0:sz-1) - Bt*tmpvec
			{
				float alpha = -1.f;
				float beta = 1.f;
				hipMemcpy(m_u.ptr() + sz0, m_g.ptr() + sz0, sz1*sizeof(float), hipMemcpyDeviceToDevice);
				cuSparseStatus = hipsparseScsrmv(m_cuSparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, sz1, sz0,
					m_Bnnzs, &alpha, m_Bt_desc, m_Bt_val.ptr(), m_Bt_RowPtr.ptr(),
					m_Bt_ColIdx.ptr(), m_tmpvec.ptr(), &beta, m_u.ptr() + sz0);
				if (cuSparseStatus != HIPSPARSE_STATUS_SUCCESS)
					printf("cuSparse error1: %d\n", cuSparseStatus);

				// solve LQ*u(sz0:sz-1) = u(sz0:sz-1)
				// note cublas use column majored matrix, we assume m_Q is column majored in this step
				hipblasStrsv(m_cublasHandle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
					sz1, m_Q.ptr(), sz1, m_u.ptr() + sz0, 1);
			}
		}
		checkNan(m_u, sz, "u");
		
		// 4.2 then we solve for L'*h=u;
		// 4.2.1 h(sz0:sz-1) = UQinv*u(sz0:sz-1)
		if (sz1 > 0)
		{
			hipMemcpy(m_h.ptr() + sz0, m_u.ptr() + sz0, sz1*sizeof(float), hipMemcpyDeviceToDevice);
			hipblasStrsv(m_cublasHandle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT,
				sz1, m_Q.ptr(), sz1, m_h.ptr() + sz0, 1);
		}
		
		// 4.2.2 h(0:sz0-1) = HdLtinv*( u(0:sz0-1) - HdLinv*B*h(sz0:sz-1) )
		// tmpvec = B*h(sz0:sz-1)
		if (sz1 > 0)
		{
			float alpha = 1.f;
			float beta = 0.f;
			cuSparseStatus = hipsparseScsrmv(m_cuSparseHandle,
				HIPSPARSE_OPERATION_NON_TRANSPOSE, sz0, sz1,
				m_Bnnzs, &alpha, m_B_desc, m_B_val.ptr(), m_B_RowPtr.ptr(),
				m_B_ColIdx.ptr(), m_h.ptr() + sz0, &beta, m_tmpvec.ptr());
			if (cuSparseStatus != HIPSPARSE_STATUS_SUCCESS)
				printf("cuSparse error2: %d\n", cuSparseStatus);
		}

		// u(0:sz0-1) = u(0:sz0-1) - HdLinv * tmpvec
		// h(0:sz0-1) = HdLtinv*u(0:sz0-1)
		if (sz0 > 0)
		{
			dim3 block(CTA_SIZE);
			dim3 grid(divUp(sz0, block.x));
			if (sz1 > 0)
			{
				calc_Hd_Linv_x_vec_kernel << <grid, block >> >(m_u.ptr(),
					m_tmpvec.ptr(), sz0, -1.f, 1.f);
				cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::calcHessian::calc_Hd_Linv_x_vec_kernel");
				calc_Hd_Ltinv_x_vec_kernel << <grid, block >> >(m_h.ptr(),
					m_u.ptr(), sz0);
				cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::calcHessian::calc_Hd_Ltinv_x_vec_kernel");
			}
			else
			{
				calc_Hd_Ltinv_x_vec_kernel << <grid, block >> >(m_h.ptr(),
					m_u.ptr(), sz0, -1.f);
				cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::calcHessian::calc_Hd_Ltinv_x_vec_kernel");
			}
		}
	}

	float GpuGaussNewtonSolver::calcTotalEnergy(float& data_energy, float& reg_energy)
	{
		float total_energy = 0.f;
		hipMemset(m_energy_vec.ptr(), 0, m_energy_vec.sizeBytes());
		{
			DataTermCombined cs;
			cs.angleThres = m_param->fusion_nonRigid_angleThreSin;
			cs.distThres = m_param->fusion_nonRigid_distThre;
			cs.Hd_ = m_Hd;
			cs.g_ = m_g;
			cs.imgHeight = m_vmap_cano->rows();
			cs.imgWidth = m_vmap_cano->cols();
			cs.intr = m_intr;
			cs.nmap_cano = *m_nmap_cano;
			cs.nmap_live = *m_nmap_live;
			cs.nmap_warp = *m_nmap_warp;
			cs.vmap_cano = *m_vmap_cano;
			cs.vmap_live = *m_vmap_live;
			cs.vmap_warp = *m_vmap_warp;
			cs.vmapKnn = m_vmapKnn;
			cs.nNodes = m_numNodes;
			cs.Tlw = m_pWarpField->get_rigidTransform();
			cs.Tlw_inv = m_pWarpField->get_rigidTransform().fast_invert();
			cs.psi_data = m_param->fusion_psi_data;
			cs.totalEnergy = m_energy_vec.ptr();

			//int zero_mem_symbol = 0;
			//hipMemcpyToSymbol(HIP_SYMBOL(g_totalEnergy), &zero_mem_symbol, sizeof(int));
			//hipMemset(&m_tmpvec[0], 0, sizeof(float));

			// 1. data term
			//////////////////////////////
			dim3 block(CTA_SIZE_X, CTA_SIZE_Y);
			dim3 grid(1, 1, 1);
			grid.x = divUp(cs.imgWidth, block.x);
			grid.y = divUp(cs.imgHeight, block.y);
			int maxK = min(WarpField::KnnK, m_pWarpField->getNumNodesInLevel(0));
			switch (maxK)
			{
			case 1:
				calcDataTermTotalEnergyKernel<1> << <grid, block >> >(cs);
				break;
			case 2:
				calcDataTermTotalEnergyKernel<2> << <grid, block >> >(cs);
				break;
			case 3:
				calcDataTermTotalEnergyKernel<3> << <grid, block >> >(cs);
				break;
			case 4:
				calcDataTermTotalEnergyKernel<4> << <grid, block >> >(cs);
				break;
			default:
				throw std::exception("non supported KnnK in data term energy!");
			}
			cudaSafeCall(hipGetLastError(), "calcDataTermTotalEnergyKernel");
		}

		if (m_Jrrows > 0)
		{
			RegTermJacobi rj;
			rj.cidx = m_Jr_ColIdx.ptr();
			rj.lambda = m_param->fusion_lambda;
			rj.nNodes = m_numNodes;
			rj.nRows = m_Jrrows;
			rj.psi_reg = m_param->fusion_psi_reg;
			rj.rows2nodeIds = m_Jr_RowMap2NodeId;
			rj.rptr = m_Jr_RowPtr.ptr();
			rj.vptr = m_Jr_val.ptr();
			rj.fptr = m_f_r.ptr();
			rj.totalEnergy = m_energy_vec.ptr() + m_vmapKnn.rows()*m_vmapKnn.cols();

			dim3 block(CTA_SIZE);
			dim3 grid(divUp(m_Jrrows / RowPerNode_RegTerm, block.x));

			calcRegTermTotalEnergy_kernel << <grid, block >> >(rj);
			cudaSafeCall(hipGetLastError(), "calcRegTermTotalEnergy_kernel");
		}

		//cudaSafeCall(hipMemcpy(&total_energy,
		//	&m_tmpvec[0], sizeof(float), hipMemcpyDeviceToHost), "copy reg totalEnergy to host");
		hipblasStatus_t st = hipblasSasum(m_cublasHandle, m_Jrrows / RowPerNode_RegTerm + 
			m_vmapKnn.rows()*m_vmapKnn.cols(),
			m_energy_vec.ptr(), 1, &total_energy);
		if (st != HIPBLAS_STATUS_SUCCESS)
			throw std::exception("cublass error, in hipblasSnrm2");

		// debug get both data and reg term energy
#if 1
		reg_energy = 0.f;
		if (m_Jrrows > 0)
		{
			hipblasSasum(m_cublasHandle, m_Jrrows / RowPerNode_RegTerm,
				m_energy_vec.ptr() + m_vmapKnn.rows()*m_vmapKnn.cols(),
				1, &reg_energy);
		}
		data_energy = total_energy - reg_energy;
#endif

		return total_energy;
	}
#pragma endregion

#pragma region --update twist

	__global__ void updateTwist_inch_kernel(float* twist, const float* h, float step, int nNodes)
	{
		int i = threadIdx.x + blockIdx.x*blockDim.x;
		if (i < nNodes)
		{
			int i6 = i * 6;
			Tbx::Vec3 r(twist[i6] + step*h[i6], twist[i6 + 1] + step*h[i6 + 1], twist[i6 + 2] + step*h[i6 + 2]);
			Tbx::Vec3 t(twist[i6+3] + step*h[i6+3], twist[i6 + 4] + step*h[i6 + 4], twist[i6 + 5] + step*h[i6 + 5]);
			Tbx::Dual_quat_cu dq;
			dq.from_twist(r, t);
			dq.to_twist(r, t);
			twist[i6] = r[0];
			twist[i6 + 1] = r[1];
			twist[i6 + 2] = r[2];
			twist[i6 + 3] = t[0];
			twist[i6 + 4] = t[1];
			twist[i6 + 5] = t[2];
		}
	}

	void GpuGaussNewtonSolver::updateTwist_inch(const float* h, float step)
	{
		dim3 block(CTA_SIZE);
		dim3 grid(divUp(m_numNodes, block.x));
		updateTwist_inch_kernel << <grid, block >> >(m_twist.ptr(), h, step, m_numNodes);
		cudaSafeCall(hipGetLastError(), "updateTwist_inch_kernel");
	}
#pragma endregion

#pragma region --factor out rigid

	__device__ float _g_common_q[8];

	template<int CTA_SIZE_, typename T>
	static __device__ __forceinline__ void reduce(volatile T* buffer)
	{
		int tid = Block::flattenedThreadId();
		T val = buffer[tid];

		if (CTA_SIZE_ >= 1024) { if (tid < 512) buffer[tid] = val = val + buffer[tid + 512]; __syncthreads(); }
		if (CTA_SIZE_ >= 512) { if (tid < 256) buffer[tid] = val = val + buffer[tid + 256]; __syncthreads(); }
		if (CTA_SIZE_ >= 256) { if (tid < 128) buffer[tid] = val = val + buffer[tid + 128]; __syncthreads(); }
		if (CTA_SIZE_ >= 128) { if (tid <  64) buffer[tid] = val = val + buffer[tid + 64]; __syncthreads(); }

		if (tid < 32){
			if (CTA_SIZE_ >= 64) { buffer[tid] = val = val + buffer[tid + 32]; }
			if (CTA_SIZE_ >= 32) { buffer[tid] = val = val + buffer[tid + 16]; }
			if (CTA_SIZE_ >= 16) { buffer[tid] = val = val + buffer[tid + 8]; }
			if (CTA_SIZE_ >= 8) { buffer[tid] = val = val + buffer[tid + 4]; }
			if (CTA_SIZE_ >= 4) { buffer[tid] = val = val + buffer[tid + 2]; }
			if (CTA_SIZE_ >= 2) { buffer[tid] = val = val + buffer[tid + 1]; }
		}
	}

	__global__ void reduce_all_nodes_kernel(const float4* nodesDqVw, int n)
	{
		const float* beg = (const float*)nodesDqVw + blockIdx.x;
		float sum = 0.f;
		for (int i = threadIdx.x; i < n; i += blockDim.x)
			sum += beg[i * 12]; // dq+vw, 12 float per node

		__shared__ float smem[GpuGaussNewtonSolver::CTA_SIZE];

		smem[threadIdx.x] = sum;
		__syncthreads();

		reduce<GpuGaussNewtonSolver::CTA_SIZE>(smem);

		if (threadIdx.x == 0)
			_g_common_q[blockIdx.x] = smem[0];
	}


	__global__ void factor_all_nodes_kernel(float4* nodesDqVw, int n, Tbx::Dual_quat_cu rigid_inv)
	{
		int i = threadIdx.x + blockIdx.x * blockDim.x;
		if (i >= n)
			return;
		
		Tbx::Dual_quat_cu dq = rigid_inv * pack_dual_quat(nodesDqVw[3 * i], nodesDqVw[3 * i + 1]);
		unpack_dual_quat(dq, nodesDqVw[3 * i], nodesDqVw[3 * i + 1]);
	}

	// optional, factor out common rigid transformations among all nodes
	void GpuGaussNewtonSolver::factor_out_rigid()
	{
		if (m_pWarpField == nullptr)
			throw std::exception("GpuGaussNewtonSolver::solve: null pointer");
		if (m_pWarpField->getNumLevels() < 2)
			throw std::exception("non-supported levels of warp field!");
		if (m_pWarpField->getNumNodesInLevel(0) == 0)
		{
			printf("no warp nodes, return\n");
			return;
		}
		const int num0 = m_pWarpField->getNumNodesInLevel(0);
		const int numAll = m_pWarpField->getNumAllNodes();

		Tbx::Dual_quat_cu dq(Tbx::Quat_cu(0,0,0,0), Tbx::Quat_cu(0,0,0,0));
		hipMemcpyToSymbol(HIP_SYMBOL(_g_common_q), &dq, sizeof(Tbx::Dual_quat_cu));

		reduce_all_nodes_kernel << <8, GpuGaussNewtonSolver::CTA_SIZE >> >(
			m_pWarpField->getNodesDqVwPtr(0), num0);
		cudaSafeCall(hipGetLastError(), "reduce_all_nodes_kernel");
		hipMemcpyFromSymbol(&dq, HIP_SYMBOL(_g_common_q), sizeof(Tbx::Dual_quat_cu));

		if (dq.get_non_dual_part().norm() > Tbx::Dual_quat_cu::epsilon())
		{
			dq.normalize();
			m_pWarpField->set_rigidTransform(
				m_pWarpField->get_rigidTransform() * dq.to_transformation());

			for (int lv = 0; lv < m_pWarpField->getNumLevels(); lv++)
			{
				int numLv = m_pWarpField->getNumNodesInLevel(lv);
				if (numLv == 0)
					break;
				factor_all_nodes_kernel << <divUp(numLv, GpuGaussNewtonSolver::CTA_SIZE),
					GpuGaussNewtonSolver::CTA_SIZE >> >(m_pWarpField->getNodesDqVwPtr(lv), numLv, dq.conjugate());
			}
			cudaSafeCall(hipGetLastError(), "factor_all_nodes_kernel");


			// re-extract info
			m_pWarpField->extract_nodes_info_no_allocation(m_nodesKnn, m_twist, m_nodesVw);
			checkNan(m_twist, numAll * 6, "twist after factoring rigid");
		}
	}
#pragma endregion

#pragma region --check linear solver
	void GpuGaussNewtonSolver::checkLinearSolver()
	{

	}
#pragma endregion
}