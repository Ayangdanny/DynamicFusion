#include "hip/hip_runtime.h"

#include <assert.h>
#include "MergeSort.h"

namespace dfusion
{
#define SHARED_SIZE_LIMIT 1024U
#define     SAMPLE_STRIDE 128
	typedef unsigned int uint;
#define MAX_SAMPLE_COUNT 32768
#define W (sizeof(uint) * 8)



	static inline __host__ __device__ uint iDivUp(uint a, uint b)
	{
		return ((a % b) == 0) ? (a / b) : (a / b + 1);
	}

	static inline __host__ __device__ uint getSampleCount(uint dividend)
	{
		return iDivUp(dividend, SAMPLE_STRIDE);
	}

	static inline __device__ uint nextPowerOfTwo(uint x)
	{
		/*
		--x;
		x |= x >> 1;
		x |= x >> 2;
		x |= x >> 4;
		x |= x >> 8;
		x |= x >> 16;
		return ++x;
		*/
		return 1U << (W - __clz(x - 1));
	}

#pragma region MergeSort
	template<class KeyType, uint sortDir> static inline __device__ uint binarySearchInclusive(
		KeyType val, KeyType *data, uint L, uint stride)
	{
		if (L == 0)
		{
			return 0;
		}

		uint pos = 0;

		for (; stride > 0; stride >>= 1)
		{
			uint newPos = min(pos + stride, L);

			if ((sortDir && (data[newPos - 1] <= val)) || (!sortDir && (data[newPos - 1] >= val)))
			{
				pos = newPos;
			}
		}

		return pos;
	}

	template<class KeyType, uint sortDir> static inline __device__ uint binarySearchExclusive(
		KeyType val, KeyType *data, uint L, uint stride)
	{
		if (L == 0)
		{
			return 0;
		}

		uint pos = 0;

		for (; stride > 0; stride >>= 1)
		{
			uint newPos = min(pos + stride, L);

			if ((sortDir && (data[newPos - 1] < val)) || (!sortDir && (data[newPos - 1] > val)))
			{
				pos = newPos;
			}
		}

		return pos;
	}

	template<class KeyType, class ValueType, uint sortDir> __global__ void mergeSortSharedKernel(
		KeyType *d_DstKey,
		ValueType *d_DstVal,
		KeyType *d_SrcKey,
		ValueType *d_SrcVal,
		uint arrayLength
		)
	{
		__shared__ KeyType s_key[SHARED_SIZE_LIMIT];
		__shared__ ValueType s_val[SHARED_SIZE_LIMIT];

		d_SrcKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
		d_SrcVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
		d_DstKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
		d_DstVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
		s_key[threadIdx.x + 0] = d_SrcKey[0];
		s_val[threadIdx.x + 0] = d_SrcVal[0];
		s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcKey[(SHARED_SIZE_LIMIT / 2)];
		s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcVal[(SHARED_SIZE_LIMIT / 2)];

		for (uint stride = 1; stride < arrayLength; stride <<= 1)
		{
			uint     lPos = threadIdx.x & (stride - 1);
			KeyType *baseKey = s_key + 2 * (threadIdx.x - lPos);
			ValueType *baseVal = s_val + 2 * (threadIdx.x - lPos);

			__syncthreads();
			KeyType keyA = baseKey[lPos + 0];
			ValueType valA = baseVal[lPos + 0];
			KeyType keyB = baseKey[lPos + stride];
			ValueType valB = baseVal[lPos + stride];
			uint posA = binarySearchExclusive<KeyType, sortDir>(keyA, baseKey + stride, stride, stride) + lPos;
			uint posB = binarySearchInclusive<KeyType, sortDir>(keyB, baseKey + 0, stride, stride) + lPos;

			__syncthreads();
			baseKey[posA] = keyA;
			baseVal[posA] = valA;
			baseKey[posB] = keyB;
			baseVal[posB] = valB;
		}

		__syncthreads();
		d_DstKey[0] = s_key[threadIdx.x + 0];
		d_DstVal[0] = s_val[threadIdx.x + 0];
		d_DstKey[(SHARED_SIZE_LIMIT / 2)] = s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
		d_DstVal[(SHARED_SIZE_LIMIT / 2)] = s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
	}

	template<class KeyType, uint sortDir> __global__ void generateSampleRanksKernel(
		uint *d_RanksA,
		uint *d_RanksB,
		KeyType *d_SrcKey,
		uint stride,
		uint N,
		uint threadCount
		)
	{
		uint pos = blockIdx.x * blockDim.x + threadIdx.x;

		if (pos >= threadCount)
		{
			return;
		}

		const uint i = pos & ((stride / SAMPLE_STRIDE) - 1);
		const uint segmentBase = (pos - i) * (2 * SAMPLE_STRIDE);
		d_SrcKey += segmentBase;
		d_RanksA += segmentBase / SAMPLE_STRIDE;
		d_RanksB += segmentBase / SAMPLE_STRIDE;

		const uint segmentElementsA = stride;
		const uint segmentElementsB = min(stride, N - segmentBase - stride);
		const uint segmentSamplesA = getSampleCount(segmentElementsA);
		const uint segmentSamplesB = getSampleCount(segmentElementsB);

		if (i < segmentSamplesA)
		{
			d_RanksA[i] = i * SAMPLE_STRIDE;
			d_RanksB[i] = binarySearchExclusive<KeyType, sortDir>(
				d_SrcKey[i * SAMPLE_STRIDE], d_SrcKey + stride,
				segmentElementsB, nextPowerOfTwo(segmentElementsB)
				);
		}

		if (i < segmentSamplesB)
		{
			d_RanksB[(stride / SAMPLE_STRIDE) + i] = i * SAMPLE_STRIDE;
			d_RanksA[(stride / SAMPLE_STRIDE) + i] = binarySearchInclusive<KeyType, sortDir>(
				d_SrcKey[stride + i * SAMPLE_STRIDE], d_SrcKey + 0,
				segmentElementsA, nextPowerOfTwo(segmentElementsA)
				);
		}
	}


	__global__ void mergeRanksAndIndicesKernel(
		uint *d_Limits,
		uint *d_Ranks,
		uint stride,
		uint N,
		uint threadCount
		)
	{
		uint pos = blockIdx.x * blockDim.x + threadIdx.x;

		if (pos >= threadCount)
		{
			return;
		}

		const uint i = pos & ((stride / SAMPLE_STRIDE) - 1);
		const uint segmentBase = (pos - i) * (2 * SAMPLE_STRIDE);
		d_Ranks += (pos - i) * 2;
		d_Limits += (pos - i) * 2;

		const uint segmentElementsA = stride;
		const uint segmentElementsB = min(stride, N - segmentBase - stride);
		const uint  segmentSamplesA = getSampleCount(segmentElementsA);
		const uint  segmentSamplesB = getSampleCount(segmentElementsB);

		if (i < segmentSamplesA)
		{
			uint dstPos = binarySearchExclusive<uint, 1U>(d_Ranks[i], d_Ranks + segmentSamplesA,
				segmentSamplesB, nextPowerOfTwo(segmentSamplesB)) + i;
			d_Limits[dstPos] = d_Ranks[i];
		}

		if (i < segmentSamplesB)
		{
			uint dstPos = binarySearchInclusive<uint, 1U>(d_Ranks[segmentSamplesA + i], d_Ranks,
				segmentSamplesA, nextPowerOfTwo(segmentSamplesA)) + i;
			d_Limits[dstPos] = d_Ranks[segmentSamplesA + i];
		}
	}


	template<class KeyType, class ValueType, uint sortDir> inline __device__ void merge(
		KeyType *dstKey,
		ValueType *dstVal,
		KeyType *srcAKey,
		ValueType *srcAVal,
		KeyType *srcBKey,
		ValueType *srcBVal,
		uint lenA,
		uint nPowTwoLenA,
		uint lenB,
		uint nPowTwoLenB
		)
	{
		KeyType keyA, keyB;
		ValueType valA, valB;
		uint dstPosA, dstPosB;

		if (threadIdx.x < lenA)
		{
			keyA = srcAKey[threadIdx.x];
			valA = srcAVal[threadIdx.x];
			dstPosA = binarySearchExclusive<KeyType, sortDir>(keyA, srcBKey, lenB, nPowTwoLenB) + threadIdx.x;
		}

		if (threadIdx.x < lenB)
		{
			keyB = srcBKey[threadIdx.x];
			valB = srcBVal[threadIdx.x];
			dstPosB = binarySearchInclusive<KeyType, sortDir>(keyB, srcAKey, lenA, nPowTwoLenA) + threadIdx.x;
		}

		__syncthreads();

		if (threadIdx.x < lenA)
		{
			dstKey[dstPosA] = keyA;
			dstVal[dstPosA] = valA;
		}

		if (threadIdx.x < lenB)
		{
			dstKey[dstPosB] = keyB;
			dstVal[dstPosB] = valB;
		}
	}

	template<class KeyType, class ValueType, uint sortDir> __global__ void mergeElementaryIntervalsKernel(
		KeyType *d_DstKey,
		ValueType *d_DstVal,
		KeyType *d_SrcKey,
		ValueType *d_SrcVal,
		uint *d_LimitsA,
		uint *d_LimitsB,
		uint stride,
		uint N
		)
	{
		__shared__ KeyType s_key[2 * SAMPLE_STRIDE];
		__shared__ ValueType s_val[2 * SAMPLE_STRIDE];

		const uint intervalI = blockIdx.x & ((2 * stride) / SAMPLE_STRIDE - 1);
		const uint segmentBase = (blockIdx.x - intervalI) * SAMPLE_STRIDE;
		d_SrcKey += segmentBase;
		d_SrcVal += segmentBase;
		d_DstKey += segmentBase;
		d_DstVal += segmentBase;

		//Set up threadblock-wide parameters
		__shared__ uint startSrcA, startSrcB, lenSrcA, lenSrcB, startDstA, startDstB;

		if (threadIdx.x == 0)
		{
			uint segmentElementsA = stride;
			uint segmentElementsB = umin(stride, N - segmentBase - stride);
			uint segmentSamplesA = getSampleCount(segmentElementsA);
			uint segmentSamplesB = getSampleCount(segmentElementsB);
			uint segmentSamples = segmentSamplesA + segmentSamplesB;

			startSrcA = d_LimitsA[blockIdx.x];
			startSrcB = d_LimitsB[blockIdx.x];
			uint endSrcA = (intervalI + 1 < segmentSamples) ? d_LimitsA[blockIdx.x + 1] : segmentElementsA;
			uint endSrcB = (intervalI + 1 < segmentSamples) ? d_LimitsB[blockIdx.x + 1] : segmentElementsB;
			lenSrcA = endSrcA - startSrcA;
			lenSrcB = endSrcB - startSrcB;
			startDstA = startSrcA + startSrcB;
			startDstB = startDstA + lenSrcA;
		}

		//Load main input data
		__syncthreads();

		if (threadIdx.x < lenSrcA)
		{
			s_key[threadIdx.x + 0] = d_SrcKey[0 + startSrcA + threadIdx.x];
			s_val[threadIdx.x + 0] = d_SrcVal[0 + startSrcA + threadIdx.x];
		}

		if (threadIdx.x < lenSrcB)
		{
			s_key[threadIdx.x + SAMPLE_STRIDE] = d_SrcKey[stride + startSrcB + threadIdx.x];
			s_val[threadIdx.x + SAMPLE_STRIDE] = d_SrcVal[stride + startSrcB + threadIdx.x];
		}

		//Merge data in shared memory
		__syncthreads();
		merge<KeyType, ValueType, sortDir>(
			s_key,
			s_val,
			s_key + 0,
			s_val + 0,
			s_key + SAMPLE_STRIDE,
			s_val + SAMPLE_STRIDE,
			lenSrcA, SAMPLE_STRIDE,
			lenSrcB, SAMPLE_STRIDE
			);

		//Store merged data
		__syncthreads();

		if (threadIdx.x < lenSrcA)
		{
			d_DstKey[startDstA + threadIdx.x] = s_key[threadIdx.x];
			d_DstVal[startDstA + threadIdx.x] = s_val[threadIdx.x];
		}

		if (threadIdx.x < lenSrcB)
		{
			d_DstKey[startDstB + threadIdx.x] = s_key[lenSrcA + threadIdx.x];
			d_DstVal[startDstB + threadIdx.x] = s_val[lenSrcA + threadIdx.x];
		}
	}

	template<class KeyType, class ValueType> class MergeSort
	{
		uint *d_RanksA, *d_RanksB, *d_LimitsA, *d_LimitsB;
		DeviceArray<char> merge_sort_buf;
		////////////////////////////////////////////////////////////////////////////////
		// Bottom-level merge sort (binary search-based)
		////////////////////////////////////////////////////////////////////////////////
		void mergeSortShared(
			KeyType *d_DstKey,
			ValueType *d_DstVal,
			KeyType *d_SrcKey,
			ValueType *d_SrcVal,
			uint batchSize,
			uint arrayLength,
			uint sortDir
			)
		{
			if (arrayLength < 2)
			{
				return;
			}

			assert(SHARED_SIZE_LIMIT % arrayLength == 0);
			assert(((batchSize * arrayLength) % SHARED_SIZE_LIMIT) == 0);
			uint  blockCount = batchSize * arrayLength / SHARED_SIZE_LIMIT;
			uint threadCount = SHARED_SIZE_LIMIT / 2;

			if (sortDir)
			{
				mergeSortSharedKernel<KeyType, ValueType, 1U> << <blockCount, threadCount >> >(
					d_DstKey, d_DstVal, d_SrcKey, d_SrcVal, arrayLength);
				cudaSafeCall(hipGetLastError(), "mergeSortShared<1><<<>>> failed\n");
			}
			else
			{
				mergeSortSharedKernel<KeyType, ValueType, 0U> << <blockCount, threadCount >> >(
					d_DstKey, d_DstVal, d_SrcKey, d_SrcVal, arrayLength);
				cudaSafeCall(hipGetLastError(), "mergeSortShared<0><<<>>> failed\n");
			}
		}


		////////////////////////////////////////////////////////////////////////////////
		// Merge step 1: generate sample ranks
		////////////////////////////////////////////////////////////////////////////////

		void generateSampleRanks(
			uint *d_RanksA,
			uint *d_RanksB,
			KeyType *d_SrcKey,
			uint stride,
			uint N,
			uint sortDir
			)
		{
			uint lastSegmentElements = N % (2 * stride);
			uint threadCount = (lastSegmentElements > stride) ?
				(N + 2 * stride - lastSegmentElements) / (2 * SAMPLE_STRIDE) :
				(N - lastSegmentElements) / (2 * SAMPLE_STRIDE);

			if (sortDir)
			{
				generateSampleRanksKernel<KeyType, 1U> << <iDivUp(threadCount, 256), 256 >> >(
					d_RanksA, d_RanksB, d_SrcKey, stride, N, threadCount);
				cudaSafeCall(hipGetLastError(), "generateSampleRanksKernel<1U><<<>>> failed\n");
			}
			else
			{
				generateSampleRanksKernel<KeyType, 0U> << <iDivUp(threadCount, 256), 256 >> >(
					d_RanksA, d_RanksB, d_SrcKey, stride, N, threadCount);
				cudaSafeCall(hipGetLastError(), "generateSampleRanksKernel<0U><<<>>> failed\n");
			}
		}

		////////////////////////////////////////////////////////////////////////////////
		// Merge step 2: generate sample ranks and indices
		////////////////////////////////////////////////////////////////////////////////

		void mergeRanksAndIndices(
			uint *d_LimitsA,
			uint *d_LimitsB,
			uint *d_RanksA,
			uint *d_RanksB,
			uint stride,
			uint N
			)
		{
			uint lastSegmentElements = N % (2 * stride);
			uint threadCount = (lastSegmentElements > stride) ?
				(N + 2 * stride - lastSegmentElements) / (2 * SAMPLE_STRIDE) :
				(N - lastSegmentElements) / (2 * SAMPLE_STRIDE);

			mergeRanksAndIndicesKernel << <iDivUp(threadCount, 256), 256 >> >(
				d_LimitsA,
				d_RanksA,
				stride,
				N,
				threadCount
				);
			cudaSafeCall(hipGetLastError(), "mergeRanksAndIndicesKernel(A)<<<>>> failed\n");

			mergeRanksAndIndicesKernel << <iDivUp(threadCount, 256), 256 >> >(
				d_LimitsB,
				d_RanksB,
				stride,
				N,
				threadCount
				);
			cudaSafeCall(hipGetLastError(), "mergeRanksAndIndicesKernel(B)<<<>>> failed\n");
		}

		////////////////////////////////////////////////////////////////////////////////
		// Merge step 3: merge elementary intervals
		////////////////////////////////////////////////////////////////////////////////

		void mergeElementaryIntervals(
			KeyType *d_DstKey,
			ValueType *d_DstVal,
			KeyType *d_SrcKey,
			ValueType *d_SrcVal,
			uint *d_LimitsA,
			uint *d_LimitsB,
			uint stride,
			uint N,
			uint sortDir
			)
		{
			uint lastSegmentElements = N % (2 * stride);
			uint mergePairs = (lastSegmentElements > stride) ? getSampleCount(N) :
				(N - lastSegmentElements) / SAMPLE_STRIDE;

			if (sortDir)
			{
				mergeElementaryIntervalsKernel<KeyType, ValueType, 1U> << <mergePairs, SAMPLE_STRIDE >> >(
					d_DstKey,
					d_DstVal,
					d_SrcKey,
					d_SrcVal,
					d_LimitsA,
					d_LimitsB,
					stride,
					N
					);
				cudaSafeCall(hipGetLastError(), "mergeElementaryIntervalsKernel<1> failed\n");
			}
			else
			{
				mergeElementaryIntervalsKernel<KeyType, ValueType, 0U> << <mergePairs, SAMPLE_STRIDE >> >(
					d_DstKey,
					d_DstVal,
					d_SrcKey,
					d_SrcVal,
					d_LimitsA,
					d_LimitsB,
					stride,
					N
					);
				cudaSafeCall(hipGetLastError(), "mergeElementaryIntervalsKernel<0> failed\n");
			}
		}


		void initMergeSort(void)
		{
			cudaSafeCall(hipMalloc((void **)&d_RanksA, MAX_SAMPLE_COUNT * sizeof(uint)));
			cudaSafeCall(hipMalloc((void **)&d_RanksB, MAX_SAMPLE_COUNT * sizeof(uint)));
			cudaSafeCall(hipMalloc((void **)&d_LimitsA, MAX_SAMPLE_COUNT * sizeof(uint)));
			cudaSafeCall(hipMalloc((void **)&d_LimitsB, MAX_SAMPLE_COUNT * sizeof(uint)));
		}

		void closeMergeSort(void)
		{
			cudaSafeCall(hipFree(d_RanksA));
			cudaSafeCall(hipFree(d_RanksB));
			cudaSafeCall(hipFree(d_LimitsB));
			cudaSafeCall(hipFree(d_LimitsA));
		}

	public:
		MergeSort()
		{
			initMergeSort();
		}
		~MergeSort()
		{
			closeMergeSort();
		}
		void mergeSort(
			KeyType *d_DstKey,
			ValueType *d_DstVal,
			KeyType *d_SrcKey,
			ValueType *d_SrcVal,
			uint N,
			uint sortDir
			)
		{
			if (N*(sizeof(KeyType)+sizeof(ValueType)) > merge_sort_buf.size())
				merge_sort_buf.create(N*(sizeof(ValueType)+sizeof(KeyType))*1.5);

			KeyType* d_BufKey = (KeyType*)merge_sort_buf.ptr();
			ValueType* d_BufVal = (ValueType*)(merge_sort_buf.ptr()+N*sizeof(KeyType));

			uint stageCount = 0;

			for (uint stride = SHARED_SIZE_LIMIT; stride < N; stride <<= 1, stageCount++);

			KeyType *ikey, *okey;
			ValueType *ival, *oval;

			if (stageCount & 1)
			{
				ikey = d_BufKey;
				ival = d_BufVal;
				okey = d_DstKey;
				oval = d_DstVal;
			}
			else
			{
				ikey = d_DstKey;
				ival = d_DstVal;
				okey = d_BufKey;
				oval = d_BufVal;
			}

			assert(N <= (SAMPLE_STRIDE * MAX_SAMPLE_COUNT));
			assert(N % SHARED_SIZE_LIMIT == 0);
			mergeSortShared(ikey, ival, d_SrcKey, d_SrcVal, N / SHARED_SIZE_LIMIT, SHARED_SIZE_LIMIT, sortDir);

			for (uint stride = SHARED_SIZE_LIMIT; stride < N; stride <<= 1)
			{
				uint lastSegmentElements = N % (2 * stride);

				//Find sample ranks and prepare for limiters merge
				generateSampleRanks(d_RanksA, d_RanksB, ikey, stride, N, sortDir);

				//Merge ranks and indices
				mergeRanksAndIndices(d_LimitsA, d_LimitsB, d_RanksA, d_RanksB, stride, N);

				//Merge elementary intervals
				mergeElementaryIntervals(okey, oval, ikey, ival, d_LimitsA, d_LimitsB, stride, N, sortDir);

				if (lastSegmentElements <= stride)
				{
					//Last merge segment consists of a single array which just needs to be passed through
					cudaSafeCall(hipMemcpy(okey + (N - lastSegmentElements), ikey + (N - lastSegmentElements), lastSegmentElements * sizeof(uint), hipMemcpyDeviceToDevice));
					cudaSafeCall(hipMemcpy(oval + (N - lastSegmentElements), ival + (N - lastSegmentElements), lastSegmentElements * sizeof(uint), hipMemcpyDeviceToDevice));
				}

				KeyType *t;
				t = ikey;
				ikey = okey;
				okey = t;
				ValueType* t1;
				t1 = ival;
				ival = oval;
				oval = t1;
			}
		}
	};

	//MergeSort<int, int> g_merge_sort_i_i;
	//MergeSort<int, float> g_merge_sort_i_f;
	//MergeSort<float, int> g_merge_sort_f_i;
	//DeviceArray<char> g_merge_sort_buf;

	void mergeSort(const int* key_in, const int* val_in, int* key_out, int* val_out, int n, bool less)
	{
		//g_merge_sort_i_i.mergeSort((int*)key_out, (int*)val_out, (int*)key_in, (int*)val_in, n, less);
	}
	void mergeSort(const int* key_in, const float* val_in, int* key_out, float* val_out, int n, bool less)
	{
		//g_merge_sort_i_f.mergeSort((int*)key_out, (float*)val_out, (int*)key_in, (float*)val_in, n, less);
	}
	void mergeSort(const float* key_in, const int* val_in, float* key_out, int* val_out, int n, bool less)
	{
		//g_merge_sort_f_i.mergeSort((float*)key_out, (int*)val_out, (float*)key_in, (int*)val_in, n, less);
	}
#pragma endregion

}