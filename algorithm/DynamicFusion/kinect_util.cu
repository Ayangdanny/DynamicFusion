#include "hip/hip_runtime.h"
#include "kinect_util.h"
#include "device_utils.h"
#include <set>
namespace dfusion
{
#pragma region --copy_kernel

	__global__ void copy_colormap_kernel(PtrStepSz<PixelRGBA> src,
		PtrStepSz<uchar4> dst)
	{
		int u = threadIdx.x + blockIdx.x * blockDim.x;
		int v = threadIdx.y + blockIdx.y * blockDim.y;

		if (u >= dst.cols || v >= dst.rows)
			return;

		PixelRGBA p = src(v, u);
		dst(v, u) = make_uchar4(p.r, p.g, p.b, p.a);
	}

	void copyColorMapToPbo(PtrStepSz<PixelRGBA> src, PtrStepSz<uchar4> dst)
	{
		dim3 block(32, 8);
		dim3 grid(1, 1, 1);
		grid.x = divUp(dst.cols, block.x);
		grid.y = divUp(dst.rows, block.y);

		copy_colormap_kernel << <grid, block >> >(src, dst);
		cudaSafeCall(hipGetLastError(), "copyColorMapToPbo");
	}
#pragma endregion

#pragma region --calculate jet
	/////////////////////////////////////////////////////////////////////////////////
	// jet calculation, for visualization
	__constant__ float g_jet_w[9][6] =
	{
		{ 0.0f, 0.0f, 0.0f, 0.0f, 0.5f, 0.5f },
		{ 0.0f, 0.0f, 0.5f, -0.5f, 0.0f, 1.0f },
		{ 0.0f, 0.0f, 0.5f, -0.5f, 0.0f, 1.0f },
		{ 0.5f, -1.5f, 0.0f, 1.0f, -0.5f, 2.5f },
		{ 0.5f, -1.5f, 0.0f, 1.0f, -0.5f, 2.5f },
		{ 0.0f, 1.0f, -0.5f, 3.5f, 0.0f, 0.0f },
		{ 0.0f, 1.0f, -0.5f, 3.5f, 0.0f, 0.0f },
		{ -0.5f, 4.5f, 0.0f, 0.0f, 0.0f, 0.0f },
		{ -0.5f, 4.5f, 0.0f, 0.0f, 0.0f, 0.0f }
	};

	__global__ void calcTemperatureJetKernel(PtrStepSz<depthtype> depth_d, PtrStepSz<uchar4> jetRgb_d, float shift, float div)
	{
		int u = threadIdx.x + blockIdx.x * blockDim.x;
		int v = threadIdx.y + blockIdx.y * blockDim.y;

		if (u >= depth_d.cols || v >= depth_d.rows)
			return;
		float val = float(depth_d(v, u)-shift)/div;
		val = 8.f * min(1.f, max(0.f, val));
		const float* const c = g_jet_w[int(val)];
		jetRgb_d(v, u) = make_uchar4(255 * (c[0] * val + c[1]), 255 * (c[2] * val + c[3]), 255 * (c[4] * val + c[5]), 255);
	}

	void calc_temperature_jet(PtrStepSz<depthtype> depth_d, PtrStepSz<uchar4> jetRgb_d, float shift, float div)
	{
		dim3 block(32, 8);
		dim3 grid(1, 1, 1);
		grid.x = divUp(jetRgb_d.cols, block.x);
		grid.y = divUp(jetRgb_d.rows, block.y);

		calcTemperatureJetKernel << <grid, block >> >(depth_d, jetRgb_d, shift, div);
		cudaSafeCall(hipGetLastError());
	}

#pragma endregion

#pragma region --generate image
	struct ImageGenerator
	{
		enum
		{
			CTA_SIZE_X = 32, CTA_SIZE_Y = 8
		};

		PtrStep<float> vmap;
		PtrStep<float> nmap;

		LightSource light;

		mutable PtrStepSz<PixelRGBA> dst;

		__device__ __forceinline__ void convertToColorHSV(float rgb[3], float value, float min_value, float max_value) const{
			if (value < min_value)
				value = min_value;
			else if (value > max_value)
				value = max_value;

			float lamda = (value - min_value) / (max_value - min_value);

			float coef = lamda * 6.0f;

			if (coef <= 1.0f)
				rgb[0] = 1.0f, rgb[1] = coef, rgb[2] = 0.0f;
			else if (coef <= 2.0f)
				rgb[0] = 1.0f - (coef - 1.0f), rgb[1] = 1.0f, rgb[2] = 0.0f;
			else if (coef <= 3.0f)
				rgb[0] = 0.0f, rgb[1] = 1.0f, rgb[2] = coef - 2.0f;
			else if (coef <= 4.0f)
				rgb[0] = 0.0f, rgb[1] = 1.0f - (coef - 3.0f), rgb[2] = 1.0f;
			else if (coef <= 5.0f)
				rgb[0] = coef - 4.0f, rgb[1] = 0.0f, rgb[2] = 1.0f;
			else
				rgb[0] = 1.0f, rgb[2] = 0.0f, rgb[2] = 1.0f - (coef - 5.0f);
		}

		__device__ __forceinline__ float int2float(int value) const
		{
			value = (value * 179426549 + 1300997) & 15487469;
			return float(value) / 15487469;
		}

		__device__ __forceinline__ void operator () () const
		{
				int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
				int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

				if (x >= dst.cols || y >= dst.rows)
					return;

				float3 v, n;
				v.x = vmap.ptr(y)[x];
				n.x = nmap.ptr(y)[x];

				PixelRGBA color;
				color.a = color.r = color.g = color.b = 0;

				if (!isnan(v.x) && !isnan(n.x))
				{
					v.y = vmap.ptr(y + dst.rows)[x];
					v.z = vmap.ptr(y + 2 * dst.rows)[x];

					n.y = nmap.ptr(y + dst.rows)[x];
					n.z = nmap.ptr(y + 2 * dst.rows)[x];

					float3 acc_vec = make_float3(0.f, 0.f, 0.f);
					{
						float3 vec = normalized(light.pos - v);
						float w = max(0.f, dot(vec, n));
						acc_vec.x += w * light.diffuse.x;
						acc_vec.y += w * light.diffuse.y;
						acc_vec.z += w * light.diffuse.z;
					}
					color.r = max(0, min(255, int(acc_vec.x*255.f)));
					color.g = max(0, min(255, int(acc_vec.y*255.f)));
					color.b = max(0, min(255, int(acc_vec.z*255.f)));
					color.a = 255;
				}

				dst.ptr(y)[x] = color;
			}
	};

	__global__ void generateImageKernel(const ImageGenerator ig)
	{
		ig();
	}

	void generateImage(const MapArr& vmap, const MapArr& nmap, ColorMap& dst, const LightSource& light)
	{
		ImageGenerator ig;
		ig.vmap = vmap;
		ig.nmap = nmap;
		ig.light = light;
		ig.dst = dst;

		dim3 block(ImageGenerator::CTA_SIZE_X, ImageGenerator::CTA_SIZE_Y);
		dim3 grid(divUp(dst.cols(), block.x), divUp(dst.rows(), block.y));

		generateImageKernel << <grid, block >> >(ig);
		cudaSafeCall(hipGetLastError(), "generateImage");
	}

#pragma endregion

#pragma region --generate normal
	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	struct NormalGenerator
	{
		enum
		{
			CTA_SIZE_X = 32, CTA_SIZE_Y = 8
		};

		PtrStep<float> nmap;
		Mat33 R;
		mutable PtrStepSz<PixelRGBA> dst;

		__device__ __forceinline__ void operator () () const
		{
			int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
			int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

			if (x >= dst.cols || y >= dst.rows)
				return;

			float3 n;
			n.x = nmap.ptr(y)[x];

			PixelRGBA color;
			color.a = color.r = color.g = color.b = 0;

			if (!isnan(n.x))
			{
				n.y = nmap.ptr(y + dst.rows)[x];
				n.z = nmap.ptr(y + 2 * dst.rows)[x];
				n = normalized(R*n);
				color.r = max(0, min(255, int(255 * (n.x * 0.5f + 0.5f))));
				color.g = max(0, min(255, int(255 * (n.y * 0.5f + 0.5f))));
				color.b = max(0, min(255, int(255 * (n.z * 0.5f + 0.5f))));
				color.a = 255;
			}

			dst.ptr(y)[x] = color;
		}
	};

	__global__ void generateNormalKernel(const NormalGenerator ig)
	{
		ig();
	}

	void generateNormalMap(const MapArr& nmap, ColorMap& dst, Mat33 R)
	{
		NormalGenerator ig;
		ig.nmap = nmap;
		ig.dst = dst;
		ig.R = R;


		dim3 block(NormalGenerator::CTA_SIZE_X, NormalGenerator::CTA_SIZE_Y);
		dim3 grid(divUp(dst.cols(), block.x), divUp(dst.rows(), block.y));

		generateNormalKernel << <grid, block >> >(ig);
		cudaSafeCall(hipGetLastError(), "generateNormal");
	}

#pragma endregion


}